#include "hip/hip_runtime.h"
#include "global_defines.cuh"
#include <cstdio>
#include <cstdlib>


void temp_compare(FLOATING *a, FLOATING *b){
	int x,y,z;
	int missed=0;
	int lx=680, ly=73, lz=73;

	for (z = 0 ; z< lz ; ++z){
		for (y = 0 ; y< ly ; ++y){
			for (x = 0 ; x< lx; ++x){
				if(abs(a[index(z,y,x)]-b[index(z,y,x)])>0.00001)
					++missed;

			}
		}
	}
	cout <<"totally missed:" << missed << endl;

	cout << " FULL MATCH!" << endl;
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
			{ 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
			{ 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
			{ 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
			{ 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
			{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
			{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
			{ 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
			{ 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
			{   -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one to run properly
	printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[7].Cores);
	return nGpuArchCoresPerSM[7].Cores;
}
// end of GPU Architecture definitions


void cuda_device_querry(){
	cout << "CUDA DEVISE TEST - START" << endl;

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		cout <<"\tcudaGetDeviceCount returned" <<(int)error_id  << " %d\n-> " <<  hipGetErrorString(error_id) << endl;
		cout <<"\tResult = FAIL\n" << endl;
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		cout << "\tThere are no available device(s) that support CUDA\n" << endl;
	}
	else
	{
		cout << "\tDetected " << deviceCount << " CUDA Capable device(s)\n" << endl;
	}

	int dev, driverVersion = 0, runtimeVersion = 0;

	for (dev = 0; dev < deviceCount; ++dev){
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		printf("\tDevice %d: \"%s\"\n", dev, deviceProp.name);

		cout <<"\tDevice " << dev << ": \" " << deviceProp.name <<    " \" "<<endl;


		// Console log
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("\tCUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
		printf("\tCUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		char msg[256];
		sprintf(msg, "\tTotal amount of global memory:                 %.0f MBytes (%llu bytes)\n",
				(float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
		printf("\t%s", msg);

		printf("\t(%2d) Multiprocessors x (%3d) CUDA Cores/MP:    %d CUDA Cores\n",
				deviceProp.multiProcessorCount,
				_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
				_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
		printf("\tGPU Clock rate:                                %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);


		printf("\tMax Texture Dimension Size (x,y,z)             1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
				deviceProp.maxTexture1D   , deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
				deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
		printf("\tMax Layered Texture Size (dim) x layers        1D=(%d) x %d, 2D=(%d,%d) x %d\n",
				deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
				deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);

		printf("\tTotal amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
		printf("\tTotal amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
		printf("\tTotal number of registers available per block: %d\n", deviceProp.regsPerBlock);
		printf("\tWarp size:                                     %d\n", deviceProp.warpSize);
		printf("\tMaximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
		printf("\tMaximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
		printf("\tMaximum sizes of each dimension of a block:    %d x %d x %d\n",
				deviceProp.maxThreadsDim[0],
				deviceProp.maxThreadsDim[1],
				deviceProp.maxThreadsDim[2]);
		printf("\tMaximum sizes of each dimension of a grid:     %d x %d x %d\n",
				deviceProp.maxGridSize[0],
				deviceProp.maxGridSize[1],
				deviceProp.maxGridSize[2]);
		printf("\tMaximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
		printf("\tTexture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
		printf("\tConcurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
		printf("\tRun time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
		printf("\tIntegrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
		printf("\tSupport host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
		printf("\tAlignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
		printf("\tDevice has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");

		printf("\tDevice supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
		printf("\tDevice PCI Bus ID / PCI location ID:           %d / %d\n", deviceProp.pciBusID, deviceProp.pciDeviceID);

		const char *sComputeMode[] =
		{
				"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
				"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
				"Prohibited (no host thread can use ::hipSetDevice() with this device)",
				"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
				"Unknown",
				NULL
		};
		printf("\tCompute Mode:\n");
		printf("\t\t< %s >\n", sComputeMode[deviceProp.computeMode]);
	}


	// csv masterlog info
	// *****************************
	// exe and CUDA driver name
	printf("\n");
	std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
	char cTemp[16];

	// driver version
	sProfileString += ", CUDA Driver Version = ";
#ifdef WIN32
	sprintf_s(cTemp, 10, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#else
	sprintf(cTemp, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#endif
	sProfileString +=  cTemp;

	// Runtime version
	sProfileString += ", CUDA Runtime Version = ";
#ifdef WIN32
	sprintf_s(cTemp, 10, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#else
	sprintf(cTemp, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#endif
	sProfileString +=  cTemp;

	// Device count
	sProfileString += ", NumDevs = ";
#ifdef WIN32
	sprintf_s(cTemp, 10, "%d", deviceCount);
#else
	sprintf(cTemp, "%d", deviceCount);
#endif
	sProfileString += cTemp;

	// Print Out all device Names
	for (dev = 0; dev < deviceCount; ++dev)
	{
#ifdef _WIN32
		sprintf_s(cTemp, 13, ", Device%d = ", dev);
#else
		sprintf(cTemp, ", Device%d = ", dev);
#endif
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		sProfileString += cTemp;
		sProfileString += deviceProp.name;
	}

	sProfileString += "\n";
	printf("%s", sProfileString.c_str());

	printf("\tResult = PASS\n");

	// finish

	cout << "CUDA DEVISE TEST - END" << endl;
}

void read_external_geometry_file_specification_for_LBM(int &lx, int &ly, int &lz, int &n_of_densities, const string filename){
	vector<string> geometry_parameters;
	ifstream conf_file(filename.c_str());
	string buff;
	if(conf_file.is_open()){
		while(conf_file>>buff){
			geometry_parameters.push_back(buff);
		}
		cout << "Geometry Parameters Read:" << endl;
		lx=atoi(geometry_parameters[0].c_str());
		cout << "\t domain length in X: " << lx << endl;

		ly=atoi(geometry_parameters[1].c_str());
		cout << "\t domain length in Y: " << ly << endl;

		lz=atoi(geometry_parameters[2].c_str());
		cout << "\t domain length in Z: " << lz << endl;

		n_of_densities=atoi(geometry_parameters[3].c_str());
		cout << "\t number of densities on each node: " << n_of_densities << endl;

		cout <<"total:" << geometry_parameters.size() << " parameters were read" << endl;
		conf_file.close();
	}else{
		cout << "The file "<< filename << " was not found" << endl;
		cout << "Create a new file at the root directory with 3 lines (one number on every line), each corresponding to the respective dimension of X,Y,Z" << endl;
		exit (-1);
	}
}


lattice::lattice(int LX,int LY, int LZ):
																																																		lx(LX), ly(LY), lz(LZ){
	Q0=new FLOATING[lz*ly*lx];
	Q1=new FLOATING[lz*ly*lx];
	Q2=new FLOATING[lz*ly*lx];
	Q3=new FLOATING[lz*ly*lx];
	Q4=new FLOATING[lz*ly*lx];
	Q5=new FLOATING[lz*ly*lx];
	Q6=new FLOATING[lz*ly*lx];
	Q7=new FLOATING[lz*ly*lx];
	Q8=new FLOATING[lz*ly*lx];
	Q9=new FLOATING[lz*ly*lx];
	Q10=new FLOATING[lz*ly*lx];
	Q11=new FLOATING[lz*ly*lx];
	Q12=new FLOATING[lz*ly*lx];
	Q13=new FLOATING[lz*ly*lx];
	Q14=new FLOATING[lz*ly*lx];
	Q15=new FLOATING[lz*ly*lx];
	Q16=new FLOATING[lz*ly*lx];
	Q17=new FLOATING[lz*ly*lx];
	Q18=new FLOATING[lz*ly*lx];

	initialise(Q0);
	initialise(Q1);
	initialise(Q2);
	initialise(Q3);
	initialise(Q4);
	initialise(Q5);
	initialise(Q6);
	initialise(Q7);
	initialise(Q8);
	initialise(Q9);
	initialise(Q10);
	initialise(Q11);
	initialise(Q12);
	initialise(Q13);
	initialise(Q14);
	initialise(Q15);
	initialise(Q16);
	initialise(Q17);
	initialise(Q18);
}

lattice::lattice(int LX,int LY, int LZ, int dump):
																																				lx(LX), ly(LY), lz(LZ){
	int FLOATING_array_size=lx*ly*lz*sizeof(FLOATING);
	hipMalloc((void **)&Q0, FLOATING_array_size);
	hipMalloc((void **)&Q1, FLOATING_array_size);
	hipMalloc((void **)&Q2, FLOATING_array_size);
	hipMalloc((void **)&Q3, FLOATING_array_size);
	hipMalloc((void **)&Q4, FLOATING_array_size);
	hipMalloc((void **)&Q5, FLOATING_array_size);
	hipMalloc((void **)&Q6, FLOATING_array_size);
	hipMalloc((void **)&Q7, FLOATING_array_size);
	hipMalloc((void **)&Q8, FLOATING_array_size);
	hipMalloc((void **)&Q9, FLOATING_array_size);
	hipMalloc((void **)&Q10, FLOATING_array_size);
	hipMalloc((void **)&Q11, FLOATING_array_size);
	hipMalloc((void **)&Q12, FLOATING_array_size);
	hipMalloc((void **)&Q13, FLOATING_array_size);
	hipMalloc((void **)&Q14, FLOATING_array_size);
	hipMalloc((void **)&Q15, FLOATING_array_size);
	hipMalloc((void **)&Q16, FLOATING_array_size);
	hipMalloc((void **)&Q17, FLOATING_array_size);
	hipMalloc((void **)&Q18, FLOATING_array_size);
}
lattice::~lattice(){
	delete [] Q0;
	delete [] Q1;
	delete [] Q2;
	delete [] Q3;
	delete [] Q4;
	delete [] Q5;
	delete [] Q6;
	delete [] Q7;
	delete [] Q8;
	delete [] Q9;
	delete [] Q10;
	delete [] Q11;
	delete [] Q12;
	delete [] Q13;
	delete [] Q14;
	delete [] Q15;
	delete [] Q16;
	delete [] Q17;
	delete [] Q18;

	printf("host memories deleted!\n");
}

void lattice::initialise(FLOATING *Q){

	for(int z=0; z<lz; ++z)
		for(int y=0; y<ly; ++y)
			for(int x=0; x<lx; ++x)
				Q[ index(z,y,x)]=0.0;
}

void LBM::create_an_example_configuration_files(const string filename){
	ofstream example_file(filename.c_str());

	if ( example_file.is_open()){
		example_file << "10" << endl;
		example_file << "100" << endl;
		example_file << "0.0175" << endl;
		example_file << "7" << endl;
		example_file << "100" << endl;
		example_file << "26" << endl;
		example_file << "59" << endl;
		example_file << "512" << endl;
		example_file << "datum_design_case_name1" << endl;
		example_file.close();
	}
}

void LBM::display_the_structure_of_an_example_configuration_file(){
	cout<< "Create a new file (with 9 lines) at the root directory following the template below:"
			<< endl;
	cout << "10 <--line 1: number of iterations" << endl;
	cout << "100 <--line 2: check frequency" << endl;
	cout << "0.0175 <--line 3: nu" << endl;
	cout << "7 <--line 4: r_small (from baffle geometry)" << endl;
	cout << "100 <--line 5: Reynolds Number" << endl;
	cout << "26 <--line 6: S (from baffle geometry)" << endl;
	cout << "59 <--line 7: baffle possition" << endl;
	cout << "512 <--line 8: CUDA threads per kernel" << endl;
	cout << "datum_design <--line 9: case name (ONE WORD!)" << endl;
}

void LBM::read_external_configuration_file_for_the_solver(const string filename) {
	vector<string> configuration_parameters;
	ifstream conf_file(filename.c_str());
	string buff;
	if (conf_file.is_open()) {
		while (conf_file >> buff) {
			configuration_parameters.push_back(buff);
		}
		cout << "Configuration Parameters Read:" << endl;
		max_iterations = atoi(configuration_parameters[0].c_str());
		cout << "\titerations: " << max_iterations << endl;
		//check step:perform check_density and export

		check_step = atoi(configuration_parameters[1].c_str());
		cout << "\tcheck step: " << check_step << endl;

		nu = atof(configuration_parameters[2].c_str());
		cout << "\tnu: " << nu << endl;

		r_small = atof(configuration_parameters[3].c_str());
		cout << "\tr_small: " << r_small << endl;

		reynolds = atof(configuration_parameters[4].c_str());
		cout << "\treynolds: " << reynolds << endl;

		s = atof(configuration_parameters[5].c_str());
		cout << "\ts: " << s << endl;

		baffle = atoi(configuration_parameters[6].c_str());
		cout << "\tbaffle position on X=" << baffle << endl;

		threads_per_kernel = atoi(configuration_parameters[7].c_str());
		cout << "\tCUDA threads per kernel: " << threads_per_kernel << endl;

		case_name = configuration_parameters[8].c_str();
		cout << "Case: " << case_name << endl;

		cout << "total:" << configuration_parameters.size()
								<< " parameters were read" << endl;
		conf_file.close();
	} else {
		cout << "The file "<<filename <<" was not found" << endl;
		display_the_structure_of_an_example_configuration_file();
		create_an_example_configuration_files(filename);
		exit(-2);
	}
}

void LBM::delete_device_data(){

	hipFree(D3_d.Q0);
	hipFree(D3_d.Q1);
	hipFree(D3_d.Q2);
	hipFree(D3_d.Q3);
	hipFree(D3_d.Q4);
	hipFree(D3_d.Q5);
	hipFree(D3_d.Q6);
	hipFree(D3_d.Q7);
	hipFree(D3_d.Q8);
	hipFree(D3_d.Q9);
	hipFree(D3_d.Q10);
	hipFree(D3_d.Q11);
	hipFree(D3_d.Q12);
	hipFree(D3_d.Q13);
	hipFree(D3_d.Q14);
	hipFree(D3_d.Q15);
	hipFree(D3_d.Q16);
	hipFree(D3_d.Q17);
	hipFree(D3_d.Q18);

	hipFree(D3_hlp_d.Q0);
	hipFree(D3_hlp_d.Q1);
	hipFree(D3_hlp_d.Q2);
	hipFree(D3_hlp_d.Q3);
	hipFree(D3_hlp_d.Q4);
	hipFree(D3_hlp_d.Q5);
	hipFree(D3_hlp_d.Q6);
	hipFree(D3_hlp_d.Q7);
	hipFree(D3_hlp_d.Q8);
	hipFree(D3_hlp_d.Q9);
	hipFree(D3_hlp_d.Q10);
	hipFree(D3_hlp_d.Q11);
	hipFree(D3_hlp_d.Q12);
	hipFree(D3_hlp_d.Q13);
	hipFree(D3_hlp_d.Q14);
	hipFree(D3_hlp_d.Q15);
	hipFree(D3_hlp_d.Q16);
	hipFree(D3_hlp_d.Q17);
	hipFree(D3_hlp_d.Q18);

	hipFree(u_current_d);
	hipFree(u_current_temp_d);
	hipFree(v_current_d);
	hipFree(w_current_d);

	hipFree(u_previous_spatial_boundary_d);
	hipFree(v_previous_spatial_boundary_d);
	hipFree(w_previous_spatial_boundary_d);

	hipFree(u_previous_temporal_boundary_d);
	hipFree(v_previous_temporal_boundary_d);
	hipFree(w_previous_temporal_boundary_d);

	hipFree(temp_check_density_d);
	hipFree(temp_check_density_d_full);

	hipFree(obstacles_d);

	hipFree(temp_Uc_d);


	printf("cuda memories deleted!\n");
}


__global__
void cuda_initialise_array(FLOATING *input_array, const int length, const FLOATING value){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	input_array[tid]=value;
	__syncthreads();

}



template <class T>
void LBM::initialise_array(T *array, const int length,const T init_value){

	for(int i=0; i<length; ++i)
		array[i]=init_value;

}


void LBM::abstract_initialise(){
	//objectives


	FLOATING temp_density=0.0;

	initialise_microscopic_density_arrays_in_the_host();

	calculate_macroscopic_density_in_the_host(temp_density);

	create_reactor_geometry_in_the_host();

#ifdef PRODUCE_OUTPUT_FILES
	geometry_file_in_VTK();
#endif //PRODUCE_OUTPUT_FILES
	cout << "0th loop" <<endl;
	//myLBM.initial_redistribute();
	fortran_redistribute(0);
	//first loop!!!
	calculate_macroscopic_density_in_the_host(temp_density);
	streaming();
	bounceback();
	initial_relaxation();
	convective_BC();
	cout <<"# iteration 0" << endl;

	count_no_obstacles_at_penultimate_x_slice();
	copy_data_from_host_to_device();

}

void LBM::abstract_check_density(){
	//check density
	if( time_unit%check_step==0){
#ifdef CPU_part
		calculate_macroscopic_density_in_the_host(density);
#endif
#ifdef GPU_part
		cuda_check_density(time_unit);
#endif
	}
}


void LBM::abstract_debug_computations(){
#ifdef DEBUG
	compare_obstacles(obstacles);
	compare_nodes_hlp(n_hlp);
	compare_nodes(node);
#endif
}

void LBM::abstract_redistribute(){
	//redistribute
#ifdef CPU_part
	redistribute();
#endif
#ifdef GPU_part
	cuda_redistribute();
#endif
}

void LBM::abstract_streaming(){
	//streaming
#ifdef CPU_part
	streaming();
#endif
#ifdef GPU_part
	cuda_streaming();
#endif
}

void LBM::abstract_bounce_back(){
	//bounce back
#ifdef CPU_part
	bounceback();
#endif
#ifdef GPU_part
	cuda_bounceback();
#endif
}

void LBM::abstract_relaxation(){
	//relaxation
#ifdef CPU_part
	relaxation();
#endif
#ifdef GPU_part
	cuda_relaxation();
#endif
}

void LBM::abstract_convective_boundary_conditions(){
	//convective BC
#ifdef CPU_part
	convective_BC();
#endif
#ifdef GPU_part
	cuda_convective_BC();
#endif
}

void LBM::core_computations(){
	//LBM CORE
	abstract_redistribute();

	abstract_streaming();

	abstract_bounce_back();

	abstract_relaxation();

	abstract_convective_boundary_conditions();
}

void LBM::compute_domain(){
	//starting from second loop!
	for (time_unit = 1; time_unit<max_iterations ; ++time_unit){
		//		cout <<"# iteration " << time_unit << endl;
		cout <<time_unit <<". ";

		abstract_check_density();

		abstract_debug_computations();

		core_computations();

	}
	cout << endl;
}

void LBM::export_solution(){


	calculate_macroscopic_quantities(time_unit);
#ifdef PRODUCE_OUTPUT_FILES
	write_VTK_SI(time_unit);
#endif //PRODUCE_OUTPUT_FILES
}

template void
LBM::initialise_array<double>(double *array, const int length,const double init_value);

template void
LBM::initialise_array<float>(float *array, const int length,const float init_value);


template <typename T>
void LBM::allocate_and_initialise(T *array, const int length){

	hipMalloc((void **)&array, length*sizeof(T));

	int n_of_threads=threads_per_kernel;
	int n_of_blocks=ceil((length*1.0)/n_of_threads);

	dim3 threads_type2(n_of_threads,1,1);
	dim3 grid_type2(n_of_blocks,1,1);
	//kane to template!
	//cuda_initialise_array<<<grid_type2,threads_type2>>>(array, length, 0.0);
}

void LBM::calculate_CUDA_quantities() {
	threads_for_streaming_collision_and_relaxation=threads_per_kernel;
	blocks_for_streaming_collision_and_relaxation= (three_dimensional_length)/threads_per_kernel;
	if ((three_dimensional_length%threads_per_kernel)!=0)
		++blocks_for_streaming_collision_and_relaxation;
	size_of_allocated_shared_memory_for_streaming_collision_and_relaxation=threads_per_kernel*sizeof(FLOATING);

	convective_boundary_conditions_blocks=two_dimensional_length/threads_per_kernel;
	if ( (two_dimensional_length%threads_per_kernel)!=0 )
		++convective_boundary_conditions_blocks;
}

void LBM::reset_convergence_file(){
	//delete previous convergence.txt and create a new one
	if( remove("LBM2_convergence.txt")!=0 )
		cout <<"couldn't delete LBM2_convergence.txt" << endl;
	else
		cout<< "creating LBM2_convergence.txt" << endl;

	ofstream convergence_file("LBM2_convergence.txt");
	convergence_file<<"#iteration ; converegence_value" << endl;
	convergence_file.close();
}

void LBM::display_CUDA_specifications(){
	cout <<"CUDA specifications:" <<endl;
	cout <<"\tstreaming/collision/relaxation:" << endl;
	cout <<"\t\tthreads: "<<threads_for_streaming_collision_and_relaxation<<endl;
	cout <<"\t\tblocks: "<<blocks_for_streaming_collision_and_relaxation<<endl;
	cout <<"\t\tshare memory size: "<< size_of_allocated_shared_memory_for_streaming_collision_and_relaxation <<endl;

	cout <<"\tconvective boundary conditions:" << endl;
	cout <<"\t\tthreads:" << threads_per_kernel << endl;
	cout <<"\t\tblocks: "<< convective_boundary_conditions_blocks << endl;
}

void LBM::initialise_host_data(){
	initialise_array(obstacles,   three_dimensional_length,0 );

	initialise_array<FLOATING>(u_current,   two_dimensional_length,0.0);
	initialise_array<FLOATING>(v_current,   two_dimensional_length,0.0);
	initialise_array<FLOATING>(w_current,   two_dimensional_length,0.0);
	// u_previous_spatial_boundary: at boundary - 1 (in x)
	initialise_array<FLOATING>(u_previous_spatial_boundary,   two_dimensional_length,0.0);
	initialise_array<FLOATING>(v_previous_spatial_boundary,   two_dimensional_length,0.0);
	initialise_array<FLOATING>(w_previous_spatial_boundary,   two_dimensional_length,0.0);
	// u_prev: at boundary - 1 (in time)
	initialise_array<FLOATING>(u_previous_temporal_boundary,   two_dimensional_length,0.0);
	initialise_array<FLOATING>(v_previous_temporal_boundary,   two_dimensional_length,0.0);
	initialise_array<FLOATING>(w_previous_temporal_boundary,   two_dimensional_length,0.0);

}

void LBM::allocate_device_arrays(){
	//allocate additional cuda memories
	hipMalloc((void **)&u_current_d, FLOATING_slice_size);

	hipMalloc((void **)&u_current_temp_d, FLOATING_slice_size);
	hipMalloc((void **)&v_current_d, FLOATING_slice_size);
	hipMalloc((void **)&w_current_d, FLOATING_slice_size);

	hipMalloc((void **)&u_previous_spatial_boundary_d, FLOATING_slice_size);
	hipMalloc((void **)&v_previous_spatial_boundary_d, FLOATING_slice_size);
	hipMalloc((void **)&w_previous_spatial_boundary_d, FLOATING_slice_size);

	hipMalloc((void **)&u_previous_temporal_boundary_d, FLOATING_slice_size);
	hipMalloc((void **)&v_previous_temporal_boundary_d, FLOATING_slice_size);
	hipMalloc((void **)&w_previous_temporal_boundary_d, FLOATING_slice_size);

	hipMalloc((void **)&temp_check_density_d, lx*ly*lz/4 *sizeof(FLOATING));
	hipMalloc((void **)&temp_check_density_d_full, lx*ly*lz*sizeof(FLOATING));

	hipMalloc((void **)&obstacles_d, int_array_size);

	hipMalloc((void **)&temp_Uc_d, 2*sizeof(FLOATING));
}

void LBM::initialise_device_data(){
	dim3 threads_type2(threads_for_streaming_collision_and_relaxation,1,1);
	dim3 grid_type2(blocks_for_streaming_collision_and_relaxation,1,1);
	cuda_initialise_array<<<grid_type2,threads_type2>>>(temp_check_density_d_full, lx*ly*lz, 0.0);
}


void LBM::initialise_all_data_arrays(){
	initialise_host_data();
	allocate_device_arrays();
	initialise_device_data();
}

void LBM::display_LBM_specifications(){
	cout << "constructing LBM(built-in quantities)" << endl;

	cout << "\tdensity" << density << endl;
	cout << "\tt_0=" << t_0 << endl;
	cout << "\tt_1=" << t_1 << endl;
	cout << "\tt_2=" << t_2 << endl;
	cout << "\tc_squ=" << c_squ << endl;
	cout << "\ttau=" << tau << endl;
	cout << "\tomega=" << omega << endl;
}



LBM::LBM(const int &LX, const int &LY, const int &LZ, const FLOATING &DENSITY, const FLOATING &T_0,
		const FLOATING &T_1, const FLOATING &T_2, const FLOATING &C_SQU):
		time_elapsed(0),
		max_iterations(1000),
		check_step(100),
		lx(LX), ly(LY), lz(LZ),
		lattice_nodes(lx*ly*lz), no_obstacle_lattices_at_penultimate_x_slice(0),
		threads_for_streaming_collision_and_relaxation(512),
		blocks_for_streaming_collision_and_relaxation(32),
		size_of_allocated_shared_memory_for_streaming_collision_and_relaxation(48*1024),
		convective_boundary_conditions_blocks(32),
		nu(0.0175), r_small(6.67897), reynolds(195.732), s(23.7849), density(DENSITY),
		t_0(density*T_0), t_1(density*T_1), t_2(density*T_2), c_squ(C_SQU), reciprocal_c_squ(1.0/c_squ),
		baffle(XBAFFLE), threads_per_kernel(MANY_THREADS), time_unit(0),
		two_dimensional_length(ly*lz),
		three_dimensional_length(lx*ly*lz),
		FLOATING_slice_size((two_dimensional_length)*sizeof(FLOATING)),
		int_array_size((three_dimensional_length)*sizeof(int)),
		tau(3.0*nu + 0.5), omega(1.0 /tau), one_minus_omega (1.0-omega),
		pr_diff(0.0), pr_out(0.0), pr_in(0.0), vor(0.0),
		D3(lx, ly, lz), D3_hlp(lx, ly, lz), obstacles(new int[lz*ly*lx]),
		u_current(new FLOATING[ly*lz]),  v_current(new FLOATING[ly*lz]),  w_current(new FLOATING[ly*lz]),
		u_previous_spatial_boundary(new FLOATING[ly*lz]), v_previous_spatial_boundary(new FLOATING[ly*lz]), w_previous_spatial_boundary(new FLOATING[ly*lz]),
		u_previous_temporal_boundary(new FLOATING[ly*lz]), v_previous_temporal_boundary(new FLOATING[ly*lz]), w_previous_temporal_boundary(new FLOATING[ly*lz]),
		u_current_d(NULL), u_current_temp_d(NULL), v_current_d(NULL), w_current_d(NULL),
		// u_previous_spatial_boundary: at boundary - 1 (in x)
		u_previous_spatial_boundary_d(NULL), v_previous_spatial_boundary_d(NULL), w_previous_spatial_boundary_d(NULL),
		// u_prev: at boundary - 1 (in time)
		u_previous_temporal_boundary_d(NULL), v_previous_temporal_boundary_d(NULL), w_previous_temporal_boundary_d(NULL),
		temp_cpu_u_current_d(NULL), temp_cpu_v_current_d(NULL), temp_cpu_w_current_d(NULL),
		temp_cpu_u_previous_temporal_boundary_d(NULL), temp_cpu_v_previous_temporal_boundary_d(NULL), temp_cpu_w_previous_temporal_boundary_d(NULL),
		temp_cpu_u_previous_spatial_boundary_d(NULL), temp_cpu_v_previous_spatial_boundary_d(NULL), temp_cpu_w_previous_spatial_boundary_d(NULL),
		temp_check_density_d(NULL), temp_check_density_d_full(NULL),
		data_location(CPU),
		temp_Uc_d(NULL), obstacles_d(NULL),
		D3_d(lx, ly, lz, 0), D3_hlp_d(lx, ly, lz, 0),
		Ux(new FLOATING[lx*ly*lz]),
		Uy(new FLOATING[lx*ly*lz]),
		Uz(new FLOATING[lx*ly*lz]),
		Pressure(new FLOATING[lx*ly*lz]),
		Wx(new FLOATING[lx*ly*lz]),
		Wy(new FLOATING[lx*ly*lz]),
		Wz(new FLOATING[lx*ly*lz]){
	cout << "***LBM Starting***" << endl;

	time (&time_start);

	read_external_configuration_file_for_the_solver("LBM2_configuration.txt");

	reset_convergence_file();

	calculate_CUDA_quantities();

	display_CUDA_specifications();

	initialise_all_data_arrays();

	display_LBM_specifications();

	abstract_initialise();
}

void LBM::delete_host_memories(){
	delete [] obstacles;

	delete [] u_current;
	delete [] v_current;
	delete [] w_current;

	delete [] u_previous_spatial_boundary;
	delete [] v_previous_spatial_boundary;
	delete [] w_previous_spatial_boundary;

	delete [] u_previous_temporal_boundary;
	delete [] v_previous_temporal_boundary;
	delete [] w_previous_temporal_boundary;

	delete [] Ux;
	delete [] Uy;
	delete [] Uz;
	delete [] Pressure;
	delete [] Wx;
	delete [] Wy;
	delete [] Wz;
}

LBM::~LBM(){

	delete_host_memories();
	delete_device_data();


	cout <<"all memories were deallocated!" <<endl;


	cout << endl << "LBM2 ended in "<< time_elapsed<< "secs !" << endl; // prints
}

void LBM::count_no_obstacles_at_penultimate_x_slice(){

	no_obstacle_lattices_at_penultimate_x_slice = 0;
#pragma unroll
	for (int z = 0 ; z< lz ; ++z){
#pragma unroll
		for (int y = 0 ; y< ly ; ++y){
			if (obstacles[index(z,y,(lx-1))]==0) {
				++no_obstacle_lattices_at_penultimate_x_slice ;
			}
		}
	}
	cout << "number of free lattices at U direction at the penultimate slice:" << no_obstacle_lattices_at_penultimate_x_slice <<endl;

}

void LBM::copy_data_from_host_to_device(){//copy data to CUDA variables
	int const array_length=(lx*ly*lz);
	int const slice_length=(ly*lz);
	int const FLOATING_array_size=array_length*sizeof(FLOATING);
	int const FLOATING_slice_size=slice_length*sizeof(FLOATING);
	int const int_array_size=array_length*sizeof(int);




	hipMemcpy(D3_d.Q0 ,D3.Q0,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q1 ,D3.Q1,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q2 ,D3.Q2,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q3 ,D3.Q3,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q4 ,D3.Q4,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q5 ,D3.Q5,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q6 ,D3.Q6,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q7 ,D3.Q7,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q8 ,D3.Q8,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q9 ,D3.Q9,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q10 ,D3.Q10,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q11 ,D3.Q11,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q12 ,D3.Q12,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q13 ,D3.Q13,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q14 ,D3.Q14,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q15 ,D3.Q15,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q16 ,D3.Q16,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q17 ,D3.Q17,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q18 ,D3.Q18,FLOATING_array_size,hipMemcpyHostToDevice);

	hipMemcpy(D3_hlp_d.Q0 ,D3_hlp.Q0,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q1 ,D3_hlp.Q1,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q2 ,D3_hlp.Q2,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q3 ,D3_hlp.Q3,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q4 ,D3_hlp.Q4,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q5 ,D3_hlp.Q5,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q6 ,D3_hlp.Q6,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q7 ,D3_hlp.Q7,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q8 ,D3_hlp.Q8,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q9 ,D3_hlp.Q9,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q10 ,D3_hlp.Q10,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q11 ,D3_hlp.Q11,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q12 ,D3_hlp.Q12,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q13 ,D3_hlp.Q13,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q14 ,D3_hlp.Q14,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q15 ,D3_hlp.Q15,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q16 ,D3_hlp.Q16,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q17 ,D3_hlp.Q17,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q18 ,D3_hlp.Q18,FLOATING_array_size,hipMemcpyHostToDevice);


	hipMemcpy(obstacles_d ,obstacles,int_array_size,hipMemcpyHostToDevice);

	hipMemcpy(u_current_d ,u_current,FLOATING_slice_size,hipMemcpyHostToDevice);
	hipMemcpy(u_current_temp_d ,u_current,FLOATING_slice_size,hipMemcpyHostToDevice);
	hipMemcpy(v_current_d ,v_current,FLOATING_slice_size,hipMemcpyHostToDevice);
	hipMemcpy(w_current_d ,w_current,FLOATING_slice_size,hipMemcpyHostToDevice);

	hipMemcpy(u_previous_spatial_boundary_d ,u_previous_spatial_boundary,FLOATING_slice_size,hipMemcpyHostToDevice);
	hipMemcpy(v_previous_spatial_boundary_d ,v_previous_spatial_boundary,FLOATING_slice_size,hipMemcpyHostToDevice);
	hipMemcpy(w_previous_spatial_boundary_d ,w_previous_spatial_boundary,FLOATING_slice_size,hipMemcpyHostToDevice);

	hipMemcpy(u_previous_temporal_boundary_d ,u_previous_temporal_boundary,FLOATING_slice_size,hipMemcpyHostToDevice);
	hipMemcpy(v_previous_temporal_boundary_d ,v_previous_temporal_boundary,FLOATING_slice_size,hipMemcpyHostToDevice);
	hipMemcpy(w_previous_temporal_boundary_d ,w_previous_temporal_boundary,FLOATING_slice_size,hipMemcpyHostToDevice);

	data_location=GPU;
	printf("all data were copied to device\n");
}

void LBM::small_copy_data_from_host_to_device(){//copy data to CUDA variables
	int const array_length=(lx*ly*lz);

	int const FLOATING_array_size=array_length*sizeof(FLOATING);



	hipMemcpy(D3_d.Q0 ,D3.Q0,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q1 ,D3.Q1,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q2 ,D3.Q2,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q3 ,D3.Q3,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q4 ,D3.Q4,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q5 ,D3.Q5,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q6 ,D3.Q6,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q7 ,D3.Q7,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q8 ,D3.Q8,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q9 ,D3.Q9,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q10 ,D3.Q10,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q11 ,D3.Q11,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q12 ,D3.Q12,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q13 ,D3.Q13,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q14 ,D3.Q14,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q15 ,D3.Q15,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q16 ,D3.Q16,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q17 ,D3.Q17,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_d.Q18 ,D3.Q18,FLOATING_array_size,hipMemcpyHostToDevice);

	hipMemcpy(D3_hlp_d.Q0 ,D3_hlp.Q0,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q1 ,D3_hlp.Q1,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q2 ,D3_hlp.Q2,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q3 ,D3_hlp.Q3,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q4 ,D3_hlp.Q4,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q5 ,D3_hlp.Q5,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q6 ,D3_hlp.Q6,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q7 ,D3_hlp.Q7,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q8 ,D3_hlp.Q8,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q9 ,D3_hlp.Q9,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q10 ,D3_hlp.Q10,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q11 ,D3_hlp.Q11,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q12 ,D3_hlp.Q12,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q13 ,D3_hlp.Q13,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q14 ,D3_hlp.Q14,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q15 ,D3_hlp.Q15,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q16 ,D3_hlp.Q16,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q17 ,D3_hlp.Q17,FLOATING_array_size,hipMemcpyHostToDevice);
	hipMemcpy(D3_hlp_d.Q18 ,D3_hlp.Q18,FLOATING_array_size,hipMemcpyHostToDevice);

	data_location=GPU;
	printf("all data were copied to device\n");
}

void LBM::copy_data_from_device_to_host(){
	int const array_length=(lx*ly*lz);
	int const slice_length=(ly*lz);
	int const FLOATING_array_size=array_length*sizeof(FLOATING);
	int const FLOATING_slice_size=slice_length*sizeof(FLOATING);
	int const int_array_size=array_length*sizeof(int);



	hipMemcpy(D3.Q0 ,D3_d.Q0,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q1 ,D3_d.Q1,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q2 ,D3_d.Q2,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q3 ,D3_d.Q3,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q4 ,D3_d.Q4,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q5 ,D3_d.Q5,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q6 ,D3_d.Q6,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q7 ,D3_d.Q7,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q8 ,D3_d.Q8,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q9 ,D3_d.Q9,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q10 ,D3_d.Q10,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q11 ,D3_d.Q11,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q12 ,D3_d.Q12,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q13 ,D3_d.Q13,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q14 ,D3_d.Q14,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q15 ,D3_d.Q15,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q16 ,D3_d.Q16,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q17 ,D3_d.Q17,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q18 ,D3_d.Q18,FLOATING_array_size,hipMemcpyDeviceToHost);

	hipMemcpy(D3_hlp.Q0 ,D3_hlp_d.Q0,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q1 ,D3_hlp_d.Q1,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q2 ,D3_hlp_d.Q2,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q3 ,D3_hlp_d.Q3,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q4 ,D3_hlp_d.Q4,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q5 ,D3_hlp_d.Q5,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q6 ,D3_hlp_d.Q6,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q7 ,D3_hlp_d.Q7,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q8 ,D3_hlp_d.Q8,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q9 ,D3_hlp_d.Q9,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q10 ,D3_hlp_d.Q10,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q11 ,D3_hlp_d.Q11,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q12 ,D3_hlp_d.Q12,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q13 ,D3_hlp_d.Q13,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q14 ,D3_hlp_d.Q14,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q15 ,D3_hlp_d.Q15,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q16 ,D3_hlp_d.Q16,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q17 ,D3_hlp_d.Q17,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q18 ,D3_hlp_d.Q18,FLOATING_array_size,hipMemcpyDeviceToHost);

	hipMemcpy(obstacles ,obstacles_d,int_array_size,hipMemcpyDeviceToHost);

	hipMemcpy(u_current ,u_current_d,FLOATING_slice_size,hipMemcpyDeviceToHost);
	hipMemcpy(v_current ,v_current_d,FLOATING_slice_size,hipMemcpyDeviceToHost);
	hipMemcpy(w_current ,w_current_d,FLOATING_slice_size,hipMemcpyDeviceToHost);

	hipMemcpy(u_previous_spatial_boundary ,u_previous_spatial_boundary_d,FLOATING_slice_size,hipMemcpyDeviceToHost);
	hipMemcpy(v_previous_spatial_boundary ,v_previous_spatial_boundary_d,FLOATING_slice_size,hipMemcpyDeviceToHost);
	hipMemcpy(w_previous_spatial_boundary ,w_previous_spatial_boundary_d,FLOATING_slice_size,hipMemcpyDeviceToHost);

	hipMemcpy(u_previous_temporal_boundary ,u_previous_temporal_boundary_d,FLOATING_slice_size,hipMemcpyDeviceToHost);
	hipMemcpy(v_previous_temporal_boundary ,v_previous_temporal_boundary_d,FLOATING_slice_size,hipMemcpyDeviceToHost);
	hipMemcpy(w_previous_temporal_boundary ,w_previous_temporal_boundary_d,FLOATING_slice_size,hipMemcpyDeviceToHost);

	data_location=CPU;
	printf("all data were copied to host\n");
}

void LBM::small_copy_data_from_device_to_host(){
	int const array_length=(lx*ly*lz);

	int const FLOATING_array_size=array_length*sizeof(FLOATING);


	hipMemcpy(D3.Q0 ,D3_d.Q0,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q1 ,D3_d.Q1,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q2 ,D3_d.Q2,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q3 ,D3_d.Q3,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q4 ,D3_d.Q4,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q5 ,D3_d.Q5,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q6 ,D3_d.Q6,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q7 ,D3_d.Q7,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q8 ,D3_d.Q8,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q9 ,D3_d.Q9,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q10 ,D3_d.Q10,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q11 ,D3_d.Q11,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q12 ,D3_d.Q12,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q13 ,D3_d.Q13,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q14 ,D3_d.Q14,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q15 ,D3_d.Q15,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q16 ,D3_d.Q16,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q17 ,D3_d.Q17,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3.Q18 ,D3_d.Q18,FLOATING_array_size,hipMemcpyDeviceToHost);

	hipMemcpy(D3_hlp.Q0 ,D3_hlp_d.Q0,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q1 ,D3_hlp_d.Q1,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q2 ,D3_hlp_d.Q2,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q3 ,D3_hlp_d.Q3,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q4 ,D3_hlp_d.Q4,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q5 ,D3_hlp_d.Q5,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q6 ,D3_hlp_d.Q6,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q7 ,D3_hlp_d.Q7,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q8 ,D3_hlp_d.Q8,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q9 ,D3_hlp_d.Q9,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q10 ,D3_hlp_d.Q10,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q11 ,D3_hlp_d.Q11,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q12 ,D3_hlp_d.Q12,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q13 ,D3_hlp_d.Q13,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q14 ,D3_hlp_d.Q14,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q15 ,D3_hlp_d.Q15,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q16 ,D3_hlp_d.Q16,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q17 ,D3_hlp_d.Q17,FLOATING_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(D3_hlp.Q18 ,D3_hlp_d.Q18,FLOATING_array_size,hipMemcpyDeviceToHost);


	data_location=CPU;
	printf("all data were copied to host\n");
}

void LBM::compare_obstacles(int *outter_obst){

	int x,y,z;

	for ( z =0 ; z < lz ; ++z){
		for (y = 0; y < ly ; ++y){
			for ( x = 0 ; x< lx ; ++x){
				if( obstacles[index(z,y,x)]!=outter_obst[index(z,y,x)]){
					cout << "obstacle miss-match @" << x << " " << y << " " << z <<endl;
					exit(-2);
				}
			}
		}
	}

	cout << "obstacles ok" << endl;

}

void LBM::compare_nodes(FLOATING *outter_node){
	int x,y,z,i;

	for ( z =0 ; z < lz ; ++z){
		for (y = 0; y < ly ; ++y){
			for ( x = 0 ; x< lx ; ++x){

				i=0;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q0[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" << i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q0[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=1;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q1[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q1[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=2;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q2[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q2[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=3;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q3[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q3[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=4;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q4[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q4[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=5;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q5[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q5[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=6;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q6[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q6[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=7;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q7[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q7[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=8;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q8[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q8[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=9;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q9[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q9[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=10;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q10[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q10[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=11;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q11[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q11[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=12;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q12[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q12[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=13;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q13[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q13[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=14;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q14[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q14[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=15;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q15[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q15[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=16;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q16[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q16[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=17;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q17[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q17[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=18;
				if( abs(outter_node[index4D(z,y,x,i)]-D3.Q18[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node[index4D(z,y,x,i)] << " vs "<< D3.Q18[index(z,y,x)] <<endl;
					exit(-1000);
				}

			}
		}
	}
	cout << "nodes ok" << endl;
}

void LBM::compare_nodes_hlp(FLOATING *outter_node_hlp){
	int x,y,z,i;

	for ( z =0 ; z < lz ; ++z){
		for (y = 0; y < ly ; ++y){
			for ( x = 0 ; x< lx ; ++x){

				i=0;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q0[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q0[index(z,y,x)] <<endl;

					exit(-1000);
				}

				i=1;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q1[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q1[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=2;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q2[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q2[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=3;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q3[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q3[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=4;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q4[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q4[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=5;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q5[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q5[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=6;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q6[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q6[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=7;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q7[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q7[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=8;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q8[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q8[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=9;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q9[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q9[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=10;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q10[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q10[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=11;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q11[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q11[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=12;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q12[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q12[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=13;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q13[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q13[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=14;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q14[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q14[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=15;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q15[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q15[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=16;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q16[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q16[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=17;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q17[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3_hlp.Q17[index(z,y,x)] <<endl;
					exit(-1000);
				}

				i=18;
				if( abs(outter_node_hlp[index4D(z,y,x,i)]-D3_hlp.Q18[index(z,y,x)])>0.00001){
					cout << "node miss-match @ x:" << x << " y:" << y << " z:" << z << " i:" <<i <<  " :" << outter_node_hlp[index4D(z,y,x,i)] << " vs "<< D3.Q18[index(z,y,x)] <<endl;
					exit(-1000);
				}

			}
		}
	}
	cout << "n_hlp ok" << endl;
}
