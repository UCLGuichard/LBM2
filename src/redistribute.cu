#include "hip/hip_runtime.h"
#include "global_defines.cuh"

__global__
void redistribute_kernel(const int lx, const int ly, const int lz,
		FLOATING reynolds, FLOATING nu, FLOATING r_small,
		FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ,

		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18){

	/************************************************************************
	 *                                                                      *
	 *     density redistribution in first lattice column                   *
	 *                                                                      *
	 *                                                                      *
	 *     Last change: 04/05/2003                                          *
	 *                                                                      *
	 ************************************************************************/
	/*
				c
				c.......directed flow is induced by density redistribution in the first
				c       lattice column. This is not too clever, since the resulting
				c       reynolds number can not be controlled and reaching steady state
				c       takes quite some time, but it is simple and it works ...
				c       use this to start with no initial field
	 */

	/*
	creates u_n, u_squ and assigns the values in node[]

	 */

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	//int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int R_big;
	int baffle_position=59;

	FLOATING    mass_flow;

	//.....local variables

	int   yc, zc, yr, zr;


	FLOATING   rho/*local density*/, u_avg,A_out,A_inn,A_anu,pi,u_xa,u_xs;
	FLOATING   u_x,u_y, u_z, u_n[19] , u_squ;


	yc= (ly+1)/2 -1;//CHANGED! ORIGINALLY IT WAS yc= (ly +1)/2; AND zc= (ly +1)/2;
	zc= (ly+1)/2 -1;
	R_big=35;
	mass_flow=0.05;
	pi = acos(0.0);
	u_avg =reynolds*nu/(2*R_big);
	u_xa= (R_big*R_big) / (R_big*R_big-r_small*r_small)*u_avg/(1+mass_flow);
	A_out=pi*R_big*R_big;
	A_inn=pi*r_small*r_small;
	A_anu=A_out-A_inn;
	u_xs=A_anu*u_xa*mass_flow/A_inn;



	//.....compute weighting factors (depending on lattice geometry) for
	//     increasing/decreasing inlet densities


	//8etei se olo to domain thn idia taxuthta, thn opoia 8a allaksei meta
	//gia ton eswteriko swlhna

	//todo: vale sto katw for, to x na paizei metaksu timwn pou prosdiorizontai apo to rank!
	//px. gia x=0...1/rank... 1/rank+margin... 2/rank.... etc!

	//	int end_of_memory=lz*ly*(baffle_position+1);





	zr=z-zc;
	yr=y-yc;

	if(x< (baffle_position+1) and yr*yr+zr*zr < r_small*r_small and tid<lx*ly*lz){
		// id = z*+y+x


		//		for( z = 0; z< lz ; ++z){
		//			for( y = 0; y< ly ; ++y){
		//				for( x = 0; x< baffle_position+1 ; ++x){



		//		rho=0.0;
		rho=Q0[index(z,y,x)]+Q1[index(z,y,x)]+Q2[index(z,y,x)]+Q3[index(z,y,x)]+
				Q4[index(z,y,x)]+Q5[index(z,y,x)]+Q6[index(z,y,x)]+Q7[index(z,y,x)]+
				Q8[index(z,y,x)]+Q9[index(z,y,x)]+Q10[index(z,y,x)]+Q11[index(z,y,x)]+
				Q12[index(z,y,x)]+Q13[index(z,y,x)]+Q14[index(z,y,x)]+Q15[index(z,y,x)]+
				Q16[index(z,y,x)]+Q17[index(z,y,x)]+Q18[index(z,y,x)];


		u_x = u_xs;
		u_y =  0.0;
		u_z =  0.0;

		u_n[0]= 0.0; //SHOULD NEVER USED!
		u_n[1] =   u_x; //u_xa
		u_n[2] =         u_y;
		u_n[3] = - u_x;
		u_n[4] =       - u_y;
		u_n[5] =   u_z;
		u_n[6] =       - u_z;
		u_n[7] =   u_x + u_y;
		u_n[8] = - u_x + u_y;
		u_n[9] = - u_x - u_y;
		u_n[10] =   u_x - u_y;
		u_n[11] =   u_x - u_z;
		u_n[12] = - u_x - u_z;
		u_n[13] = - u_x + u_z;
		u_n[14] =   u_x + u_z;
		u_n[15] =   u_z + u_y;
		u_n[16] = - u_z + u_y;
		u_n[17] = - u_z - u_y;
		u_n[18] =   u_z - u_y;

		u_squ = u_x*u_x + u_y*u_y + u_z*u_z;


		Q0[index(z,y,x)]=(FLOATING) (t_0  * rho *(1.0  - u_squ / (2.0  * c_squ)));

		//...........axis speeds (factor: t_1)

		Q1[index(z,y,x)]=(FLOATING) (t_1 * rho *	 (1.0+ ( u_n[1]/c_squ ) +  0.5* ( (u_n[1]*u_n[1])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));;
		Q2[index(z,y,x)]=(FLOATING) (t_1 * rho *	 (1.0+ ( u_n[2]/c_squ ) +  0.5* ( (u_n[2]*u_n[2])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));;
		Q3[index(z,y,x)]=(FLOATING) (t_1 * rho *	 (1.0+ ( u_n[3]/c_squ ) +  0.5* ( (u_n[3]*u_n[3])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));;
		Q4[index(z,y,x)]=(FLOATING) (t_1 * rho *	 (1.0+ ( u_n[4]/c_squ ) +  0.5* ( (u_n[4]*u_n[4])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));;
		Q5[index(z,y,x)]=(FLOATING) (t_1 * rho *	 (1.0+ ( u_n[5]/c_squ ) +  0.5* ( (u_n[5]*u_n[5])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));;
		Q6[index(z,y,x)]=(FLOATING) (t_1 * rho *	 (1.0+ ( u_n[6]/c_squ ) +  0.5* ( (u_n[6]*u_n[6])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));;

		//...........diagonal speeds (factor: t_2)
		Q7[index(z,y,x)]=(FLOATING) (t_2 * rho *	 (1.0+ ( u_n[7]/c_squ ) +  0.5* ( (u_n[7]*u_n[7])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));;
		Q8[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[8]/c_squ ) +  0.5* ( (u_n[8]*u_n[8])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q9[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[9]/c_squ ) +  0.5* ( (u_n[9]*u_n[9])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q10[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[10]/c_squ ) +  0.5* ( (u_n[10]*u_n[10])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q11[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[11]/c_squ ) +  0.5* ( (u_n[11]*u_n[11])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q12[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[12]/c_squ ) +  0.5* ( (u_n[12]*u_n[12])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q13[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[13]/c_squ ) +  0.5* ( (u_n[13]*u_n[13])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q14[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[14]/c_squ ) +  0.5* ( (u_n[14]*u_n[14])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q15[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[15]/c_squ ) +  0.5* ( (u_n[15]*u_n[15])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q16[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[16]/c_squ ) +  0.5* ( (u_n[16]*u_n[16])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q17[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[17]/c_squ ) +  0.5* ( (u_n[17]*u_n[17])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));
		Q18[index(z,y,x)]=(FLOATING) (t_2  * rho * (1.0+ ( u_n[18]/c_squ ) +  0.5* ( (u_n[18]*u_n[18])/(c_squ*c_squ) ) - 0.5 * ( u_squ/c_squ ) ));


		//				}
		//			}
		//		}
	}


}


void LBM::cuda_redistribute(){

	if(data_location==CPU)
		copy_data_from_host_to_device();

//	int lattice_nodes=lz*ly*lx;
//
//	int n_of_threads=128;
//	int n_of_blocks=ceil((lattice_nodes*1.0)/n_of_threads);
//	dim3 threads_type2(n_of_threads,1,1);
//	dim3 grid_type2(n_of_blocks,1,1);
//#ifdef REPORT
//	cout << "redistribute kernel with:" << lattice_nodes << " lattice nodes" << endl;
//	cout << "\tthreads:" << n_of_threads << endl;
//	cout << "\tblocks:" << n_of_blocks << endl;
//#endif

	dim3 threads_type2(threads_for_streaming_collision_and_relaxation,1,1);
		dim3 grid_type2(blocks_for_streaming_collision_and_relaxation,1,1);


	redistribute_kernel<<<grid_type2, threads_type2>>>(lx, ly, lz, reynolds, nu, r_small,
			t_0, t_1, t_2, c_squ,
			D3_d.Q0, D3_d.Q1, D3_d.Q2, D3_d.Q3,
			D3_d.Q4, D3_d.Q5, D3_d.Q6, D3_d.Q7,
			D3_d.Q8, D3_d.Q9, D3_d.Q10, D3_d.Q11,
			D3_d.Q12, D3_d.Q13, D3_d.Q14, D3_d.Q15,
			D3_d.Q16, D3_d.Q17, D3_d.Q18);

	hipDeviceSynchronize();

}
