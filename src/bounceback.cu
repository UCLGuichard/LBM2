#include "hip/hip_runtime.h"
#include "global_defines.cuh"


void LBM::bounceback(){
	/*Fluid densities are rotated. By the next propagation step, this  *
	 *     results in a bounce back from obstacle nodes.*/

	/*
			.......bounce back from obstacles: this is the no-slip boundary-
			condition.
			The velocity vector of all fluid densities is inverted, so all
			the fluid densities will be sent back to the node  where they
			were located before the last propagation step, but with opposite
			velocity vector
			... there exist lots of other possibilities.
		 */

	if(data_location==GPU)
		copy_data_from_device_to_host();

	int  x,y,z;

	//.....loop over all nodes
	for( z = 0; z< lz ; ++z){
		for( y = 0; y< ly ; ++y){
			for( x = 0; x< lx ; ++x){

				//.........consider only obstacle nodes
				if (obstacles[index(z,y,x)]==1){

					//...........rotate all ensities and write back to node
					D3.Q1[index(z,y,x)] = D3_hlp.Q3[index(z,y,x)];
					D3.Q2[index(z,y,x)] = D3_hlp.Q4[index(z,y,x)];
					D3.Q3[index(z,y,x)] = D3_hlp.Q1[index(z,y,x)];
					D3.Q4[index(z,y,x)] = D3_hlp.Q2[index(z,y,x)];
					D3.Q5[index(z,y,x)] = D3_hlp.Q6[index(z,y,x)];
					D3.Q6[index(z,y,x)] = D3_hlp.Q5[index(z,y,x)];
					D3.Q7[index(z,y,x)] = D3_hlp.Q9[index(z,y,x)];
					D3.Q8[index(z,y,x)] = D3_hlp.Q10[index(z,y,x)];
					D3.Q9[index(z,y,x)] = D3_hlp.Q7[index(z,y,x)];
					D3.Q10[index(z,y,x)] = D3_hlp.Q8[index(z,y,x)];
					D3.Q11[index(z,y,x)] = D3_hlp.Q13[index(z,y,x)];
					D3.Q12[index(z,y,x)] = D3_hlp.Q14[index(z,y,x)];
					D3.Q13[index(z,y,x)] = D3_hlp.Q11[index(z,y,x)];
					D3.Q14[index(z,y,x)] = D3_hlp.Q12[index(z,y,x)];
					D3.Q15[index(z,y,x)] = D3_hlp.Q17[index(z,y,x)];
					D3.Q16[index(z,y,x)] = D3_hlp.Q18[index(z,y,x)];
					D3.Q17[index(z,y,x)] = D3_hlp.Q15[index(z,y,x)];
					D3.Q18[index(z,y,x)] = D3_hlp.Q16[index(z,y,x)];
				}
			}
		}
	}
#ifdef DEBUG
	cout << " #LBM bounceback OK!" << endl;
#endif
}









__global__
void bounceback_kernel_v4_shared(const int end_of_memory, const CUDA_FLOATING *source_data, CUDA_FLOATING *destination_data,
		const int *obstacles){
	/*Fluid densities are rotated. By the next propagation step, this  *
	 *     results in a bounce back from obstacle nodes.*/

	/*
		.......bounce back from obstacles: this is the no-slip boundary-
		condition.
		The velocity vector of all fluid densities is inverted, so all
		the fluid densities will be sent back to the node  where they
		were located before the last propagation step, but with opposite
		velocity vector
		... there exist lots of other possibilities.
	 */

	const int tid=blockIdx.x*blockDim.x+threadIdx.x;
	extern __shared__ CUDA_FLOATING shared_buffer[];
	shared_buffer[threadIdx.x]=source_data[tid];

	__syncthreads();
	if (tid<end_of_memory and obstacles[tid]){
		destination_data[tid]=shared_buffer[threadIdx.x];
	}
}



void LBM::cuda_bounceback(){

	if(data_location==CPU)
		copy_data_from_host_to_device();

	dim3 threads_type2(threads_for_streaming_collision_and_relaxation,1,1);
	dim3 grid_type2(blocks_for_streaming_collision_and_relaxation,1,1);



	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q3, D3_d.Q1,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q4, D3_d.Q2,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q1, D3_d.Q3,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q2, D3_d.Q4,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q6, D3_d.Q5,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q5, D3_d.Q6,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q9, D3_d.Q7,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q10, D3_d.Q8,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q7, D3_d.Q9,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q8, D3_d.Q10,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q13, D3_d.Q11,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q14, D3_d.Q12,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q11, D3_d.Q13,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q12, D3_d.Q14,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q17, D3_d.Q15,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q18, D3_d.Q16,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q15, D3_d.Q17,   obstacles_d);
	bounceback_kernel_v4_shared<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>(lattice_nodes,
			D3_hlp_d.Q16, D3_d.Q18,   obstacles_d);

	hipDeviceSynchronize();
#ifdef DEBUG
cout << " #LBM bounceback OK!" << endl;
#endif
}

