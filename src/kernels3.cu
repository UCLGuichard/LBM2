#include "hip/hip_runtime.h"
/*
 * Copyright 2008, Karen Hains, UWA . All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws. Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * WE MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE. IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.
 */

#ifndef _REDUCE_KERNELS_
#define _REDUCE_KERNELS_

#include "global_defines.cuh"
#include <cstdio>
#include <cmath>

///////////////////////////////////////////////////////////
// Simple Hello World kernel
// @param gpu_odata output data in global memory
///////////////////////////////////////////////////////////

#define SUM
//#define MINIMUM
//#define MAXIMUM


#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

#ifndef MAX
#define MAX(x,y) ((x > y) ? x : y)
#endif

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
	__device__ inline operator       T*()
	{
		extern __shared__ int __smem[];
		return (T*)__smem;
	}

	__device__ inline operator const T*() const
    																		{
		extern __shared__ int __smem[];
		return (T*)__smem;
    																		}
};

// specialize for FLOATING to avoid unaligned memory
// access compile errors

#ifdef USING_DOUBLE
template<>
struct SharedMemory<double>
{
	__device__ inline operator       double*()
    														{
		extern __shared__ double __smem_d[];
		return (double*)__smem_d;
    														}

	__device__ inline operator const double*() const
    														{
		extern __shared__ double __smem_d[];
		return (double*)__smem_d;
    														}
};
#endif //USING_DOUBLE

#ifndef USING_DOUBLE
template<>
struct SharedMemory<float>
{
	__device__ inline operator       float*()
    														{
		extern __shared__ float __smem_d[];
		return (float*)__smem_d;
    														}

	__device__ inline operator const float*() const
    														{
		extern __shared__ float __smem_d[];
		return (float*)__smem_d;
    														}
};
#endif //USING_FLOAT

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
 */

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved
   inactivity means that no whole warps are active, which is also very
   inefficient */

unsigned int nextPow2( unsigned int x ) {
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
	//get device capability, to avoid block/grid size excceed the upbound
	hipDeviceProp_t prop;
	int device;
	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);

	if (whichKernel < 3)
	{
		threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
		blocks = (n + threads - 1) / threads;
	}
	else
	{
		threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
		blocks = (n + (threads * 2 - 1)) / (threads * 2);
	}

	if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	{
		printf("n is too large, please choose a smaller number!\n");
	}

	if (blocks > prop.maxGridSize[0])
	{
		printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
				blocks, prop.maxGridSize[0], threads*2, threads);

		blocks /= 2;
		threads *= 2;
	}

	if (whichKernel == 6)
	{
		blocks = MIN(maxBlocks, blocks);
	}
#ifdef REPORT
	printf("CUDA Kernels will be launched with:\n");
	printf("\tnumber of blocks=%d\n", blocks);
	printf("\tnumber of threads=%d\n", threads);
#endif //REPORT

}

template <class T>
__global__ void
reduce0(T *g_idata, T *g_odata, unsigned int n)
{
	T *sdata = SharedMemory<T>();

	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

#ifdef SUM
	sdata[tid] = (i < n) ? g_idata[i] : 0;
#endif
#ifdef MINIMUM
	sdata[tid] = (i < n) ? g_idata[i] : 0;
#endif
#ifdef MAXIMUM
	sdata[tid] = (i < n) ? g_idata[i] : 0;
#endif

	__syncthreads();

	// do reduction in shared mem
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		// modulo arithmetic is slow!
		if ((tid % (2*s)) == 0) {
#ifdef SUM
			sdata[tid] += sdata[tid + s];
#endif
#ifdef MINIMUM
			sdata[tid] = min(sdata[tid] ,  sdata[tid + s]);
#endif
#ifdef MAXIMUM
			sdata[tid] = max(sdata[tid] ,  sdata[tid + s]);
#endif

		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce61(T *g_idata, T *g_odata, unsigned int n)
{
	//original
	T *sdata = SharedMemory<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	T mySum = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{
		mySum += g_idata[i];

		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n)
			mySum += g_idata[i+blockSize];

		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = mySum;
	__syncthreads();


	// do reduction in shared mem
	if (blockSize >= 512)
	{
		if (tid < 256)
		{
			sdata[tid] = mySum = mySum + sdata[tid + 256];
		}

		__syncthreads();
	}

	if (blockSize >= 256)
	{
		if (tid < 128)
		{
			sdata[tid] = mySum = mySum + sdata[tid + 128];
		}

		__syncthreads();
	}

	if (blockSize >= 128)
	{
		if (tid <  64)
		{
			sdata[tid] = mySum = mySum + sdata[tid +  64];
		}

		__syncthreads();
	}

	if (tid < 32)
	{
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		volatile T *smem = sdata;

		if (blockSize >=  64)
		{
			smem[tid] = mySum = mySum + smem[tid + 32];
		}

		if (blockSize >=  32)
		{
			smem[tid] = mySum = mySum + smem[tid + 16];
		}

		if (blockSize >=  16)
		{
			smem[tid] = mySum = mySum + smem[tid +  8];
		}

		if (blockSize >=   8)
		{
			smem[tid] = mySum = mySum + smem[tid +  4];
		}

		if (blockSize >=   4)
		{
			smem[tid] = mySum = mySum + smem[tid +  2];
		}

		if (blockSize >=   2)
		{
			smem[tid] = mySum = mySum + smem[tid +  1];
		}
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}



template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce6(T *g_idata, T *g_odata, unsigned int n)
{
	T *sdata = SharedMemory<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

#ifdef SUM
	T mySum = 0;
#endif
#ifdef MINIMUM
	T myMin = g_idata[i];//g_idata[0];
#endif
#ifdef MAXIMUM
	T myMax = g_idata[i];//g_idata[0];
#endif
	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{
#ifdef SUM
		mySum = mySum+ g_idata[i];
#endif
#ifdef MINIMUM
		myMin=min(myMin, g_idata[i]);
#endif
#ifdef MAXIMUM
		myMax=max(myMax, g_idata[i]);
#endif

		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n){
#ifdef SUM
			mySum = mySum + g_idata[i+blockSize];
#endif
#ifdef MINIMUM
			myMin=min(myMin, g_idata[i+blockSize]);
#endif
#ifdef MAXIMUM
			myMax=max(myMax, g_idata[i+blockSize]);
#endif
		}
		i += gridSize;
	}

	// each thread puts its local sum into shared memory
#ifdef SUM
	sdata[tid] = mySum;
#endif
#ifdef MINIMUM
	sdata[tid] = myMin;
#endif
#ifdef MAXIMUM
	sdata[tid] = myMax;
#endif
	__syncthreads();


	// do reduction in shared mem
#ifdef SUM
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
#endif
#ifdef MINIMUM
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = myMin = min(myMin , sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = myMin = min(myMin , sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = myMin = min(myMin , sdata[tid +  64]); } __syncthreads(); }
#endif
#ifdef MAXIMUM
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] = myMax = max(myMax , sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] = myMax = max(myMax , sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] = myMax = max(myMax , sdata[tid +  64]); } __syncthreads(); }
#endif
	if (tid < 32)
	{
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		volatile T* smem = sdata;
#ifdef SUM
		if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
		if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
		if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
		if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
		if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
		if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
#endif
#ifdef MINIMUM
		if (blockSize >=  64) { smem[tid] = myMin = min(myMin , smem[tid + 32]); }
		if (blockSize >=  32) { smem[tid] = myMin = min(myMin , smem[tid + 16]); }
		if (blockSize >=  16) { smem[tid] = myMin = min(myMin , smem[tid +  8]); }
		if (blockSize >=   8) { smem[tid] = myMin = min(myMin , smem[tid +  4]); }
		if (blockSize >=   4) { smem[tid] = myMin = min(myMin , smem[tid +  2]); }
		if (blockSize >=   2) { smem[tid] = myMin = min(myMin , smem[tid +  1]); }
#endif
#ifdef MAXIMUM
		if (blockSize >=  64) { smem[tid] = myMax = max(myMax , smem[tid + 32]); }
		if (blockSize >=  32) { smem[tid] = myMax = max(myMax , smem[tid + 16]); }
		if (blockSize >=  16) { smem[tid] = myMax = max(myMax , smem[tid +  8]); }
		if (blockSize >=   8) { smem[tid] = myMax = max(myMax , smem[tid +  4]); }
		if (blockSize >=   4) { smem[tid] = myMax = max(myMax , smem[tid +  2]); }
		if (blockSize >=   2) { smem[tid] = myMax = max(myMax , smem[tid +  1]); }
#endif
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}






bool isPow2(unsigned int x)
{
	return ((x&(x-1))==0);
}



////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void
reduce(int size, int threads, int blocks,
		int whichKernel, T *d_idata, T *d_odata)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

	// choose which of the optimized versions of reduction to launch
	switch (whichKernel)
	{
	case 0:
		reduce0<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
		break;
	case 6:
	default:
		if (isPow2(size))
		{
			switch (threads)
			{
			case 512:
				reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 256:
				reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 128:
				reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 64:
				reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 32:
				reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 16:
				reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  8:
				reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  4:
				reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  2:
				reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  1:
				reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			}
		}
		else
		{
			switch (threads)
			{
			case 512:
				reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 256:
				reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 128:
				reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 64:
				reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 32:
				reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case 16:
				reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  8:
				reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  4:
				reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  2:
				reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			case  1:
				reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
			}
		}
		break;
	}
}




// Instantiate the reduction function for 3 types
template void
reduce<int>(int size, int threads, int blocks,
		int whichKernel, int *d_idata, int *d_odata);

template void
reduce<float>(int size, int threads, int blocks,
		int whichKernel, float *d_idata, float *d_odata);

template void
reduce<double>(int size, int threads, int blocks,
		int whichKernel, double *d_idata, double *d_odata);


//void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads);



FLOATING reduce_sum(FLOATING *d_idata, const int big_array_length){
	int whichKernel = 6;
	int maxThreads = 256;  // number of threads per block
	int maxBlocks = min(33554432/maxThreads , 65535);
	int numBlocks, numThreads;
	int cpuFinalThreshold=1;


	getNumBlocksAndThreads(whichKernel, big_array_length, maxBlocks, maxThreads, numBlocks, numThreads);

	if (numBlocks == 1) cpuFinalThreshold = 1;
	FLOATING* d_odata = NULL;
	hipMalloc((void**) &d_odata, numBlocks*sizeof(FLOATING));
	FLOATING* h_odata = (FLOATING*) malloc(numBlocks*sizeof(FLOATING));

	reduce<FLOATING>(big_array_length, numThreads, numBlocks, whichKernel, d_idata, d_odata);
	hipDeviceSynchronize();

	FLOATING gpu_result_sum=0.0;
	bool needReadBack = true;

	int s=numBlocks;
	int kernel = whichKernel;

	while (s > cpuFinalThreshold)
	{
		int threads = 0, blocks = 0;
		getNumBlocksAndThreads(kernel, s, maxBlocks, maxThreads, blocks, threads);

		reduce<FLOATING>(s, threads, blocks, kernel, d_odata, d_odata);

		if (kernel < 3)
		{
			s = (s + threads - 1) / threads;
		}
		else
		{
			s = (s + (threads*2-1)) / (threads*2);
		}
	}

	if (s > 1)
	{
		// copy result from device to host
		hipMemcpy(h_odata, d_odata, s * sizeof(FLOATING), hipMemcpyDeviceToHost);

		for (int i=0; i < s; i++)
		{
			gpu_result_sum += h_odata[i];
		}

		needReadBack = false;
	}

	if (needReadBack)
	{
		// copy final sum from device to host
		hipMemcpy(&gpu_result_sum, d_odata, sizeof(FLOATING), hipMemcpyDeviceToHost);
	}

	hipDeviceSynchronize();


#ifdef REPORT
	printf ( " [universal]gpu_result_sum: %30.10f \n", gpu_result_sum);
#endif //REPORT
	hipFree(d_odata);
	return gpu_result_sum;
}


#endif // #ifndef _HELLOWORLD_KERNEL_H_
