#include "hip/hip_runtime.h"
#include "global_defines.cuh"
#include <numeric>




void LBM::relaxation(){





	/*One-step density relaxation process

				c.......density relaxation: a single time relaxation with relaxation
				c       parameter omega is applied here. This step is only "local",
				c       nothing is propagated through the lattice.
				c*/

	if(data_location==GPU)
		copy_data_from_device_to_host();

	int  x,y,z;
	FLOATING  u_x=0.0, u_y=0.0, u_z=0.0, u_squ=0.0, rho=0.0, reciprocal_rho=0.0;




	FLOATING u_n[19];
	//	FLOATING n_equ[19];
	FLOATING buff[19];

	//	FLOATING u_n_squared[19];
	//FLOATING two_x_c_squ_sqared;
	const FLOATING omega_x_t_0=omega*t_0, omega_x_t_1=omega*t_1, omega_x_t_2=omega*t_2;
	FLOATING omega_x_rho_x_t_0, omega_x_rho_x_t_1, omega_x_rho_x_t_2;
	FLOATING temp_factor;
	FLOATING u_n__over__c_squ[19];
	FLOATING u_n__over__c_squ__squared_and_halved[19];

	//....square speed of sound
	/*      compute the out let velocity with a convevtive boundary condition
					c.....loop over all nodes
					c.....attention: actual densities are stored after the propagation
					c                step in the help-array n_hlp !*/


#pragma unroll
	for (z = 0 ; z< lz ; ++z){
#pragma unroll
		for (y = 0 ; y< ly ; ++y){
#pragma unroll
			for (x = 0 ; x< lx; ++x){

				/*c.........only free nodes are considered here
					!if (.not. obstacles[z][y][x]) then
					c...........integral local density
					c...........initialize variable ro*/
				//memory optimised implementation
				buff[0]=D3_hlp.Q0[index(z,y,x)];
				buff[1]=D3_hlp.Q1[index(z,y,x)];
				buff[2]=D3_hlp.Q2[index(z,y,x)];
				buff[3]=D3_hlp.Q3[index(z,y,x)];
				buff[4]=D3_hlp.Q4[index(z,y,x)];
				buff[5]=D3_hlp.Q5[index(z,y,x)];
				buff[6]=D3_hlp.Q6[index(z,y,x)];
				buff[7]=D3_hlp.Q7[index(z,y,x)];
				buff[8]=D3_hlp.Q8[index(z,y,x)];
				buff[9]=D3_hlp.Q9[index(z,y,x)];
				buff[10]=D3_hlp.Q10[index(z,y,x)];
				buff[11]=D3_hlp.Q11[index(z,y,x)];
				buff[12]=D3_hlp.Q12[index(z,y,x)];
				buff[13]=D3_hlp.Q13[index(z,y,x)];
				buff[14]=D3_hlp.Q14[index(z,y,x)];
				buff[15]=D3_hlp.Q15[index(z,y,x)];
				buff[16]=D3_hlp.Q16[index(z,y,x)];
				buff[17]=D3_hlp.Q17[index(z,y,x)];
				buff[18]=D3_hlp.Q18[index(z,y,x)];

				rho=accumulate(buff, buff+DENSITIES, 0.0);

				reciprocal_rho=1.0/rho;




				switch(obstacles[index(z,y,x)]){
				case 1:
					u_x = 0.0;
					u_y = 0.0;
					u_z = 0.0;
					break;
				default:
					u_x = 0.0;
					u_x =  reciprocal_rho*(buff[1] + buff[7] + buff[10] +buff[11] + buff[14]-
							(buff[3] + buff[8] + buff[9] +buff[12] + buff[13]));

					u_y = 0.0;
					u_y =  reciprocal_rho*(buff[2]+buff[8]+buff[7]+buff[16] + buff[15] -
							(buff[4] + buff[9] + buff[10] +buff[17] + buff[18]));

					u_z = 0.0;
					u_z =  reciprocal_rho*(buff[5]+buff[13]+buff[14]+buff[15]+buff[18]-
							(buff[6]+buff[12]+buff[11]+buff[16]+buff[17]));
					break;
				}//switch(obstacles[index(z,y,x)])

				//original implementation
				//				rho=0.0;
				//				rho+=D3_hlp.Q0[index(z,y,x)]+D3_hlp.Q1[index(z,y,x)]+D3_hlp.Q2[index(z,y,x)]+D3_hlp.Q3[index(z,y,x)];
				//				rho+=D3_hlp.Q4[index(z,y,x)]+D3_hlp.Q5[index(z,y,x)]+D3_hlp.Q6[index(z,y,x)]+D3_hlp.Q7[index(z,y,x)];
				//				rho+=D3_hlp.Q8[index(z,y,x)]+D3_hlp.Q9[index(z,y,x)]+D3_hlp.Q10[index(z,y,x)]+D3_hlp.Q11[index(z,y,x)];
				//				rho+=D3_hlp.Q12[index(z,y,x)]+D3_hlp.Q13[index(z,y,x)]+D3_hlp.Q14[index(z,y,x)]+D3_hlp.Q15[index(z,y,x)];
				//				rho+=D3_hlp.Q16[index(z,y,x)]+D3_hlp.Q17[index(z,y,x)]+D3_hlp.Q18[index(z,y,x)];
				//				reciprocal_rho=1.0/rho;

				//...........x-, and y- velocity components

				//				switch(obstacles[index(z,y,x)]){
				//				case 1:
				//					u_x = 0.0;
				//					u_y = 0.0;
				//					u_z = 0.0;
				//					break;
				//				default:
				//					u_x = (FLOATING) reciprocal_rho*(D3_hlp.Q1[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
				//							D3_hlp.Q11[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] -
				//							(D3_hlp.Q3[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] +
				//									D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)]));
				//
				//					u_y = (FLOATING) reciprocal_rho*(D3_hlp.Q2[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] +
				//							D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q15[index(z,y,x)] -
				//							(D3_hlp.Q4[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
				//									D3_hlp.Q17[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)]));
				//
				//					u_z = (FLOATING) reciprocal_rho*(D3_hlp.Q5[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] +
				//							D3_hlp.Q15[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)] -
				//							(D3_hlp.Q6[index(z,y,x)] + D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q11[index(z,y,x)] +
				//									D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q17[index(z,y,x)]));
				//					break;
				//				}//switch(obstacles[index(z,y,x)])




				u_squ = (FLOATING)  u_x*u_x + u_y*u_y + u_z*u_z;
				temp_factor= 0.5*(2.0* c_squ - u_squ)/c_squ;
				//u_squ = (FLOATING)  pow(u_x,2) + pow(u_y,2) + pow(u_z,2);


				/*...........n- velocity compnents (n = lattice node connection vectors)
					c...........this is only necessary for clearence, and only 3 speeds would
					c...........be necessary*/


				//WARNING!!!! o pinakas autos exei tropopoihmena indices!!!!
				u_n[0]= 0.0; //SHOULD NEVER USED!
				u_n[1] =   u_x;
				u_n[2] =         u_y;
				u_n[3] = - u_x;
				u_n[4] =       - u_y;
				u_n[5] =   u_z;
				u_n[6] =       - u_z;
				u_n[7] =   u_x + u_y;
				u_n[8] = - u_x + u_y;
				u_n[9] = - u_x - u_y;
				u_n[10] =   u_x - u_y;
				u_n[11] =   u_x - u_z;
				u_n[12] = - u_x - u_z;
				u_n[13] = - u_x + u_z;
				u_n[14] =   u_x + u_z;
				u_n[15] =   u_z + u_y;
				u_n[16] = - u_z + u_y;
				u_n[17] = - u_z - u_y;
				u_n[18] =   u_z - u_y;

#pragma unroll
				for(int i=0; i<DENSITIES; ++i){
					u_n__over__c_squ[i]=reciprocal_c_squ*u_n[i];
					u_n__over__c_squ__squared_and_halved[i]=0.5*u_n__over__c_squ[i]*u_n__over__c_squ[i];
				}

				/*c...........equilibrium densities
					c...........this can be rewritten to improve computational performance
					c...........considerabely !
					c
					c...........zero velocity density
					c*/
				//memory optimised implementation! WARNING!!! different from the original case!

				//two_x_c_squ_sqared=2.0*c_squ*c_squ;
				omega_x_rho_x_t_0=omega_x_t_0*rho;
				omega_x_rho_x_t_1=omega_x_t_1*rho;
				omega_x_rho_x_t_2=omega_x_t_2*rho;


				//				//...........relaxation step


				//omega_x_rho_x_t_0*(1.0 - 0.5*u_squ/c_squ);
				D3.Q0[index(z,y,x)]=buff[0]*one_minus_omega+omega_x_rho_x_t_0*(u_n__over__c_squ__squared_and_halved[0]+u_n__over__c_squ[0]+temp_factor);




				D3.Q1[index(z,y,x)]=buff[1]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[1]+u_n__over__c_squ[1]+temp_factor);
				D3.Q2[index(z,y,x)]=buff[2]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[2]+u_n__over__c_squ[2]+temp_factor);
				D3.Q3[index(z,y,x)]=buff[3]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[3]+u_n__over__c_squ[3]+temp_factor);
				D3.Q4[index(z,y,x)]=buff[4]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[4]+u_n__over__c_squ[4]+temp_factor);
				D3.Q5[index(z,y,x)]=buff[5]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[5]+u_n__over__c_squ[5]+temp_factor);
				D3.Q6[index(z,y,x)]=buff[6]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[6]+u_n__over__c_squ[6]+temp_factor);



				D3.Q7[index(z,y,x)]= buff[ 7]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 7]+u_n__over__c_squ[ 7]+temp_factor);
				D3.Q8[index(z,y,x)]= buff[ 8]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 8]+u_n__over__c_squ[ 8]+temp_factor);
				D3.Q9[index(z,y,x)]= buff[ 9]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 9]+u_n__over__c_squ[ 9]+temp_factor);
				D3.Q10[index(z,y,x)]=buff[10]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[10]+u_n__over__c_squ[10]+temp_factor);
				D3.Q11[index(z,y,x)]=buff[11]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[11]+u_n__over__c_squ[11]+temp_factor);
				D3.Q12[index(z,y,x)]=buff[12]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[12]+u_n__over__c_squ[12]+temp_factor);
				D3.Q13[index(z,y,x)]=buff[13]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[13]+u_n__over__c_squ[13]+temp_factor);
				D3.Q14[index(z,y,x)]=buff[14]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[14]+u_n__over__c_squ[14]+temp_factor);
				D3.Q15[index(z,y,x)]=buff[15]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[15]+u_n__over__c_squ[15]+temp_factor);
				D3.Q16[index(z,y,x)]=buff[16]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[16]+u_n__over__c_squ[16]+temp_factor);
				D3.Q17[index(z,y,x)]=buff[17]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[17]+u_n__over__c_squ[17]+temp_factor);
				D3.Q18[index(z,y,x)]=buff[18]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[18]+u_n__over__c_squ[18]+temp_factor);

				//original implementation
				//				n_equ[0] = t_0  * rho*(1.0 - u_squ / (2.0 * c_squ));
				//
				//								//...........axis speeds (factor: t_1)
				//				#pragma unroll
				//								for (int i = 1 ; i< 7; ++i){
				//									n_equ[i] = t_1 * rho*(1.0 + u_n[i] / c_squ
				//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
				//											- u_squ / (2.0  * c_squ));
				//								}
				//
				//								//...........diagonal speeds (factor: t_2)
				//				#pragma unroll
				//								for (int i = 7 ; i< 19; ++i){
				//									n_equ[i] = t_2  * rho*(1.0 + u_n[i] / c_squ
				//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
				//											- u_squ / (2.0  * c_squ));
				//								}

				//				D3.Q0[index(z,y,x)]=D3_hlp.Q0[index(z,y,x)]+omega*(n_equ[0] - D3_hlp.Q0[index(z,y,x)]);
				//								D3.Q1[index(z,y,x)]=D3_hlp.Q1[index(z,y,x)]+omega*(n_equ[1] - D3_hlp.Q1[index(z,y,x)]);
				//								D3.Q2[index(z,y,x)]=D3_hlp.Q2[index(z,y,x)]+omega*(n_equ[2] - D3_hlp.Q2[index(z,y,x)]);
				//								D3.Q3[index(z,y,x)]=D3_hlp.Q3[index(z,y,x)]+omega*(n_equ[3] - D3_hlp.Q3[index(z,y,x)]);
				//								D3.Q4[index(z,y,x)]=D3_hlp.Q4[index(z,y,x)]+omega*(n_equ[4] - D3_hlp.Q4[index(z,y,x)]);
				//								D3.Q5[index(z,y,x)]=D3_hlp.Q5[index(z,y,x)]+omega*(n_equ[5] - D3_hlp.Q5[index(z,y,x)]);
				//								D3.Q6[index(z,y,x)]=D3_hlp.Q6[index(z,y,x)]+omega*(n_equ[6] - D3_hlp.Q6[index(z,y,x)]);
				//								D3.Q7[index(z,y,x)]=D3_hlp.Q7[index(z,y,x)]+omega*(n_equ[7] - D3_hlp.Q7[index(z,y,x)]);
				//								D3.Q8[index(z,y,x)]=D3_hlp.Q8[index(z,y,x)]+omega*(n_equ[8] - D3_hlp.Q8[index(z,y,x)]);
				//								D3.Q9[index(z,y,x)]=D3_hlp.Q9[index(z,y,x)]+omega*(n_equ[9] - D3_hlp.Q9[index(z,y,x)]);
				//								D3.Q10[index(z,y,x)]=D3_hlp.Q10[index(z,y,x)]+omega*(n_equ[10] - D3_hlp.Q10[index(z,y,x)]);
				//								D3.Q11[index(z,y,x)]=D3_hlp.Q11[index(z,y,x)]+omega*(n_equ[11] - D3_hlp.Q11[index(z,y,x)]);
				//								D3.Q12[index(z,y,x)]=D3_hlp.Q12[index(z,y,x)]+omega*(n_equ[12] - D3_hlp.Q12[index(z,y,x)]);
				//								D3.Q13[index(z,y,x)]=D3_hlp.Q13[index(z,y,x)]+omega*(n_equ[13] - D3_hlp.Q13[index(z,y,x)]);
				//								D3.Q14[index(z,y,x)]=D3_hlp.Q14[index(z,y,x)]+omega*(n_equ[14] - D3_hlp.Q14[index(z,y,x)]);
				//								D3.Q15[index(z,y,x)]=D3_hlp.Q15[index(z,y,x)]+omega*(n_equ[15] - D3_hlp.Q15[index(z,y,x)]);
				//								D3.Q16[index(z,y,x)]=D3_hlp.Q16[index(z,y,x)]+omega*(n_equ[16] - D3_hlp.Q16[index(z,y,x)]);
				//								D3.Q17[index(z,y,x)]=D3_hlp.Q17[index(z,y,x)]+omega*(n_equ[17] - D3_hlp.Q17[index(z,y,x)]);
				//								D3.Q18[index(z,y,x)]=D3_hlp.Q18[index(z,y,x)]+omega*(n_equ[18] - D3_hlp.Q18[index(z,y,x)]);

				if (x == lx-2) {
					u_previous_spatial_boundary[index2D(z,y)] = u_x;
					v_previous_spatial_boundary[index2D(z,y)] = u_y;
					w_previous_spatial_boundary[index2D(z,y)] = u_z;

					u_current[index2D(z,y)] = u_x;
					v_current[index2D(z,y)] = u_y;
					w_current[index2D(z,y)] = u_z;

				}//if (x == lx-2)
			}//for (x = 0 ; x< lx; ++x)
		}//for (y = 0 ; y< ly ; ++y)
	}//for (z = 0 ; z< lz ; ++z)
#ifdef DEBUG
	cout << " #LBM relaxation OK!" << endl;
#endif
}

void LBM::initial_relaxation(){





	/*One-step density relaxation process

				c.......density relaxation: a single time relaxation with relaxation
				c       parameter omega is applied here. This step is only "local",
				c       nothing is propagated through the lattice.
				c*/

	int  x,y,z;
	FLOATING  u_x=0.0, u_y=0.0, u_z=0.0, u_squ=0.0, rho=0.0;
	const FLOATING  tau=3.0*nu + 0.5, omega = 1.0 /tau; //	omega=1.0/(3.0*nu+0.5);



	FLOATING u_n[19], n_equ[19];

	//....square speed of sound
	/*      compute the out let velocity with a convevtive boundary condition
					c.....loop over all nodes
					c.....attention: actual densities are stored after the propagation
					c                step in the help-array n_hlp !*/



	for (z = 0 ; z< lz ; ++z){
		for (y = 0 ; y< ly ; ++y){
			for (x = 0 ; x< lx; ++x){

				/*c.........only free nodes are considered here
					!if (.not. obstacles[z][y][x]) then
					c...........integral local density
					c...........initialize variable ro*/


				rho=0.0;
				rho+=D3_hlp.Q0[index(z,y,x)]+D3_hlp.Q1[index(z,y,x)]+D3_hlp.Q2[index(z,y,x)]+D3_hlp.Q3[index(z,y,x)];
				rho+=D3_hlp.Q4[index(z,y,x)]+D3_hlp.Q5[index(z,y,x)]+D3_hlp.Q6[index(z,y,x)]+D3_hlp.Q7[index(z,y,x)];
				rho+=D3_hlp.Q8[index(z,y,x)]+D3_hlp.Q9[index(z,y,x)]+D3_hlp.Q10[index(z,y,x)]+D3_hlp.Q11[index(z,y,x)];
				rho+=D3_hlp.Q12[index(z,y,x)]+D3_hlp.Q13[index(z,y,x)]+D3_hlp.Q14[index(z,y,x)]+D3_hlp.Q15[index(z,y,x)];
				rho+=D3_hlp.Q16[index(z,y,x)]+D3_hlp.Q17[index(z,y,x)]+D3_hlp.Q18[index(z,y,x)];


				//...........x-, and y- velocity components


				if ( obstacles[index(z,y,x)]==1 ) {
					u_x = 0.0;
					u_y = 0.0;
					u_z = 0.0;
				}else{





					u_x = (FLOATING) (D3_hlp.Q1[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
							D3_hlp.Q11[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] -
							(D3_hlp.Q3[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] +
									D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)])) / rho;

					u_y = (FLOATING) (D3_hlp.Q2[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] +
							D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q15[index(z,y,x)] -
							(D3_hlp.Q4[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
									D3_hlp.Q17[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)])) / rho;

					u_z = (FLOATING) (D3_hlp.Q5[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] +
							D3_hlp.Q15[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)] -
							(D3_hlp.Q6[index(z,y,x)] + D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q11[index(z,y,x)] +
									D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q17[index(z,y,x)])) / rho;

				}

				u_squ = (FLOATING)  u_x*u_x + u_y*u_y + u_z*u_z;
				//u_squ = (FLOATING)  pow(u_x,2) + pow(u_y,2) + pow(u_z,2);


				/*...........n- velocity compnents (n = lattice node connection vectors)
					c...........this is only necessary for clearence, and only 3 speeds would
					c...........be necessary*/

				//WARNING!!!! o pinakas autos exei tropopoihmena indices!!!!
				u_n[0]= 0.0; //SHOULD NEVER USED!
				u_n[1] =   u_x;
				u_n[2] =         u_y;
				u_n[3] = - u_x;
				u_n[4] =       - u_y;
				u_n[5] =   u_z;
				u_n[6] =       - u_z;
				u_n[7] =   u_x + u_y;
				u_n[8] = - u_x + u_y;
				u_n[9] = - u_x - u_y;
				u_n[10] =   u_x - u_y;
				u_n[11] =   u_x - u_z;
				u_n[12] = - u_x - u_z;
				u_n[13] = - u_x + u_z;
				u_n[14] =   u_x + u_z;
				u_n[15] =   u_z + u_y;
				u_n[16] = - u_z + u_y;
				u_n[17] = - u_z - u_y;
				u_n[18] =   u_z - u_y;

				/*c...........equilibrium densities
					c...........this can be rewritten to improve computational performance
					c...........considerabely !
					c
					c...........zero velocity density
					c*/
				n_equ[0] = t_0  * rho*(1.0 - u_squ / (2.0 * c_squ));

				//...........axis speeds (factor: t_1)
				for (int i = 1 ; i< 7; ++i){
					n_equ[i] = t_1 * rho*(1.0 + u_n[i] / c_squ
							+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
							- u_squ / (2.0  * c_squ));
				}

				//...........diagonal speeds (factor: t_2)
				for (int i = 7 ; i< 19; ++i){
					n_equ[i] = t_2  * rho*(1.0 + u_n[i] / c_squ
							+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
							- u_squ / (2.0  * c_squ));
				}


				//...........relaxation step




				D3.Q0[index(z,y,x)]=D3_hlp.Q0[index(z,y,x)]+omega*(n_equ[0] - D3_hlp.Q0[index(z,y,x)]);
				D3.Q1[index(z,y,x)]=D3_hlp.Q1[index(z,y,x)]+omega*(n_equ[1] - D3_hlp.Q1[index(z,y,x)]);
				D3.Q2[index(z,y,x)]=D3_hlp.Q2[index(z,y,x)]+omega*(n_equ[2] - D3_hlp.Q2[index(z,y,x)]);
				D3.Q3[index(z,y,x)]=D3_hlp.Q3[index(z,y,x)]+omega*(n_equ[3] - D3_hlp.Q3[index(z,y,x)]);
				D3.Q4[index(z,y,x)]=D3_hlp.Q4[index(z,y,x)]+omega*(n_equ[4] - D3_hlp.Q4[index(z,y,x)]);
				D3.Q5[index(z,y,x)]=D3_hlp.Q5[index(z,y,x)]+omega*(n_equ[5] - D3_hlp.Q5[index(z,y,x)]);
				D3.Q6[index(z,y,x)]=D3_hlp.Q6[index(z,y,x)]+omega*(n_equ[6] - D3_hlp.Q6[index(z,y,x)]);
				D3.Q7[index(z,y,x)]=D3_hlp.Q7[index(z,y,x)]+omega*(n_equ[7] - D3_hlp.Q7[index(z,y,x)]);
				D3.Q8[index(z,y,x)]=D3_hlp.Q8[index(z,y,x)]+omega*(n_equ[8] - D3_hlp.Q8[index(z,y,x)]);
				D3.Q9[index(z,y,x)]=D3_hlp.Q9[index(z,y,x)]+omega*(n_equ[9] - D3_hlp.Q9[index(z,y,x)]);
				D3.Q10[index(z,y,x)]=D3_hlp.Q10[index(z,y,x)]+omega*(n_equ[10] - D3_hlp.Q10[index(z,y,x)]);
				D3.Q11[index(z,y,x)]=D3_hlp.Q11[index(z,y,x)]+omega*(n_equ[11] - D3_hlp.Q11[index(z,y,x)]);
				D3.Q12[index(z,y,x)]=D3_hlp.Q12[index(z,y,x)]+omega*(n_equ[12] - D3_hlp.Q12[index(z,y,x)]);
				D3.Q13[index(z,y,x)]=D3_hlp.Q13[index(z,y,x)]+omega*(n_equ[13] - D3_hlp.Q13[index(z,y,x)]);
				D3.Q14[index(z,y,x)]=D3_hlp.Q14[index(z,y,x)]+omega*(n_equ[14] - D3_hlp.Q14[index(z,y,x)]);
				D3.Q15[index(z,y,x)]=D3_hlp.Q15[index(z,y,x)]+omega*(n_equ[15] - D3_hlp.Q15[index(z,y,x)]);
				D3.Q16[index(z,y,x)]=D3_hlp.Q16[index(z,y,x)]+omega*(n_equ[16] - D3_hlp.Q16[index(z,y,x)]);
				D3.Q17[index(z,y,x)]=D3_hlp.Q17[index(z,y,x)]+omega*(n_equ[17] - D3_hlp.Q17[index(z,y,x)]);
				D3.Q18[index(z,y,x)]=D3_hlp.Q18[index(z,y,x)]+omega*(n_equ[18] - D3_hlp.Q18[index(z,y,x)]);

				//at the penultimat slice, save previous and current slices
				if (x == lx-2) {
					u_previous_spatial_boundary[index2D(z,y)] = u_x;
					v_previous_spatial_boundary[index2D(z,y)] = u_y;
					w_previous_spatial_boundary[index2D(z,y)] = u_z;

					u_current[index2D(z,y)] = u_x;
					v_current[index2D(z,y)] = u_y;
					w_current[index2D(z,y)] = u_z;

					//the following 3 lines correspond to time_unit==0!!!
					u_previous_temporal_boundary[index2D(z,y)] = u_current[index2D(z,y)];
					v_previous_temporal_boundary[index2D(z,y)] = v_current[index2D(z,y)];
					w_previous_temporal_boundary[index2D(z,y)] = w_current[index2D(z,y)];

				}//if (x == lx-2)
			}//for (x = 0 ; x< lx; ++x)
		}//for (y = 0 ; y< ly ; ++y)
	}//for (z = 0 ; z< lz ; ++z)
#ifdef DEBUG
	cout << " #LBM relaxation OK!" << endl;
#endif
}

//__global__
//void relaxation_kernel(int lx, int ly, int lz, FLOATING reynolds, FLOATING nu, FLOATING r_small,
//		FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ, FLOATING omega, FLOATING one_minus_omega,
//		FLOATING reciprocal_c_squ,lattice D3, lattice D3_hlp, int *obstacles_d,
//		FLOATING *u_previous_spatial_boundary, FLOATING *v_previous_spatial_boundary, FLOATING *w_previous_spatial_boundary,
//		FLOATING *u_current, FLOATING *v_current, FLOATING *w_current){
//
//
//
//
//
//	/*One-step density relaxation process
//
//				c.......density relaxation: a single time relaxation with relaxation
//				c       parameter omega is applied here. This step is only "local",
//				c       nothing is propagated through the lattice.
//				c*/
//
//	int  x,y,z;
//	FLOATING  u_x=0.0, u_y=0.0, u_z=0.0, u_squ=0.0, rho=0.0, reciprocal_rho=0.0;
//
//
//
//
//	FLOATING u_n[19];
//	//	FLOATING n_equ[19];
//	FLOATING buff[19];
//
//	//	FLOATING u_n_squared[19];
//	//FLOATING two_x_c_squ_sqared;
//	const FLOATING omega_x_t_0=omega*t_0, omega_x_t_1=omega*t_1, omega_x_t_2=omega*t_2;
//	FLOATING omega_x_rho_x_t_0, omega_x_rho_x_t_1, omega_x_rho_x_t_2;
//	FLOATING temp_factor;
//	FLOATING u_n__over__c_squ[19];
//	FLOATING u_n__over__c_squ__squared_and_halved[19];
//
//	//....square speed of sound
//	/*      compute the out let velocity with a convevtive boundary condition
//					c.....loop over all nodes
//					c.....attention: actual densities are stored after the propagation
//					c                step in the help-array n_hlp !*/
//
//
//#pragma unroll
//	for (z = 0 ; z< lz ; ++z){
//#pragma unroll
//		for (y = 0 ; y< ly ; ++y){
//#pragma unroll
//			for (x = 0 ; x< lx; ++x){
//
//				/*c.........only free nodes are considered here
//					!if (.not. obstacles[z][y][x]) then
//					c...........integral local density
//					c...........initialize variable ro*/
//				//memory optimised implementation
//				buff[0]=D3_hlp.Q0[index(z,y,x)];
//				buff[1]=D3_hlp.Q1[index(z,y,x)];
//				buff[2]=D3_hlp.Q2[index(z,y,x)];
//				buff[3]=D3_hlp.Q3[index(z,y,x)];
//				buff[4]=D3_hlp.Q4[index(z,y,x)];
//				buff[5]=D3_hlp.Q5[index(z,y,x)];
//				buff[6]=D3_hlp.Q6[index(z,y,x)];
//				buff[7]=D3_hlp.Q7[index(z,y,x)];
//				buff[8]=D3_hlp.Q8[index(z,y,x)];
//				buff[9]=D3_hlp.Q9[index(z,y,x)];
//				buff[10]=D3_hlp.Q10[index(z,y,x)];
//				buff[11]=D3_hlp.Q11[index(z,y,x)];
//				buff[12]=D3_hlp.Q12[index(z,y,x)];
//				buff[13]=D3_hlp.Q13[index(z,y,x)];
//				buff[14]=D3_hlp.Q14[index(z,y,x)];
//				buff[15]=D3_hlp.Q15[index(z,y,x)];
//				buff[16]=D3_hlp.Q16[index(z,y,x)];
//				buff[17]=D3_hlp.Q17[index(z,y,x)];
//				buff[18]=D3_hlp.Q18[index(z,y,x)];
//
//				rho=0.0;
//				for(int k=0; k<DENSITIES; ++k)
//					rho+=buff[k];
//
//				//	rho=accumulate(buff, buff+DENSITIES, 0.0);
//
//				reciprocal_rho=1.0/rho;
//
//
//
//
//				switch(obstacles_d[index(z,y,x)]){
//				case 1:
//					u_x = 0.0;
//					u_y = 0.0;
//					u_z = 0.0;
//					break;
//				default:
//					u_x = 0.0;
//					u_x =  reciprocal_rho*(buff[1] + buff[7] + buff[10] +buff[11] + buff[14]-
//							(buff[3] + buff[8] + buff[9] +buff[12] + buff[13]));
//
//					u_y = 0.0;
//					u_y =  reciprocal_rho*(buff[2]+buff[8]+buff[7]+buff[16] + buff[15] -
//							(buff[4] + buff[9] + buff[10] +buff[17] + buff[18]));
//
//					u_z = 0.0;
//					u_z =  reciprocal_rho*(buff[5]+buff[13]+buff[14]+buff[15]+buff[18]-
//							(buff[6]+buff[12]+buff[11]+buff[16]+buff[17]));
//					break;
//				}//switch(obstacles[index(z,y,x)])
//
//				//original implementation
//				//				rho=0.0;
//				//				rho+=D3_hlp.Q0[index(z,y,x)]+D3_hlp.Q1[index(z,y,x)]+D3_hlp.Q2[index(z,y,x)]+D3_hlp.Q3[index(z,y,x)];
//				//				rho+=D3_hlp.Q4[index(z,y,x)]+D3_hlp.Q5[index(z,y,x)]+D3_hlp.Q6[index(z,y,x)]+D3_hlp.Q7[index(z,y,x)];
//				//				rho+=D3_hlp.Q8[index(z,y,x)]+D3_hlp.Q9[index(z,y,x)]+D3_hlp.Q10[index(z,y,x)]+D3_hlp.Q11[index(z,y,x)];
//				//				rho+=D3_hlp.Q12[index(z,y,x)]+D3_hlp.Q13[index(z,y,x)]+D3_hlp.Q14[index(z,y,x)]+D3_hlp.Q15[index(z,y,x)];
//				//				rho+=D3_hlp.Q16[index(z,y,x)]+D3_hlp.Q17[index(z,y,x)]+D3_hlp.Q18[index(z,y,x)];
//				//				reciprocal_rho=1.0/rho;
//
//				//...........x-, and y- velocity components
//
//				//				switch(obstacles[index(z,y,x)]){
//				//				case 1:
//				//					u_x = 0.0;
//				//					u_y = 0.0;
//				//					u_z = 0.0;
//				//					break;
//				//				default:
//				//					u_x = (FLOATING) reciprocal_rho*(D3_hlp.Q1[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
//				//							D3_hlp.Q11[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] -
//				//							(D3_hlp.Q3[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] +
//				//									D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)]));
//				//
//				//					u_y = (FLOATING) reciprocal_rho*(D3_hlp.Q2[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] +
//				//							D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q15[index(z,y,x)] -
//				//							(D3_hlp.Q4[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
//				//									D3_hlp.Q17[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)]));
//				//
//				//					u_z = (FLOATING) reciprocal_rho*(D3_hlp.Q5[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] +
//				//							D3_hlp.Q15[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)] -
//				//							(D3_hlp.Q6[index(z,y,x)] + D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q11[index(z,y,x)] +
//				//									D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q17[index(z,y,x)]));
//				//					break;
//				//				}//switch(obstacles[index(z,y,x)])
//
//
//
//
//				u_squ = (FLOATING)  u_x*u_x + u_y*u_y + u_z*u_z;
//				temp_factor= 0.5*(2.0* c_squ - u_squ)/c_squ;
//				//u_squ = (FLOATING)  pow(u_x,2) + pow(u_y,2) + pow(u_z,2);
//
//
//				/*...........n- velocity compnents (n = lattice node connection vectors)
//					c...........this is only necessary for clearence, and only 3 speeds would
//					c...........be necessary*/
//
//
//				//WARNING!!!! o pinakas autos exei tropopoihmena indices!!!!
//				u_n[0]= 0.0; //SHOULD NEVER USED!
//				u_n[1] =   u_x;
//				u_n[2] =         u_y;
//				u_n[3] = - u_x;
//				u_n[4] =       - u_y;
//				u_n[5] =   u_z;
//				u_n[6] =       - u_z;
//				u_n[7] =   u_x + u_y;
//				u_n[8] = - u_x + u_y;
//				u_n[9] = - u_x - u_y;
//				u_n[10] =   u_x - u_y;
//				u_n[11] =   u_x - u_z;
//				u_n[12] = - u_x - u_z;
//				u_n[13] = - u_x + u_z;
//				u_n[14] =   u_x + u_z;
//				u_n[15] =   u_z + u_y;
//				u_n[16] = - u_z + u_y;
//				u_n[17] = - u_z - u_y;
//				u_n[18] =   u_z - u_y;
//
//#pragma unroll
//				for(int i=0; i<DENSITIES; ++i){
//					u_n__over__c_squ[i]=reciprocal_c_squ*u_n[i];
//					u_n__over__c_squ__squared_and_halved[i]=0.5*u_n__over__c_squ[i]*u_n__over__c_squ[i];
//				}
//
//				/*c...........equilibrium densities
//					c...........this can be rewritten to improve computational performance
//					c...........considerabely !
//					c
//					c...........zero velocity density
//					c*/
//				//memory optimised implementation! WARNING!!! different from the original case!
//
//				//two_x_c_squ_sqared=2.0*c_squ*c_squ;
//				omega_x_rho_x_t_0=omega_x_t_0*rho;
//				omega_x_rho_x_t_1=omega_x_t_1*rho;
//				omega_x_rho_x_t_2=omega_x_t_2*rho;
//
//
//				//				//...........relaxation step
//
//
//				//omega_x_rho_x_t_0*(1.0 - 0.5*u_squ/c_squ);
//				D3.Q0[index(z,y,x)]=buff[0]*one_minus_omega+omega_x_rho_x_t_0*(u_n__over__c_squ__squared_and_halved[0]+u_n__over__c_squ[0]+temp_factor);
//
//
//
//
//				D3.Q1[index(z,y,x)]=buff[1]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[1]+u_n__over__c_squ[1]+temp_factor);
//				D3.Q2[index(z,y,x)]=buff[2]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[2]+u_n__over__c_squ[2]+temp_factor);
//				D3.Q3[index(z,y,x)]=buff[3]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[3]+u_n__over__c_squ[3]+temp_factor);
//				D3.Q4[index(z,y,x)]=buff[4]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[4]+u_n__over__c_squ[4]+temp_factor);
//				D3.Q5[index(z,y,x)]=buff[5]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[5]+u_n__over__c_squ[5]+temp_factor);
//				D3.Q6[index(z,y,x)]=buff[6]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[6]+u_n__over__c_squ[6]+temp_factor);
//
//
//
//				D3.Q7[index(z,y,x)]= buff[ 7]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 7]+u_n__over__c_squ[ 7]+temp_factor);
//				D3.Q8[index(z,y,x)]= buff[ 8]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 8]+u_n__over__c_squ[ 8]+temp_factor);
//				D3.Q9[index(z,y,x)]= buff[ 9]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 9]+u_n__over__c_squ[ 9]+temp_factor);
//				D3.Q10[index(z,y,x)]=buff[10]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[10]+u_n__over__c_squ[10]+temp_factor);
//				D3.Q11[index(z,y,x)]=buff[11]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[11]+u_n__over__c_squ[11]+temp_factor);
//				D3.Q12[index(z,y,x)]=buff[12]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[12]+u_n__over__c_squ[12]+temp_factor);
//				D3.Q13[index(z,y,x)]=buff[13]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[13]+u_n__over__c_squ[13]+temp_factor);
//				D3.Q14[index(z,y,x)]=buff[14]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[14]+u_n__over__c_squ[14]+temp_factor);
//				D3.Q15[index(z,y,x)]=buff[15]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[15]+u_n__over__c_squ[15]+temp_factor);
//				D3.Q16[index(z,y,x)]=buff[16]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[16]+u_n__over__c_squ[16]+temp_factor);
//				D3.Q17[index(z,y,x)]=buff[17]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[17]+u_n__over__c_squ[17]+temp_factor);
//				D3.Q18[index(z,y,x)]=buff[18]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[18]+u_n__over__c_squ[18]+temp_factor);
//
//				//original implementation
//				//				n_equ[0] = t_0  * rho*(1.0 - u_squ / (2.0 * c_squ));
//				//
//				//								//...........axis speeds (factor: t_1)
//				//				#pragma unroll
//				//								for (int i = 1 ; i< 7; ++i){
//				//									n_equ[i] = t_1 * rho*(1.0 + u_n[i] / c_squ
//				//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
//				//											- u_squ / (2.0  * c_squ));
//				//								}
//				//
//				//								//...........diagonal speeds (factor: t_2)
//				//				#pragma unroll
//				//								for (int i = 7 ; i< 19; ++i){
//				//									n_equ[i] = t_2  * rho*(1.0 + u_n[i] / c_squ
//				//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
//				//											- u_squ / (2.0  * c_squ));
//				//								}
//
//				//				D3.Q0[index(z,y,x)]=D3_hlp.Q0[index(z,y,x)]+omega*(n_equ[0] - D3_hlp.Q0[index(z,y,x)]);
//				//								D3.Q1[index(z,y,x)]=D3_hlp.Q1[index(z,y,x)]+omega*(n_equ[1] - D3_hlp.Q1[index(z,y,x)]);
//				//								D3.Q2[index(z,y,x)]=D3_hlp.Q2[index(z,y,x)]+omega*(n_equ[2] - D3_hlp.Q2[index(z,y,x)]);
//				//								D3.Q3[index(z,y,x)]=D3_hlp.Q3[index(z,y,x)]+omega*(n_equ[3] - D3_hlp.Q3[index(z,y,x)]);
//				//								D3.Q4[index(z,y,x)]=D3_hlp.Q4[index(z,y,x)]+omega*(n_equ[4] - D3_hlp.Q4[index(z,y,x)]);
//				//								D3.Q5[index(z,y,x)]=D3_hlp.Q5[index(z,y,x)]+omega*(n_equ[5] - D3_hlp.Q5[index(z,y,x)]);
//				//								D3.Q6[index(z,y,x)]=D3_hlp.Q6[index(z,y,x)]+omega*(n_equ[6] - D3_hlp.Q6[index(z,y,x)]);
//				//								D3.Q7[index(z,y,x)]=D3_hlp.Q7[index(z,y,x)]+omega*(n_equ[7] - D3_hlp.Q7[index(z,y,x)]);
//				//								D3.Q8[index(z,y,x)]=D3_hlp.Q8[index(z,y,x)]+omega*(n_equ[8] - D3_hlp.Q8[index(z,y,x)]);
//				//								D3.Q9[index(z,y,x)]=D3_hlp.Q9[index(z,y,x)]+omega*(n_equ[9] - D3_hlp.Q9[index(z,y,x)]);
//				//								D3.Q10[index(z,y,x)]=D3_hlp.Q10[index(z,y,x)]+omega*(n_equ[10] - D3_hlp.Q10[index(z,y,x)]);
//				//								D3.Q11[index(z,y,x)]=D3_hlp.Q11[index(z,y,x)]+omega*(n_equ[11] - D3_hlp.Q11[index(z,y,x)]);
//				//								D3.Q12[index(z,y,x)]=D3_hlp.Q12[index(z,y,x)]+omega*(n_equ[12] - D3_hlp.Q12[index(z,y,x)]);
//				//								D3.Q13[index(z,y,x)]=D3_hlp.Q13[index(z,y,x)]+omega*(n_equ[13] - D3_hlp.Q13[index(z,y,x)]);
//				//								D3.Q14[index(z,y,x)]=D3_hlp.Q14[index(z,y,x)]+omega*(n_equ[14] - D3_hlp.Q14[index(z,y,x)]);
//				//								D3.Q15[index(z,y,x)]=D3_hlp.Q15[index(z,y,x)]+omega*(n_equ[15] - D3_hlp.Q15[index(z,y,x)]);
//				//								D3.Q16[index(z,y,x)]=D3_hlp.Q16[index(z,y,x)]+omega*(n_equ[16] - D3_hlp.Q16[index(z,y,x)]);
//				//								D3.Q17[index(z,y,x)]=D3_hlp.Q17[index(z,y,x)]+omega*(n_equ[17] - D3_hlp.Q17[index(z,y,x)]);
//				//								D3.Q18[index(z,y,x)]=D3_hlp.Q18[index(z,y,x)]+omega*(n_equ[18] - D3_hlp.Q18[index(z,y,x)]);
//
//				if (x == lx-2) {
//					u_previous_spatial_boundary[index2D(z,y)] = u_x;
//					v_previous_spatial_boundary[index2D(z,y)] = u_y;
//					w_previous_spatial_boundary[index2D(z,y)] = u_z;
//
//					u_current[index2D(z,y)] = u_x;
//					v_current[index2D(z,y)] = u_y;
//					w_current[index2D(z,y)] = u_z;
//
//				}//if (x == lx-2)
//			}//for (x = 0 ; x< lx; ++x)
//		}//for (y = 0 ; y< ly ; ++y)
//	}//for (z = 0 ; z< lz ; ++z)
//#ifdef DEBUG
//	cout << " #LBM relaxation OK!" << endl;
//#endif
//}
//
//__global__
//void relaxation_kernel_v2(int lx, int ly, int lz, FLOATING reynolds, FLOATING nu, FLOATING r_small,
//		FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ, FLOATING omega, FLOATING one_minus_omega,
//		FLOATING reciprocal_c_squ,
//		FLOATING *hlp_Q0, FLOATING *hlp_Q1, FLOATING *hlp_Q2, FLOATING *hlp_Q3,
//		FLOATING *hlp_Q4, FLOATING *hlp_Q5, FLOATING *hlp_Q6, FLOATING *hlp_Q7,
//		FLOATING *hlp_Q8, FLOATING *hlp_Q9, FLOATING *hlp_Q10, FLOATING *hlp_Q11,
//		FLOATING *hlp_Q12, FLOATING *hlp_Q13, FLOATING *hlp_Q14, FLOATING *hlp_Q15,
//		FLOATING *hlp_Q16, FLOATING *hlp_Q17, FLOATING *hlp_Q18,
//		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
//		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
//		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
//		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
//		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18,
//		int *obstacles_d,
//		FLOATING *u_previous_spatial_boundary, FLOATING *v_previous_spatial_boundary, FLOATING *w_previous_spatial_boundary,
//		FLOATING *u_current, FLOATING *v_current, FLOATING *w_current){
//
//
//
//
//
//	/*One-step density relaxation process
//
//				c.......density relaxation: a single time relaxation with relaxation
//				c       parameter omega is applied here. This step is only "local",
//				c       nothing is propagated through the lattice.
//				c*/
//
//	int  x,y,z;
//	FLOATING  u_x=0.0, u_y=0.0, u_z=0.0, u_squ=0.0, rho=0.0, reciprocal_rho=0.0;
//
//
//
//
//	FLOATING u_n[19];
//	//	FLOATING n_equ[19];
//	FLOATING buff[19];
//
//	//	FLOATING u_n_squared[19];
//	//FLOATING two_x_c_squ_sqared;
//	const FLOATING omega_x_t_0=omega*t_0, omega_x_t_1=omega*t_1, omega_x_t_2=omega*t_2;
//	FLOATING omega_x_rho_x_t_0, omega_x_rho_x_t_1, omega_x_rho_x_t_2;
//	FLOATING temp_factor;
//	FLOATING u_n__over__c_squ[19];
//	FLOATING u_n__over__c_squ__squared_and_halved[19];
//
//	//....square speed of sound
//	/*      compute the out let velocity with a convevtive boundary condition
//					c.....loop over all nodes
//					c.....attention: actual densities are stored after the propagation
//					c                step in the help-array n_hlp !*/
//
//	//
//	//#pragma unroll
//	//	for (z = 0 ; z< lz ; ++z){
//	//#pragma unroll
//	//		for (y = 0 ; y< ly ; ++y){
//	//#pragma unroll
//	//			for (x = 0 ; x< lx; ++x){
//
//	const int tid=blockIdx.x*blockDim.x+threadIdx.x;
//	int rest;
//	int end_of_memory=lz*ly*(lx);
//
//	z=(int) (tid/(ly*lx));
//	rest=tid-z;
//	y=(int)(rest/lx);
//	x=rest-y;
//
//	if (tid<end_of_memory){
//		/*c.........only free nodes are considered here
//					!if (.not. obstacles[z][y][x]) then
//					c...........integral local density
//					c...........initialize variable ro*/
//		//memory optimised implementation
//		buff[0]=hlp_Q0[index(z,y,x)];
//		buff[1]=hlp_Q1[index(z,y,x)];
//		buff[2]=hlp_Q2[index(z,y,x)];
//		buff[3]=hlp_Q3[index(z,y,x)];
//		buff[4]=hlp_Q4[index(z,y,x)];
//		buff[5]=hlp_Q5[index(z,y,x)];
//		buff[6]=hlp_Q6[index(z,y,x)];
//		buff[7]=hlp_Q7[index(z,y,x)];
//		buff[8]=hlp_Q8[index(z,y,x)];
//		buff[9]=hlp_Q9[index(z,y,x)];
//		buff[10]=hlp_Q10[index(z,y,x)];
//		buff[11]=hlp_Q11[index(z,y,x)];
//		buff[12]=hlp_Q12[index(z,y,x)];
//		buff[13]=hlp_Q13[index(z,y,x)];
//		buff[14]=hlp_Q14[index(z,y,x)];
//		buff[15]=hlp_Q15[index(z,y,x)];
//		buff[16]=hlp_Q16[index(z,y,x)];
//		buff[17]=hlp_Q17[index(z,y,x)];
//		buff[18]=hlp_Q18[index(z,y,x)];
//
//		rho=0.0;
//		for(int k=0; k<DENSITIES; ++k)
//			rho+=buff[k];
//
//		//	rho=accumulate(buff, buff+DENSITIES, 0.0);
//
//		reciprocal_rho=1.0/rho;
//
//
//
//
//		switch(obstacles_d[index(z,y,x)]){
//		case 1:
//			u_x = 0.0;
//			u_y = 0.0;
//			u_z = 0.0;
//			break;
//		default:
//			u_x = 0.0;
//			u_x =  reciprocal_rho*(buff[1] + buff[7] + buff[10] +buff[11] + buff[14]-
//					(buff[3] + buff[8] + buff[9] +buff[12] + buff[13]));
//
//			u_y = 0.0;
//			u_y =  reciprocal_rho*(buff[2]+buff[8]+buff[7]+buff[16] + buff[15] -
//					(buff[4] + buff[9] + buff[10] +buff[17] + buff[18]));
//
//			u_z = 0.0;
//			u_z =  reciprocal_rho*(buff[5]+buff[13]+buff[14]+buff[15]+buff[18]-
//					(buff[6]+buff[12]+buff[11]+buff[16]+buff[17]));
//			break;
//		}//switch(obstacles[index(z,y,x)])
//
//		//original implementation
//		//				rho=0.0;
//		//				rho+=D3_hlp.Q0[index(z,y,x)]+D3_hlp.Q1[index(z,y,x)]+D3_hlp.Q2[index(z,y,x)]+D3_hlp.Q3[index(z,y,x)];
//		//				rho+=D3_hlp.Q4[index(z,y,x)]+D3_hlp.Q5[index(z,y,x)]+D3_hlp.Q6[index(z,y,x)]+D3_hlp.Q7[index(z,y,x)];
//		//				rho+=D3_hlp.Q8[index(z,y,x)]+D3_hlp.Q9[index(z,y,x)]+D3_hlp.Q10[index(z,y,x)]+D3_hlp.Q11[index(z,y,x)];
//		//				rho+=D3_hlp.Q12[index(z,y,x)]+D3_hlp.Q13[index(z,y,x)]+D3_hlp.Q14[index(z,y,x)]+D3_hlp.Q15[index(z,y,x)];
//		//				rho+=D3_hlp.Q16[index(z,y,x)]+D3_hlp.Q17[index(z,y,x)]+D3_hlp.Q18[index(z,y,x)];
//		//				reciprocal_rho=1.0/rho;
//
//		//...........x-, and y- velocity components
//
//		//				switch(obstacles[index(z,y,x)]){
//		//				case 1:
//		//					u_x = 0.0;
//		//					u_y = 0.0;
//		//					u_z = 0.0;
//		//					break;
//		//				default:
//		//					u_x = (FLOATING) reciprocal_rho*(D3_hlp.Q1[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
//		//							D3_hlp.Q11[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] -
//		//							(D3_hlp.Q3[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] +
//		//									D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)]));
//		//
//		//					u_y = (FLOATING) reciprocal_rho*(D3_hlp.Q2[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] +
//		//							D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q15[index(z,y,x)] -
//		//							(D3_hlp.Q4[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
//		//									D3_hlp.Q17[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)]));
//		//
//		//					u_z = (FLOATING) reciprocal_rho*(D3_hlp.Q5[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] +
//		//							D3_hlp.Q15[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)] -
//		//							(D3_hlp.Q6[index(z,y,x)] + D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q11[index(z,y,x)] +
//		//									D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q17[index(z,y,x)]));
//		//					break;
//		//				}//switch(obstacles[index(z,y,x)])
//
//
//
//
//		u_squ = (FLOATING)  u_x*u_x + u_y*u_y + u_z*u_z;
//		temp_factor= 0.5*(2.0* c_squ - u_squ)/c_squ;
//		//u_squ = (FLOATING)  pow(u_x,2) + pow(u_y,2) + pow(u_z,2);
//
//
//		/*...........n- velocity compnents (n = lattice node connection vectors)
//					c...........this is only necessary for clearence, and only 3 speeds would
//					c...........be necessary*/
//
//
//		//WARNING!!!! o pinakas autos exei tropopoihmena indices!!!!
//		u_n[0]= 0.0; //SHOULD NEVER USED!
//		u_n[1] =   u_x;
//		u_n[2] =         u_y;
//		u_n[3] = - u_x;
//		u_n[4] =       - u_y;
//		u_n[5] =   u_z;
//		u_n[6] =       - u_z;
//		u_n[7] =   u_x + u_y;
//		u_n[8] = - u_x + u_y;
//		u_n[9] = - u_x - u_y;
//		u_n[10] =   u_x - u_y;
//		u_n[11] =   u_x - u_z;
//		u_n[12] = - u_x - u_z;
//		u_n[13] = - u_x + u_z;
//		u_n[14] =   u_x + u_z;
//		u_n[15] =   u_z + u_y;
//		u_n[16] = - u_z + u_y;
//		u_n[17] = - u_z - u_y;
//		u_n[18] =   u_z - u_y;
//
//#pragma unroll
//		for(int i=0; i<DENSITIES; ++i){
//			u_n__over__c_squ[i]=reciprocal_c_squ*u_n[i];
//			u_n__over__c_squ__squared_and_halved[i]=0.5*u_n__over__c_squ[i]*u_n__over__c_squ[i];
//		}
//
//		/*c...........equilibrium densities
//					c...........this can be rewritten to improve computational performance
//					c...........considerabely !
//					c
//					c...........zero velocity density
//					c*/
//		//memory optimised implementation! WARNING!!! different from the original case!
//
//		//two_x_c_squ_sqared=2.0*c_squ*c_squ;
//		omega_x_rho_x_t_0=omega_x_t_0*rho;
//		omega_x_rho_x_t_1=omega_x_t_1*rho;
//		omega_x_rho_x_t_2=omega_x_t_2*rho;
//
//
//		//				//...........relaxation step
//
//
//		//omega_x_rho_x_t_0*(1.0 - 0.5*u_squ/c_squ);
//		Q0[index(z,y,x)]=buff[0]*one_minus_omega+omega_x_rho_x_t_0*(u_n__over__c_squ__squared_and_halved[0]+u_n__over__c_squ[0]+temp_factor);
//
//
//
//
//		Q1[index(z,y,x)]=buff[1]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[1]+u_n__over__c_squ[1]+temp_factor);
//		Q2[index(z,y,x)]=buff[2]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[2]+u_n__over__c_squ[2]+temp_factor);
//		Q3[index(z,y,x)]=buff[3]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[3]+u_n__over__c_squ[3]+temp_factor);
//		Q4[index(z,y,x)]=buff[4]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[4]+u_n__over__c_squ[4]+temp_factor);
//		Q5[index(z,y,x)]=buff[5]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[5]+u_n__over__c_squ[5]+temp_factor);
//		Q6[index(z,y,x)]=buff[6]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[6]+u_n__over__c_squ[6]+temp_factor);
//
//
//
//		Q7[index(z,y,x)]= buff[ 7]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 7]+u_n__over__c_squ[ 7]+temp_factor);
//		Q8[index(z,y,x)]= buff[ 8]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 8]+u_n__over__c_squ[ 8]+temp_factor);
//		Q9[index(z,y,x)]= buff[ 9]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 9]+u_n__over__c_squ[ 9]+temp_factor);
//		Q10[index(z,y,x)]=buff[10]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[10]+u_n__over__c_squ[10]+temp_factor);
//		Q11[index(z,y,x)]=buff[11]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[11]+u_n__over__c_squ[11]+temp_factor);
//		Q12[index(z,y,x)]=buff[12]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[12]+u_n__over__c_squ[12]+temp_factor);
//		Q13[index(z,y,x)]=buff[13]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[13]+u_n__over__c_squ[13]+temp_factor);
//		Q14[index(z,y,x)]=buff[14]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[14]+u_n__over__c_squ[14]+temp_factor);
//		Q15[index(z,y,x)]=buff[15]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[15]+u_n__over__c_squ[15]+temp_factor);
//		Q16[index(z,y,x)]=buff[16]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[16]+u_n__over__c_squ[16]+temp_factor);
//		Q17[index(z,y,x)]=buff[17]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[17]+u_n__over__c_squ[17]+temp_factor);
//		Q18[index(z,y,x)]=buff[18]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[18]+u_n__over__c_squ[18]+temp_factor);
//
//		//original implementation
//		//				n_equ[0] = t_0  * rho*(1.0 - u_squ / (2.0 * c_squ));
//		//
//		//								//...........axis speeds (factor: t_1)
//		//				#pragma unroll
//		//								for (int i = 1 ; i< 7; ++i){
//		//									n_equ[i] = t_1 * rho*(1.0 + u_n[i] / c_squ
//		//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
//		//											- u_squ / (2.0  * c_squ));
//		//								}
//		//
//		//								//...........diagonal speeds (factor: t_2)
//		//				#pragma unroll
//		//								for (int i = 7 ; i< 19; ++i){
//		//									n_equ[i] = t_2  * rho*(1.0 + u_n[i] / c_squ
//		//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
//		//											- u_squ / (2.0  * c_squ));
//		//								}
//
//		//				D3.Q0[index(z,y,x)]=D3_hlp.Q0[index(z,y,x)]+omega*(n_equ[0] - D3_hlp.Q0[index(z,y,x)]);
//		//								D3.Q1[index(z,y,x)]=D3_hlp.Q1[index(z,y,x)]+omega*(n_equ[1] - D3_hlp.Q1[index(z,y,x)]);
//		//								D3.Q2[index(z,y,x)]=D3_hlp.Q2[index(z,y,x)]+omega*(n_equ[2] - D3_hlp.Q2[index(z,y,x)]);
//		//								D3.Q3[index(z,y,x)]=D3_hlp.Q3[index(z,y,x)]+omega*(n_equ[3] - D3_hlp.Q3[index(z,y,x)]);
//		//								D3.Q4[index(z,y,x)]=D3_hlp.Q4[index(z,y,x)]+omega*(n_equ[4] - D3_hlp.Q4[index(z,y,x)]);
//		//								D3.Q5[index(z,y,x)]=D3_hlp.Q5[index(z,y,x)]+omega*(n_equ[5] - D3_hlp.Q5[index(z,y,x)]);
//		//								D3.Q6[index(z,y,x)]=D3_hlp.Q6[index(z,y,x)]+omega*(n_equ[6] - D3_hlp.Q6[index(z,y,x)]);
//		//								D3.Q7[index(z,y,x)]=D3_hlp.Q7[index(z,y,x)]+omega*(n_equ[7] - D3_hlp.Q7[index(z,y,x)]);
//		//								D3.Q8[index(z,y,x)]=D3_hlp.Q8[index(z,y,x)]+omega*(n_equ[8] - D3_hlp.Q8[index(z,y,x)]);
//		//								D3.Q9[index(z,y,x)]=D3_hlp.Q9[index(z,y,x)]+omega*(n_equ[9] - D3_hlp.Q9[index(z,y,x)]);
//		//								D3.Q10[index(z,y,x)]=D3_hlp.Q10[index(z,y,x)]+omega*(n_equ[10] - D3_hlp.Q10[index(z,y,x)]);
//		//								D3.Q11[index(z,y,x)]=D3_hlp.Q11[index(z,y,x)]+omega*(n_equ[11] - D3_hlp.Q11[index(z,y,x)]);
//		//								D3.Q12[index(z,y,x)]=D3_hlp.Q12[index(z,y,x)]+omega*(n_equ[12] - D3_hlp.Q12[index(z,y,x)]);
//		//								D3.Q13[index(z,y,x)]=D3_hlp.Q13[index(z,y,x)]+omega*(n_equ[13] - D3_hlp.Q13[index(z,y,x)]);
//		//								D3.Q14[index(z,y,x)]=D3_hlp.Q14[index(z,y,x)]+omega*(n_equ[14] - D3_hlp.Q14[index(z,y,x)]);
//		//								D3.Q15[index(z,y,x)]=D3_hlp.Q15[index(z,y,x)]+omega*(n_equ[15] - D3_hlp.Q15[index(z,y,x)]);
//		//								D3.Q16[index(z,y,x)]=D3_hlp.Q16[index(z,y,x)]+omega*(n_equ[16] - D3_hlp.Q16[index(z,y,x)]);
//		//								D3.Q17[index(z,y,x)]=D3_hlp.Q17[index(z,y,x)]+omega*(n_equ[17] - D3_hlp.Q17[index(z,y,x)]);
//		//								D3.Q18[index(z,y,x)]=D3_hlp.Q18[index(z,y,x)]+omega*(n_equ[18] - D3_hlp.Q18[index(z,y,x)]);
//
//		if (x == lx-2) {
//			u_previous_spatial_boundary[index2D(z,y)] = u_x;
//			v_previous_spatial_boundary[index2D(z,y)] = u_y;
//			w_previous_spatial_boundary[index2D(z,y)] = u_z;
//
//			u_current[index2D(z,y)] = u_x;
//			v_current[index2D(z,y)] = u_y;
//			w_current[index2D(z,y)] = u_z;
//
//		}//if (x == lx-2)
//	}//if memory!
//	//			}//for (x = 0 ; x< lx; ++x)
//	//		}//for (y = 0 ; y< ly ; ++y)
//	//	}//for (z = 0 ; z< lz ; ++z)
//
//}
//
//__global__
//void relaxation_kernel_v3(int lx, int ly, int lz, FLOATING reynolds, FLOATING nu, FLOATING r_small,
//		FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ, FLOATING omega, FLOATING one_minus_omega,
//		FLOATING reciprocal_c_squ,
//		FLOATING *hlp_Q0, FLOATING *hlp_Q1, FLOATING *hlp_Q2, FLOATING *hlp_Q3,
//		FLOATING *hlp_Q4, FLOATING *hlp_Q5, FLOATING *hlp_Q6, FLOATING *hlp_Q7,
//		FLOATING *hlp_Q8, FLOATING *hlp_Q9, FLOATING *hlp_Q10, FLOATING *hlp_Q11,
//		FLOATING *hlp_Q12, FLOATING *hlp_Q13, FLOATING *hlp_Q14, FLOATING *hlp_Q15,
//		FLOATING *hlp_Q16, FLOATING *hlp_Q17, FLOATING *hlp_Q18,
//		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
//		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
//		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
//		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
//		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18,
//		int *obstacles,
//		FLOATING *u_previous_spatial_boundary, FLOATING *v_previous_spatial_boundary, FLOATING *w_previous_spatial_boundary,
//		FLOATING *u_current, FLOATING *v_current, FLOATING *w_current){
//
//
//
//
//
//	/*One-step density relaxation process
//
//				c.......density relaxation: a single time relaxation with relaxation
//				c       parameter omega is applied here. This step is only "local",
//				c       nothing is propagated through the lattice.
//				c*/
//
//	int  x,y,z;
//	FLOATING  u_x=0.0, u_y=0.0, u_z=0.0, u_squ=0.0, rho=0.0, reciprocal_rho=0.0;
//
//
//
//
//	FLOATING u_n[19];
//	//	FLOATING n_equ[19];
//	FLOATING buff[19];
//
//	//	FLOATING u_n_squared[19];
//	//FLOATING two_x_c_squ_sqared;
//	const FLOATING omega_x_t_0=omega*t_0, omega_x_t_1=omega*t_1, omega_x_t_2=omega*t_2;
//	FLOATING omega_x_rho_x_t_0, omega_x_rho_x_t_1, omega_x_rho_x_t_2;
//	FLOATING temp_factor;
//	FLOATING u_n__over__c_squ[19];
//	FLOATING u_n__over__c_squ__squared_and_halved[19];
//
//	//....square speed of sound
//	/*      compute the out let velocity with a convevtive boundary condition
//					c.....loop over all nodes
//					c.....attention: actual densities are stored after the propagation
//					c                step in the help-array n_hlp !*/
//
//	//
//	//#pragma unroll
//	//	for (z = 0 ; z< lz ; ++z){
//	//#pragma unroll
//	//		for (y = 0 ; y< ly ; ++y){
//	//#pragma unroll
//	//			for (x = 0 ; x< lx; ++x){
//
//	const int tid=blockIdx.x*blockDim.x+threadIdx.x;
//	int rest;
//	int end_of_memory=lz*ly*(lx);
//
//	z=(int) (tid/(ly*lx));
//	rest=tid-z;
//	y=(int)(rest/lx);
//	x=rest-y;
//
//	if (tid<end_of_memory){
//		/*c.........only free nodes are considered here
//					!if (.not. obstacles[z][y][x]) then
//					c...........integral local density
//					c...........initialize variable ro*/
//		//memory optimised implementation
//		buff[0]=hlp_Q0[tid];
//		buff[1]=hlp_Q1[tid];
//		buff[2]=hlp_Q2[tid];
//		buff[3]=hlp_Q3[tid];
//		buff[4]=hlp_Q4[tid];
//		buff[5]=hlp_Q5[tid];
//		buff[6]=hlp_Q6[tid];
//		buff[7]=hlp_Q7[tid];
//		buff[8]=hlp_Q8[tid];
//		buff[9]=hlp_Q9[tid];
//		buff[10]=hlp_Q10[tid];
//		buff[11]=hlp_Q11[tid];
//		buff[12]=hlp_Q12[tid];
//		buff[13]=hlp_Q13[tid];
//		buff[14]=hlp_Q14[tid];
//		buff[15]=hlp_Q15[tid];
//		buff[16]=hlp_Q16[tid];
//		buff[17]=hlp_Q17[tid];
//		buff[18]=hlp_Q18[tid];
//
//		rho=0.0;
//		for(int k=0; k<DENSITIES; ++k)
//			rho+=buff[k];
//
//		//	rho=accumulate(buff, buff+DENSITIES, 0.0);
//
//		reciprocal_rho=1.0/rho;
//
//
//
//
//		switch(obstacles[tid]){
//		case 1:
//			u_x = 0.0;
//			u_y = 0.0;
//			u_z = 0.0;
//			break;
//		default:
//			u_x = 0.0;
//			u_x =  reciprocal_rho*(buff[1] + buff[7] + buff[10] +buff[11] + buff[14]-
//					(buff[3] + buff[8] + buff[9] +buff[12] + buff[13]));
//
//			u_y = 0.0;
//			u_y =  reciprocal_rho*(buff[2]+buff[8]+buff[7]+buff[16] + buff[15] -
//					(buff[4] + buff[9] + buff[10] +buff[17] + buff[18]));
//
//			u_z = 0.0;
//			u_z =  reciprocal_rho*(buff[5]+buff[13]+buff[14]+buff[15]+buff[18]-
//					(buff[6]+buff[12]+buff[11]+buff[16]+buff[17]));
//			break;
//		}//switch(obstacles[index(z,y,x)])
//
//		//original implementation
//		//				rho=0.0;
//		//				rho+=D3_hlp.Q0[index(z,y,x)]+D3_hlp.Q1[index(z,y,x)]+D3_hlp.Q2[index(z,y,x)]+D3_hlp.Q3[index(z,y,x)];
//		//				rho+=D3_hlp.Q4[index(z,y,x)]+D3_hlp.Q5[index(z,y,x)]+D3_hlp.Q6[index(z,y,x)]+D3_hlp.Q7[index(z,y,x)];
//		//				rho+=D3_hlp.Q8[index(z,y,x)]+D3_hlp.Q9[index(z,y,x)]+D3_hlp.Q10[index(z,y,x)]+D3_hlp.Q11[index(z,y,x)];
//		//				rho+=D3_hlp.Q12[index(z,y,x)]+D3_hlp.Q13[index(z,y,x)]+D3_hlp.Q14[index(z,y,x)]+D3_hlp.Q15[index(z,y,x)];
//		//				rho+=D3_hlp.Q16[index(z,y,x)]+D3_hlp.Q17[index(z,y,x)]+D3_hlp.Q18[index(z,y,x)];
//		//				reciprocal_rho=1.0/rho;
//
//		//...........x-, and y- velocity components
//
//		//				switch(obstacles[index(z,y,x)]){
//		//				case 1:
//		//					u_x = 0.0;
//		//					u_y = 0.0;
//		//					u_z = 0.0;
//		//					break;
//		//				default:
//		//					u_x = (FLOATING) reciprocal_rho*(D3_hlp.Q1[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
//		//							D3_hlp.Q11[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] -
//		//							(D3_hlp.Q3[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] +
//		//									D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)]));
//		//
//		//					u_y = (FLOATING) reciprocal_rho*(D3_hlp.Q2[index(z,y,x)] + D3_hlp.Q8[index(z,y,x)] + D3_hlp.Q7[index(z,y,x)] +
//		//							D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q15[index(z,y,x)] -
//		//							(D3_hlp.Q4[index(z,y,x)] + D3_hlp.Q9[index(z,y,x)] + D3_hlp.Q10[index(z,y,x)] +
//		//									D3_hlp.Q17[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)]));
//		//
//		//					u_z = (FLOATING) reciprocal_rho*(D3_hlp.Q5[index(z,y,x)] + D3_hlp.Q13[index(z,y,x)] + D3_hlp.Q14[index(z,y,x)] +
//		//							D3_hlp.Q15[index(z,y,x)] + D3_hlp.Q18[index(z,y,x)] -
//		//							(D3_hlp.Q6[index(z,y,x)] + D3_hlp.Q12[index(z,y,x)] + D3_hlp.Q11[index(z,y,x)] +
//		//									D3_hlp.Q16[index(z,y,x)] + D3_hlp.Q17[index(z,y,x)]));
//		//					break;
//		//				}//switch(obstacles[index(z,y,x)])
//
//
//
//
//		u_squ = (FLOATING)  u_x*u_x + u_y*u_y + u_z*u_z;
//		temp_factor= 0.5*(2.0* c_squ - u_squ)/c_squ;
//		//u_squ = (FLOATING)  pow(u_x,2) + pow(u_y,2) + pow(u_z,2);
//
//
//		/*...........n- velocity compnents (n = lattice node connection vectors)
//					c...........this is only necessary for clearence, and only 3 speeds would
//					c...........be necessary*/
//
//
//		//WARNING!!!! o pinakas autos exei tropopoihmena indices!!!!
//		u_n[0]= 0.0; //SHOULD NEVER USED!
//		u_n[1] =   u_x;
//		u_n[2] =         u_y;
//		u_n[3] = - u_x;
//		u_n[4] =       - u_y;
//		u_n[5] =   u_z;
//		u_n[6] =       - u_z;
//		u_n[7] =   u_x + u_y;
//		u_n[8] = - u_x + u_y;
//		u_n[9] = - u_x - u_y;
//		u_n[10] =   u_x - u_y;
//		u_n[11] =   u_x - u_z;
//		u_n[12] = - u_x - u_z;
//		u_n[13] = - u_x + u_z;
//		u_n[14] =   u_x + u_z;
//		u_n[15] =   u_z + u_y;
//		u_n[16] = - u_z + u_y;
//		u_n[17] = - u_z - u_y;
//		u_n[18] =   u_z - u_y;
//
//#pragma unroll
//		for(int i=0; i<DENSITIES; ++i){
//			u_n__over__c_squ[i]=reciprocal_c_squ*u_n[i];
//			u_n__over__c_squ__squared_and_halved[i]=0.5*u_n__over__c_squ[i]*u_n__over__c_squ[i];
//		}
//
//		/*c...........equilibrium densities
//					c...........this can be rewritten to improve computational performance
//					c...........considerabely !
//					c
//					c...........zero velocity density
//					c*/
//		//memory optimised implementation! WARNING!!! different from the original case!
//
//		//two_x_c_squ_sqared=2.0*c_squ*c_squ;
//		omega_x_rho_x_t_0=omega_x_t_0*rho;
//		omega_x_rho_x_t_1=omega_x_t_1*rho;
//		omega_x_rho_x_t_2=omega_x_t_2*rho;
//
//
//		//				//...........relaxation step
//
//
//		//omega_x_rho_x_t_0*(1.0 - 0.5*u_squ/c_squ);
//		Q0[tid]=buff[0]*one_minus_omega+omega_x_rho_x_t_0*(u_n__over__c_squ__squared_and_halved[0]+u_n__over__c_squ[0]+temp_factor);
//
//
//
//
//		Q1[tid]=buff[1]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[1]+u_n__over__c_squ[1]+temp_factor);
//		Q2[tid]=buff[2]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[2]+u_n__over__c_squ[2]+temp_factor);
//		Q3[tid]=buff[3]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[3]+u_n__over__c_squ[3]+temp_factor);
//		Q4[tid]=buff[4]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[4]+u_n__over__c_squ[4]+temp_factor);
//		Q5[tid]=buff[5]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[5]+u_n__over__c_squ[5]+temp_factor);
//		Q6[tid]=buff[6]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[6]+u_n__over__c_squ[6]+temp_factor);
//
//
//
//		Q7[tid]= buff[ 7]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 7]+u_n__over__c_squ[ 7]+temp_factor);
//		Q8[tid]= buff[ 8]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 8]+u_n__over__c_squ[ 8]+temp_factor);
//		Q9[tid]= buff[ 9]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 9]+u_n__over__c_squ[ 9]+temp_factor);
//		Q10[tid]=buff[10]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[10]+u_n__over__c_squ[10]+temp_factor);
//		Q11[tid]=buff[11]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[11]+u_n__over__c_squ[11]+temp_factor);
//		Q12[tid]=buff[12]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[12]+u_n__over__c_squ[12]+temp_factor);
//		Q13[tid]=buff[13]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[13]+u_n__over__c_squ[13]+temp_factor);
//		Q14[tid]=buff[14]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[14]+u_n__over__c_squ[14]+temp_factor);
//		Q15[tid]=buff[15]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[15]+u_n__over__c_squ[15]+temp_factor);
//		Q16[tid]=buff[16]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[16]+u_n__over__c_squ[16]+temp_factor);
//		Q17[tid]=buff[17]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[17]+u_n__over__c_squ[17]+temp_factor);
//		Q18[tid]=buff[18]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[18]+u_n__over__c_squ[18]+temp_factor);
//
//		//original implementation
//		//				n_equ[0] = t_0  * rho*(1.0 - u_squ / (2.0 * c_squ));
//		//
//		//								//...........axis speeds (factor: t_1)
//		//				#pragma unroll
//		//								for (int i = 1 ; i< 7; ++i){
//		//									n_equ[i] = t_1 * rho*(1.0 + u_n[i] / c_squ
//		//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
//		//											- u_squ / (2.0  * c_squ));
//		//								}
//		//
//		//								//...........diagonal speeds (factor: t_2)
//		//				#pragma unroll
//		//								for (int i = 7 ; i< 19; ++i){
//		//									n_equ[i] = t_2  * rho*(1.0 + u_n[i] / c_squ
//		//											+ (u_n[i]*u_n[i]) / (2.0 * c_squ *c_squ)
//		//											- u_squ / (2.0  * c_squ));
//		//								}
//
//		//				D3.Q0[index(z,y,x)]=D3_hlp.Q0[index(z,y,x)]+omega*(n_equ[0] - D3_hlp.Q0[index(z,y,x)]);
//		//								D3.Q1[index(z,y,x)]=D3_hlp.Q1[index(z,y,x)]+omega*(n_equ[1] - D3_hlp.Q1[index(z,y,x)]);
//		//								D3.Q2[index(z,y,x)]=D3_hlp.Q2[index(z,y,x)]+omega*(n_equ[2] - D3_hlp.Q2[index(z,y,x)]);
//		//								D3.Q3[index(z,y,x)]=D3_hlp.Q3[index(z,y,x)]+omega*(n_equ[3] - D3_hlp.Q3[index(z,y,x)]);
//		//								D3.Q4[index(z,y,x)]=D3_hlp.Q4[index(z,y,x)]+omega*(n_equ[4] - D3_hlp.Q4[index(z,y,x)]);
//		//								D3.Q5[index(z,y,x)]=D3_hlp.Q5[index(z,y,x)]+omega*(n_equ[5] - D3_hlp.Q5[index(z,y,x)]);
//		//								D3.Q6[index(z,y,x)]=D3_hlp.Q6[index(z,y,x)]+omega*(n_equ[6] - D3_hlp.Q6[index(z,y,x)]);
//		//								D3.Q7[index(z,y,x)]=D3_hlp.Q7[index(z,y,x)]+omega*(n_equ[7] - D3_hlp.Q7[index(z,y,x)]);
//		//								D3.Q8[index(z,y,x)]=D3_hlp.Q8[index(z,y,x)]+omega*(n_equ[8] - D3_hlp.Q8[index(z,y,x)]);
//		//								D3.Q9[index(z,y,x)]=D3_hlp.Q9[index(z,y,x)]+omega*(n_equ[9] - D3_hlp.Q9[index(z,y,x)]);
//		//								D3.Q10[index(z,y,x)]=D3_hlp.Q10[index(z,y,x)]+omega*(n_equ[10] - D3_hlp.Q10[index(z,y,x)]);
//		//								D3.Q11[index(z,y,x)]=D3_hlp.Q11[index(z,y,x)]+omega*(n_equ[11] - D3_hlp.Q11[index(z,y,x)]);
//		//								D3.Q12[index(z,y,x)]=D3_hlp.Q12[index(z,y,x)]+omega*(n_equ[12] - D3_hlp.Q12[index(z,y,x)]);
//		//								D3.Q13[index(z,y,x)]=D3_hlp.Q13[index(z,y,x)]+omega*(n_equ[13] - D3_hlp.Q13[index(z,y,x)]);
//		//								D3.Q14[index(z,y,x)]=D3_hlp.Q14[index(z,y,x)]+omega*(n_equ[14] - D3_hlp.Q14[index(z,y,x)]);
//		//								D3.Q15[index(z,y,x)]=D3_hlp.Q15[index(z,y,x)]+omega*(n_equ[15] - D3_hlp.Q15[index(z,y,x)]);
//		//								D3.Q16[index(z,y,x)]=D3_hlp.Q16[index(z,y,x)]+omega*(n_equ[16] - D3_hlp.Q16[index(z,y,x)]);
//		//								D3.Q17[index(z,y,x)]=D3_hlp.Q17[index(z,y,x)]+omega*(n_equ[17] - D3_hlp.Q17[index(z,y,x)]);
//		//								D3.Q18[index(z,y,x)]=D3_hlp.Q18[index(z,y,x)]+omega*(n_equ[18] - D3_hlp.Q18[index(z,y,x)]);
//
//		if (x == lx-2) {
//			u_previous_spatial_boundary[index2D(z,y)] = u_x;
//			v_previous_spatial_boundary[index2D(z,y)] = u_y;
//			w_previous_spatial_boundary[index2D(z,y)] = u_z;
//
//			u_current[index2D(z,y)] = u_x;
//			v_current[index2D(z,y)] = u_y;
//			w_current[index2D(z,y)] = u_z;
//
//		}//if (x == lx-2)
//	}//if memory!
//	//			}//for (x = 0 ; x< lx; ++x)
//	//		}//for (y = 0 ; y< ly ; ++y)
//	//	}//for (z = 0 ; z< lz ; ++z)
//
//}

__global__
void relaxation_kernel_v4(const int lx, const int ly, const int lz, const FLOATING reynolds, FLOATING nu, FLOATING r_small,
		FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ, FLOATING omega, FLOATING one_minus_omega,
		FLOATING reciprocal_c_squ,
		const FLOATING *hlp_Q0, const FLOATING *hlp_Q1, const FLOATING *hlp_Q2, const FLOATING *hlp_Q3,
		const FLOATING *hlp_Q4, const FLOATING *hlp_Q5, const FLOATING *hlp_Q6, const FLOATING *hlp_Q7,
		const FLOATING *hlp_Q8, const FLOATING *hlp_Q9, const FLOATING *hlp_Q10, const FLOATING *hlp_Q11,
		const FLOATING *hlp_Q12, const FLOATING *hlp_Q13, const FLOATING *hlp_Q14, const FLOATING *hlp_Q15,
		const FLOATING *hlp_Q16, const FLOATING *hlp_Q17, const FLOATING *hlp_Q18,
		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18,
		const int *obstacles,
		FLOATING *u_previous_spatial_boundary, FLOATING *v_previous_spatial_boundary, FLOATING *w_previous_spatial_boundary,
		FLOATING *u_current, FLOATING *v_current, FLOATING *w_current, FLOATING *u_current_temp){





	/*One-step density relaxation process

				c.......density relaxation: a single time relaxation with relaxation
				c       parameter omega is applied here. This step is only "local",
				c       nothing is propagated through the lattice.
				c*/

	int  x,y,z;
	FLOATING  u_x=0.0, u_y=0.0, u_z=0.0, u_squ=0.0, rho=0.0, reciprocal_rho=0.0;




	__shared__ FLOATING u_n[19];

	__shared__ FLOATING buff[19];

	//	extern __shared__ FLOATING shared_buffer[];

	const FLOATING omega_x_t_0=omega*t_0, omega_x_t_1=omega*t_1, omega_x_t_2=omega*t_2;
	FLOATING omega_x_rho_x_t_0, omega_x_rho_x_t_1, omega_x_rho_x_t_2;
	FLOATING temp_factor;
	FLOATING u_n__over__c_squ[19];
	FLOATING u_n__over__c_squ__squared_and_halved[19];

	//....square speed of sound
	/*      compute the out let velocity with a convevtive boundary condition
					c.....loop over all nodes
					c.....attention: actual densities are stored after the propagation
					c                step in the help-array n_hlp !*/



	const int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int rest;
	int end_of_memory=lz*ly*(lx);

	z=(int) (tid/(ly*lx));
	rest=tid-z;
	y=(int)(rest/lx);
	x=rest-y;

	if (tid<end_of_memory){
		/*c.........only free nodes are considered here
					!if (.not. obstacles[z][y][x]) then
					c...........integral local density
					c...........initialize variable ro*/
		//memory optimised implementation
		buff[0]=hlp_Q0[tid];
		buff[1]=hlp_Q1[tid];
		buff[2]=hlp_Q2[tid];
		buff[3]=hlp_Q3[tid];
		buff[4]=hlp_Q4[tid];
		buff[5]=hlp_Q5[tid];
		buff[6]=hlp_Q6[tid];
		buff[7]=hlp_Q7[tid];
		buff[8]=hlp_Q8[tid];
		buff[9]=hlp_Q9[tid];
		buff[10]=hlp_Q10[tid];
		buff[11]=hlp_Q11[tid];
		buff[12]=hlp_Q12[tid];
		buff[13]=hlp_Q13[tid];
		buff[14]=hlp_Q14[tid];
		buff[15]=hlp_Q15[tid];
		buff[16]=hlp_Q16[tid];
		buff[17]=hlp_Q17[tid];
		buff[18]=hlp_Q18[tid];

		rho=0.0;
#pragma unroll
		for(int k=0; k<DENSITIES; ++k)
			rho+=buff[k];



		reciprocal_rho=1.0/rho;




		u_x =  (1-obstacles[tid])*reciprocal_rho*(buff[1] + buff[7] + buff[10] +buff[11] + buff[14]-
				(buff[3] + buff[8] + buff[9] +buff[12] + buff[13]));


		u_y =   (1-obstacles[tid])*reciprocal_rho*(buff[2]+buff[8]+buff[7]+buff[16] + buff[15] -
				(buff[4] + buff[9] + buff[10] +buff[17] + buff[18]));


		u_z =   (1-obstacles[tid])*reciprocal_rho*(buff[5]+buff[13]+buff[14]+buff[15]+buff[18]-
				(buff[6]+buff[12]+buff[11]+buff[16]+buff[17]));






		u_squ = (FLOATING)  u_x*u_x + u_y*u_y + u_z*u_z;
		temp_factor= 0.5*(2.0* c_squ - u_squ)/c_squ;
		//u_squ = (FLOATING)  pow(u_x,2) + pow(u_y,2) + pow(u_z,2);


		/*...........n- velocity compnents (n = lattice node connection vectors)
					c...........this is only necessary for clearence, and only 3 speeds would
					c...........be necessary*/


		//WARNING!!!! o pinakas autos exei tropopoihmena indices!!!!
		u_n[0]= 0.0; //SHOULD NEVER USED!
		u_n[1] =   u_x;
		u_n[2] =         u_y;
		u_n[3] = - u_x;
		u_n[4] =       - u_y;
		u_n[5] =   u_z;
		u_n[6] =       - u_z;
		u_n[7] =   u_x + u_y;
		u_n[8] = - u_x + u_y;
		u_n[9] = - u_x - u_y;
		u_n[10] =   u_x - u_y;
		u_n[11] =   u_x - u_z;
		u_n[12] = - u_x - u_z;
		u_n[13] = - u_x + u_z;
		u_n[14] =   u_x + u_z;
		u_n[15] =   u_z + u_y;
		u_n[16] = - u_z + u_y;
		u_n[17] = - u_z - u_y;
		u_n[18] =   u_z - u_y;

#pragma unroll
		for(int i=0; i<DENSITIES; ++i){
			u_n__over__c_squ[i]=reciprocal_c_squ*u_n[i];
			u_n__over__c_squ__squared_and_halved[i]=0.5*u_n__over__c_squ[i]*u_n__over__c_squ[i];
		}

		/*c...........equilibrium densities
					c...........this can be rewritten to improve computational performance
					c...........considerabely !
					c
					c...........zero velocity density
					c*/
		//memory optimised implementation! WARNING!!! different from the original case!

		//two_x_c_squ_sqared=2.0*c_squ*c_squ;
		omega_x_rho_x_t_0=omega_x_t_0*rho;
		omega_x_rho_x_t_1=omega_x_t_1*rho;
		omega_x_rho_x_t_2=omega_x_t_2*rho;


		//				//...........relaxation step


		//omega_x_rho_x_t_0*(1.0 - 0.5*u_squ/c_squ);
		Q0[tid]=buff[0]*one_minus_omega+omega_x_rho_x_t_0*(u_n__over__c_squ__squared_and_halved[0]+u_n__over__c_squ[0]+temp_factor);




		Q1[tid]=buff[1]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[1]+u_n__over__c_squ[1]+temp_factor);
		Q2[tid]=buff[2]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[2]+u_n__over__c_squ[2]+temp_factor);
		Q3[tid]=buff[3]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[3]+u_n__over__c_squ[3]+temp_factor);
		Q4[tid]=buff[4]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[4]+u_n__over__c_squ[4]+temp_factor);
		Q5[tid]=buff[5]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[5]+u_n__over__c_squ[5]+temp_factor);
		Q6[tid]=buff[6]*one_minus_omega+omega_x_rho_x_t_1*(u_n__over__c_squ__squared_and_halved[6]+u_n__over__c_squ[6]+temp_factor);



		Q7[tid]= buff[ 7]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 7]+u_n__over__c_squ[ 7]+temp_factor);
		Q8[tid]= buff[ 8]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 8]+u_n__over__c_squ[ 8]+temp_factor);
		Q9[tid]= buff[ 9]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[ 9]+u_n__over__c_squ[ 9]+temp_factor);
		Q10[tid]=buff[10]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[10]+u_n__over__c_squ[10]+temp_factor);
		Q11[tid]=buff[11]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[11]+u_n__over__c_squ[11]+temp_factor);
		Q12[tid]=buff[12]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[12]+u_n__over__c_squ[12]+temp_factor);
		Q13[tid]=buff[13]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[13]+u_n__over__c_squ[13]+temp_factor);
		Q14[tid]=buff[14]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[14]+u_n__over__c_squ[14]+temp_factor);
		Q15[tid]=buff[15]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[15]+u_n__over__c_squ[15]+temp_factor);
		Q16[tid]=buff[16]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[16]+u_n__over__c_squ[16]+temp_factor);
		Q17[tid]=buff[17]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[17]+u_n__over__c_squ[17]+temp_factor);
		Q18[tid]=buff[18]*one_minus_omega+omega_x_rho_x_t_2*(u_n__over__c_squ__squared_and_halved[18]+u_n__over__c_squ[18]+temp_factor);


		//todo : improve the following if by having a kernel to collect all the necessary data
		if (x == lx-2 and obstacles[index(z,y,(lx-1))]==0) {
			u_previous_spatial_boundary[index2D(z,y)] = u_x;
			v_previous_spatial_boundary[index2D(z,y)] = u_y;
			w_previous_spatial_boundary[index2D(z,y)] = u_z;

			u_current[index2D(z,y)] = u_x;
			v_current[index2D(z,y)] = u_y;
			w_current[index2D(z,y)] = u_z;

			u_current_temp[index2D(z,y)] = u_x;


		}//if (x == lx-2)
	}//if memory!


}


void LBM::cuda_relaxation(){

	if(data_location==CPU)
		copy_data_from_host_to_device();

	dim3 threads_type2(threads_for_streaming_collision_and_relaxation,1,1);
	dim3 grid_type2(blocks_for_streaming_collision_and_relaxation,1,1);

	relaxation_kernel_v4<<<grid_type2, threads_type2>>>( lx,  ly,  lz,  reynolds,  nu,  r_small,
			t_0,  t_1,  t_2,  c_squ,  omega,  one_minus_omega,
			reciprocal_c_squ,
			D3_hlp_d.Q0, D3_hlp_d.Q1, D3_hlp_d.Q2, D3_hlp_d.Q3,
			D3_hlp_d.Q4, D3_hlp_d.Q5, D3_hlp_d.Q6, D3_hlp_d.Q7,
			D3_hlp_d.Q8, D3_hlp_d.Q9, D3_hlp_d.Q10, D3_hlp_d.Q11,
			D3_hlp_d.Q12, D3_hlp_d.Q13, D3_hlp_d.Q14, D3_hlp_d.Q15,
			D3_hlp_d.Q16, D3_hlp_d.Q17, D3_hlp_d.Q18,
			D3_d.Q0, D3_d.Q1, D3_d.Q2, D3_d.Q3,
			D3_d.Q4, D3_d.Q5, D3_d.Q6, D3_d.Q7,
			D3_d.Q8, D3_d.Q9, D3_d.Q10, D3_d.Q11,
			D3_d.Q12, D3_d.Q13, D3_d.Q14, D3_d.Q15,
			D3_d.Q16, D3_d.Q17, D3_d.Q18,
			obstacles_d,
			u_previous_spatial_boundary_d, v_previous_spatial_boundary_d, w_previous_spatial_boundary_d,
			u_current_d, v_current_d, w_current_d, u_current_temp_d);




	hipDeviceSynchronize();
}
