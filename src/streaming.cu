#include "hip/hip_runtime.h"
#include "global_defines.cuh"


void LBM::streaming(){



	/*Propagate fluid densities to their next neighbour nodes */
	/*c
	c.......density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.
	c*/

	if(data_location==GPU)
		copy_data_from_device_to_host();


	int  x,y,z;
	int x_e/*east*/,x_w/*west*/;
	int y_n/*north*/,y_s/*south*/;
	int z_l/*left*/,z_r/*right*/;





	//todo rwta ton Dirk pws to kanei auto to vhma
	for (z = 0 ; z< lz ; ++z){
		z_l =  (z+1)%lz; //1 8esh meta to trexon x
		z_r =   (z+lz-1) %(  lz) ;

		for (y = 0; y< ly; ++y){
			y_n =  (y+1)%ly; //1 8esh meta to trexon y
			y_s =   (y+ly-1) %(  ly) ;

			for (x = 0; x< lx; ++x){
				x_e =  (x+1)%lx; //1 8esh meta to trexon z
				x_w =   (x+lx-1) %(  lx) ;

				//regular streaming process

				/*
					.........density propagation

					.........zero: just copy
				 */
				D3_hlp.Q0[index(z,y,x)]=D3.Q0[index(z,y,x)];
				//node_hlp_grid[index4D(z,y,x,0)]=node_grid[index4D(z,y,x,0)];

				//.........in the x,y and z directions
				D3_hlp.Q1[index(z  ,y  ,x_e)] = D3.Q1[index(z,y,x)];
				D3_hlp.Q2[index(z  ,y_n,x  )] = D3.Q2[index(z,y,x)];
				D3_hlp.Q3[index(z  ,y  ,x_w)] = D3.Q3[index(z,y,x)];
				D3_hlp.Q4[index(z  ,y_s,x  )] = D3.Q4[index(z,y,x)];
				D3_hlp.Q5[index(z_l,y  ,x  )] = D3.Q5[index(z,y,x)];
				D3_hlp.Q6[index(z_r,y  ,x  )] = D3.Q6[index(z,y,x)];


				//......... in the x,y diagonals


				D3_hlp.Q7[ index(z  ,y_n,x_e)] = D3.Q7[ index(z,y,x)];
				D3_hlp.Q8[ index(z  ,y_n,x_w)] = D3.Q8[ index(z,y,x)];
				D3_hlp.Q9[ index(z  ,y_s,x_w)] = D3.Q9[ index(z,y,x)];
				D3_hlp.Q10[index(z  ,y_s,x_e)] = D3.Q10[index(z,y,x)];

				//......... in the x,z diagonals

				D3_hlp.Q11[index(z_r,y  ,x_e)] = D3.Q11[index(z,y,x)];
				D3_hlp.Q12[index(z_r,y  ,x_w)] = D3.Q12[index(z,y,x)];
				D3_hlp.Q13[index(z_l,y  ,x_w)] = D3.Q13[index(z,y,x)];
				D3_hlp.Q14[index(z_l,y  ,x_e)] = D3.Q14[index(z,y,x)];

				//......... in the y,z diagonals

				D3_hlp.Q15[index(z_l,y_n,x  )] = D3.Q15[index(z,y,x)];
				D3_hlp.Q16[index(z_r,y_n,x  )] = D3.Q16[index(z,y,x)];
				D3_hlp.Q17[index(z_r,y_s,x  )] = D3.Q17[index(z,y,x)];
				D3_hlp.Q18[index(z_l,y_s,x  )] = D3.Q18[index(z,y,x)];



			}//z-loop
		}//y loop
	}//x loop

	for (z = 0 ; z< lz ; ++z){
		//loop for x=0 and x=lx-1!!!! (first and last slice)
		for (y = 0; y< ly; ++y){
			//toslice 0 antigrafetai sto 0
			D3_hlp.Q0[index(z,y,0)]=D3.Q0[index(z,y,0)];
			D3_hlp.Q1[index(z,y,0)]=D3.Q1[index(z,y,0)];
			D3_hlp.Q2[index(z,y,0)]=D3.Q2[index(z,y,0)];
			D3_hlp.Q3[index(z,y,0)]=D3.Q3[index(z,y,0)];
			D3_hlp.Q4[index(z,y,0)]=D3.Q4[index(z,y,0)];
			D3_hlp.Q5[index(z,y,0)]=D3.Q5[index(z,y,0)];
			D3_hlp.Q6[index(z,y,0)]=D3.Q6[index(z,y,0)];
			D3_hlp.Q7[index(z,y,0)]=D3.Q7[index(z,y,0)];
			D3_hlp.Q8[index(z,y,0)]=D3.Q8[index(z,y,0)];
			D3_hlp.Q9[index(z,y,0)]=D3.Q9[index(z,y,0)];
			D3_hlp.Q10[index(z,y,0)]=D3.Q10[index(z,y,0)];
			D3_hlp.Q11[index(z,y,0)]=D3.Q11[index(z,y,0)];
			D3_hlp.Q12[index(z,y,0)]=D3.Q12[index(z,y,0)];
			D3_hlp.Q13[index(z,y,0)]=D3.Q13[index(z,y,0)];
			D3_hlp.Q14[index(z,y,0)]=D3.Q14[index(z,y,0)];
			D3_hlp.Q15[index(z,y,0)]=D3.Q15[index(z,y,0)];
			D3_hlp.Q16[index(z,y,0)]=D3.Q16[index(z,y,0)];
			D3_hlp.Q17[index(z,y,0)]=D3.Q17[index(z,y,0)];
			D3_hlp.Q18[index(z,y,0)]=D3.Q18[index(z,y,0)];
			//at x= lx I set the incomming density as the one (equilibrum) calculated after the collision
			D3_hlp.Q3[index(z,y,lx-1)] = D3.Q3[index(z,y,lx-1)];
			D3_hlp.Q8[index(z,y,lx-1)] = D3.Q8[index(z,y,lx-1)] ;
			D3_hlp.Q9[index(z,y,lx-1)] = D3.Q9[index(z,y,lx-1)] ;
			D3_hlp.Q12[index(z,y,lx-1)] = D3.Q12[index(z,y,lx-1)] ;
			D3_hlp.Q13[index(z,y,lx-1)] = D3.Q13[index(z,y,lx-1)] ;
			//			gia x=lx-1: the densities 0,   2,4,5,6,15,16,17,18
			//				aplws 8a metadw8oune
			//				"ka8eta" sto slice kai de 8a ginoun propagate se
			//				alla slices (tuxainei na voleuei auto)
			//				auto to kommati exei HDH ginei pio panw!
			//
			//
			//				during streaming some of the indices at
			//				(fixed) x=lx-1, y=ly-1, do not get updated.
			//				it doesn't matter since the last slice on x is not useful.
		}
	}

	/*
	for (z = 0 ; z< lz ; ++z){
		z_l =  (z+1)%lz; //1 8esh meta to trexon x
		z_r =   (z+lz-1) %(  lz) ;

		//loop for x=0 and x=lx-1!!!! (first and last slice)
		for (y = 0; y< ly; ++y){
			y_n =  (y+1)%ly; //1 8esh meta to trexon x
			y_s =   (y+ly-1) %(  ly) ;



			//toslice 0 antigrafetai sto 0


				D3_hlp.Q0[index(z,y,0)]=D3.Q0[index(z,y,0)];
				D3_hlp.Q1[index(z,y,0)]=D3.Q1[index(z,y,0)];
				D3_hlp.Q2[index(z,y,0)]=D3.Q2[index(z,y,0)];
				D3_hlp.Q3[index(z,y,0)]=D3.Q3[index(z,y,0)];
				D3_hlp.Q4[index(z,y,0)]=D3.Q4[index(z,y,0)];
				D3_hlp.Q5[index(z,y,0)]=D3.Q5[index(z,y,0)];
				D3_hlp.Q6[index(z,y,0)]=D3.Q6[index(z,y,0)];
				D3_hlp.Q7[index(z,y,0)]=D3.Q7[index(z,y,0)];
				D3_hlp.Q8[index(z,y,0)]=D3.Q8[index(z,y,0)];
				D3_hlp.Q9[index(z,y,0)]=D3.Q9[index(z,y,0)];
				D3_hlp.Q10[index(z,y,0)]=D3.Q10[index(z,y,0)];
				D3_hlp.Q11[index(z,y,0)]=D3.Q11[index(z,y,0)];
				D3_hlp.Q12[index(z,y,0)]=D3.Q12[index(z,y,0)];
				D3_hlp.Q13[index(z,y,0)]=D3.Q13[index(z,y,0)];
				D3_hlp.Q14[index(z,y,0)]=D3.Q14[index(z,y,0)];
				D3_hlp.Q15[index(z,y,0)]=D3.Q15[index(z,y,0)];
				D3_hlp.Q16[index(z,y,0)]=D3.Q16[index(z,y,0)];
				D3_hlp.Q17[index(z,y,0)]=D3.Q17[index(z,y,0)];
				D3_hlp.Q18[index(z,y,0)]=D3.Q18[index(z,y,0)];


			//at x= lx I set the incomming density as the one (equilibrum) calculated after the collision
			D3_hlp.Q3[index(z,y,lx-1)] = D3.Q3[index(z,y,lx-1)];
			D3_hlp.Q8[index(z,y,lx-1)] = D3.Q8[index(z,y,lx-1)] ;
			D3_hlp.Q9[index(z,y,lx-1)] = D3.Q9[index(z,y,lx-1)] ;
			D3_hlp.Q12[index(z,y,lx-1)] = D3.Q12[index(z,y,lx-1)] ;
			D3_hlp.Q13[index(z,y,lx-1)] = D3.Q13[index(z,y,lx-1)] ;

			//tis aristeres densities apo to slice 0  tis metaferei sto telos

			D3_hlp.Q3[ index(z  ,y  ,lx-1)] = D3.Q3[ index(z,y,0)];
			D3_hlp.Q8[ index(z  ,y_n,lx-1)] = D3.Q8[ index(z,y,0)];
			D3_hlp.Q9[ index(z  ,y_s,lx-1)] = D3.Q9[ index(z,y,0)];
			D3_hlp.Q12[index(z_r,y  ,lx-1)] = D3.Q12[index(z,y,0)];
			D3_hlp.Q13[index(z_l,y  ,lx-1)] = D3.Q13[index(z,y,0)];

			//tis deksies densities tou teleutaiou slice tis metaferei sto slice 0
			D3_hlp.Q1[ index(z  ,y  ,0)] = D3.Q1[ index(z,y,lx-1)];
			D3_hlp.Q7[ index(z  ,y_n,0)] = D3.Q7[ index(z,y,lx-1)];
			D3_hlp.Q10[index(z  ,y_s,0)] = D3.Q10[index(z,y,lx-1)];
			D3_hlp.Q11[index(z_r,y  ,0)] = D3.Q11[index(z,y,lx-1)];
			D3_hlp.Q14[index(z_l,y  ,0)] = D3.Q14[index(z,y,lx-1)];

//			gia x=lx-1: the densities 0,   2,4,5,6,15,16,17,18
//				aplws 8a metadw8oune
//				"ka8eta" sto slice kai de 8a ginoun propagate se
//				alla slices (tuxainei na voleuei auto)
//				auto to kommati exei HDH ginei pio panw!
//
//
//				during streaming some of the indices at
//				(fixed) x=lx-1, y=ly-1, do not get updated.
//				it doesn't matter since the last slice on x is not useful.

		}
	}*/
#ifdef DEBUG
	cout << " #LBM streaming OK!" << endl;
#endif
}

void LBM::streaming_first_part(){



	/*Propagate fluid densities to their next neighbour nodes */
	/*c
	c.......density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.
	c*/
	int  x,y,z;
	int x_e/*east*/,x_w/*west*/;
	int y_n/*north*/,y_s/*south*/;
	int z_l/*left*/,z_r/*right*/;
	//todo rwta ton Dirk pws to kanei auto to vhma
	for (z = 0 ; z< lz ; ++z){
		z_l =  (z+1)%lz; //1 8esh meta to trexon x
		z_r =   (z+lz-1) %(  lz) ;

		for (y = 0; y< ly; ++y){
			y_n =  (y+1)%ly; //1 8esh meta to trexon y
			y_s =   (y+ly-1) %(  ly) ;

			for (x = 0; x< lx; ++x){
				x_e =  (x+1)%lx; //1 8esh meta to trexon z
				x_w =   (x+lx-1) %(  lx) ;

				//regular streaming process

				/*
					.........density propagation

					.........zero: just copy
				 */
				D3_hlp.Q0[index(z,y,x)]=D3.Q0[index(z,y,x)];
				//				node_hlp_grid[index4D(z,y,x,0)]=node_grid[index4D(z,y,x,0)];

				//.........in the x,y and z directions
				D3_hlp.Q1[index(z  ,y  ,x_e)] = D3.Q1[index(z,y,x)];
				D3_hlp.Q2[index(z  ,y_n,x  )] = D3.Q2[index(z,y,x)];
				D3_hlp.Q3[index(z  ,y  ,x_w)] = D3.Q3[index(z,y,x)];
				D3_hlp.Q4[index(z  ,y_s,x  )] = D3.Q4[index(z,y,x)];
				D3_hlp.Q5[index(z_l,y  ,x  )] = D3.Q5[index(z,y,x)];
				D3_hlp.Q6[index(z_r,y  ,x  )] = D3.Q6[index(z,y,x)];


				//......... in the x,y diagonals


				D3_hlp.Q7[ index(z  ,y_n,x_e)] = D3.Q7[ index(z,y,x)];
				D3_hlp.Q8[ index(z  ,y_n,x_w)] = D3.Q8[ index(z,y,x)];
				D3_hlp.Q9[ index(z  ,y_s,x_w)] = D3.Q9[ index(z,y,x)];
				D3_hlp.Q10[index(z  ,y_s,x_e)] = D3.Q10[index(z,y,x)];

				//......... in the x,z diagonals

				D3_hlp.Q11[index(z_r,y  ,x_e)] = D3.Q11[index(z,y,x)];
				D3_hlp.Q12[index(z_r,y  ,x_w)] = D3.Q12[index(z,y,x)];
				D3_hlp.Q13[index(z_l,y  ,x_w)] = D3.Q13[index(z,y,x)];
				D3_hlp.Q14[index(z_l,y  ,x_e)] = D3.Q14[index(z,y,x)];

				//......... in the y,z diagonals

				D3_hlp.Q15[index(z_l,y_n,x  )] = D3.Q15[index(z,y,x)];
				D3_hlp.Q16[index(z_r,y_n,x  )] = D3.Q16[index(z,y,x)];
				D3_hlp.Q17[index(z_r,y_s,x  )] = D3.Q17[index(z,y,x)];
				D3_hlp.Q18[index(z_l,y_s,x  )] = D3.Q18[index(z,y,x)];



			}//z-loop
		}//y loop
	}//x loop


#ifdef DEBUG
	cout << " #LBM streaming OK!" << endl;
#endif
}

void LBM::streaming_last_part(){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c
	c.......density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.
	c*/

	int   y,z;

	for (z = 0 ; z< lz ; ++z){
		//loop for x=0 and x=lx-1!!!! (first and last slice)
		for (y = 0; y< ly; ++y){
			//toslice 0 antigrafetai sto 0
			D3_hlp.Q0[index(z,y,0)]=D3.Q0[index(z,y,0)];
			D3_hlp.Q1[index(z,y,0)]=D3.Q1[index(z,y,0)];
			D3_hlp.Q2[index(z,y,0)]=D3.Q2[index(z,y,0)];
			D3_hlp.Q3[index(z,y,0)]=D3.Q3[index(z,y,0)];
			D3_hlp.Q4[index(z,y,0)]=D3.Q4[index(z,y,0)];
			D3_hlp.Q5[index(z,y,0)]=D3.Q5[index(z,y,0)];
			D3_hlp.Q6[index(z,y,0)]=D3.Q6[index(z,y,0)];
			D3_hlp.Q7[index(z,y,0)]=D3.Q7[index(z,y,0)];
			D3_hlp.Q8[index(z,y,0)]=D3.Q8[index(z,y,0)];
			D3_hlp.Q9[index(z,y,0)]=D3.Q9[index(z,y,0)];
			D3_hlp.Q10[index(z,y,0)]=D3.Q10[index(z,y,0)];
			D3_hlp.Q11[index(z,y,0)]=D3.Q11[index(z,y,0)];
			D3_hlp.Q12[index(z,y,0)]=D3.Q12[index(z,y,0)];
			D3_hlp.Q13[index(z,y,0)]=D3.Q13[index(z,y,0)];
			D3_hlp.Q14[index(z,y,0)]=D3.Q14[index(z,y,0)];
			D3_hlp.Q15[index(z,y,0)]=D3.Q15[index(z,y,0)];
			D3_hlp.Q16[index(z,y,0)]=D3.Q16[index(z,y,0)];
			D3_hlp.Q17[index(z,y,0)]=D3.Q17[index(z,y,0)];
			D3_hlp.Q18[index(z,y,0)]=D3.Q18[index(z,y,0)];
			//at x= lx I set the incomming density as the one (equilibrum) calculated after the collision
			D3_hlp.Q3[index(z,y,lx-1)] = D3.Q3[index(z,y,lx-1)];
			D3_hlp.Q8[index(z,y,lx-1)] = D3.Q8[index(z,y,lx-1)] ;
			D3_hlp.Q9[index(z,y,lx-1)] = D3.Q9[index(z,y,lx-1)] ;
			D3_hlp.Q12[index(z,y,lx-1)] = D3.Q12[index(z,y,lx-1)] ;
			D3_hlp.Q13[index(z,y,lx-1)] = D3.Q13[index(z,y,lx-1)] ;
			//			gia x=lx-1: the densities 0,   2,4,5,6,15,16,17,18
			//				aplws 8a metadw8oune
			//				"ka8eta" sto slice kai de 8a ginoun propagate se
			//				alla slices (tuxainei na voleuei auto)
			//				auto to kommati exei HDH ginei pio panw!
			//
			//
			//				during streaming some of the indices at
			//				(fixed) x=lx-1, y=ly-1, do not get updated.
			//				it doesn't matter since the last slice on x is not useful.
		}
	}
}




__global__
void streaming_kernel(int lx, int ly, int lz, FLOATING reynolds, FLOATING nu, FLOATING r_small, FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ, lattice D3, lattice D3_hlp){



	/*Propagate fluid densities to their next neighbour nodes */
	/*c
	c.......density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.
	c*/




	int  x,y,z;
	int x_e/*east*/,x_w/*west*/;
	int y_n/*north*/,y_s/*south*/;
	int z_l/*left*/,z_r/*right*/;


	int rest;
	int end_of_memory=lz*ly*(lx);
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	z=(int) (tid/(ly*lx));
	rest=tid-z;
	y=(int)(rest/lx);
	x=rest-y;

	//if(tid<end_of_memory){
	if( z<lz and y<ly and x<lx){

		//todo rwta ton Dirk pws to kanei auto to vhma
		//		for (z = 0 ; z< lz ; ++z){
		z_l =  (z+1)%lz; //1 8esh meta to trexon x
		z_r =   (z+lz-1) %(  lz) ;

		//			for (y = 0; y< ly; ++y){
		y_n =  (y+1)%ly; //1 8esh meta to trexon y
		y_s =   (y+ly-1) %(  ly) ;

		//				for (x = 0; x< lx; ++x){
		x_e =  (x+1)%lx; //1 8esh meta to trexon z
		x_w =   (x+lx-1) %(  lx) ;

		//regular streaming process

		/*
					.........density propagation

					.........zero: just copy
		 */
		D3_hlp.Q0[index(z,y,x)]=D3.Q0[index(z,y,x)];
		//node_hlp_grid[index4D(z,y,x,0)]=node_grid[index4D(z,y,x,0)];

		//.........in the x,y and z directions
		D3_hlp.Q1[index(z  ,y  ,x_e)] = D3.Q1[index(z,y,x)];
		D3_hlp.Q2[index(z  ,y_n,x  )] = D3.Q2[index(z,y,x)];
		D3_hlp.Q3[index(z  ,y  ,x_w)] = D3.Q3[index(z,y,x)];
		D3_hlp.Q4[index(z  ,y_s,x  )] = D3.Q4[index(z,y,x)];
		D3_hlp.Q5[index(z_l,y  ,x  )] = D3.Q5[index(z,y,x)];
		D3_hlp.Q6[index(z_r,y  ,x  )] = D3.Q6[index(z,y,x)];


		//......... in the x,y diagonals


		D3_hlp.Q7[ index(z  ,y_n,x_e)] = D3.Q7[ index(z,y,x)];
		D3_hlp.Q8[ index(z  ,y_n,x_w)] = D3.Q8[ index(z,y,x)];
		D3_hlp.Q9[ index(z  ,y_s,x_w)] = D3.Q9[ index(z,y,x)];
		D3_hlp.Q10[index(z  ,y_s,x_e)] = D3.Q10[index(z,y,x)];

		//......... in the x,z diagonals

		D3_hlp.Q11[index(z_r,y  ,x_e)] = D3.Q11[index(z,y,x)];
		D3_hlp.Q12[index(z_r,y  ,x_w)] = D3.Q12[index(z,y,x)];
		D3_hlp.Q13[index(z_l,y  ,x_w)] = D3.Q13[index(z,y,x)];
		D3_hlp.Q14[index(z_l,y  ,x_e)] = D3.Q14[index(z,y,x)];

		//......... in the y,z diagonals

		D3_hlp.Q15[index(z_l,y_n,x  )] = D3.Q15[index(z,y,x)];
		D3_hlp.Q16[index(z_r,y_n,x  )] = D3.Q16[index(z,y,x)];
		D3_hlp.Q17[index(z_r,y_s,x  )] = D3.Q17[index(z,y,x)];
		D3_hlp.Q18[index(z_l,y_s,x  )] = D3.Q18[index(z,y,x)];



		//				}//z-loop
		//			}//y loop
		//		}//x loop
	}

	if(tid<end_of_memory and x==0 ){
		//		for (z = 0 ; z< lz ; ++z){
		//
		//
		//			//loop for x=0 and x=lx-1!!!! (first and last slice)
		//			for (y = 0; y< ly; ++y){
		//toslice 0 antigrafetai sto 0
		D3_hlp.Q0[index(z,y,0)]=D3.Q0[index(z,y,0)];
		D3_hlp.Q1[index(z,y,0)]=D3.Q1[index(z,y,0)];
		D3_hlp.Q2[index(z,y,0)]=D3.Q2[index(z,y,0)];
		D3_hlp.Q3[index(z,y,0)]=D3.Q3[index(z,y,0)];
		D3_hlp.Q4[index(z,y,0)]=D3.Q4[index(z,y,0)];
		D3_hlp.Q5[index(z,y,0)]=D3.Q5[index(z,y,0)];
		D3_hlp.Q6[index(z,y,0)]=D3.Q6[index(z,y,0)];
		D3_hlp.Q7[index(z,y,0)]=D3.Q7[index(z,y,0)];
		D3_hlp.Q8[index(z,y,0)]=D3.Q8[index(z,y,0)];
		D3_hlp.Q9[index(z,y,0)]=D3.Q9[index(z,y,0)];
		D3_hlp.Q10[index(z,y,0)]=D3.Q10[index(z,y,0)];
		D3_hlp.Q11[index(z,y,0)]=D3.Q11[index(z,y,0)];
		D3_hlp.Q12[index(z,y,0)]=D3.Q12[index(z,y,0)];
		D3_hlp.Q13[index(z,y,0)]=D3.Q13[index(z,y,0)];
		D3_hlp.Q14[index(z,y,0)]=D3.Q14[index(z,y,0)];
		D3_hlp.Q15[index(z,y,0)]=D3.Q15[index(z,y,0)];
		D3_hlp.Q16[index(z,y,0)]=D3.Q16[index(z,y,0)];
		D3_hlp.Q17[index(z,y,0)]=D3.Q17[index(z,y,0)];
		D3_hlp.Q18[index(z,y,0)]=D3.Q18[index(z,y,0)];



		//			gia x=lx-1: the densities 0,   2,4,5,6,15,16,17,18
		//				aplws 8a metadw8oune
		//				"ka8eta" sto slice kai de 8a ginoun propagate se
		//				alla slices (tuxainei na voleuei auto)
		//				auto to kommati exei HDH ginei pio panw!
		//
		//
		//				during streaming some of the indices at
		//				(fixed) x=lx-1, y=ly-1, do not get updated.
		//				it doesn't matter since the last slice on x is not useful.

		//			}
		//		}
	}
	if(tid<end_of_memory and   x==lx-1  ){
		//at x= lx I set the incomming density as the one (equilibrum) calculated after the collision
		D3_hlp.Q3[index(z,y,lx-1)] = D3.Q3[index(z,y,lx-1)];
		D3_hlp.Q8[index(z,y,lx-1)] = D3.Q8[index(z,y,lx-1)] ;
		D3_hlp.Q9[index(z,y,lx-1)] = D3.Q9[index(z,y,lx-1)] ;
		D3_hlp.Q12[index(z,y,lx-1)] = D3.Q12[index(z,y,lx-1)] ;
		D3_hlp.Q13[index(z,y,lx-1)] = D3.Q13[index(z,y,lx-1)] ;
	}


#ifdef DEBUG
	cout << " #LBM streaming OK!" << endl;
#endif
}






__global__
void streaming_kernel_single_threaded_p1(int lx, int ly, int lz,
		FLOATING *hlp_Q0, FLOATING *hlp_Q1, FLOATING *hlp_Q2, FLOATING *hlp_Q3,
		FLOATING *hlp_Q4, FLOATING *hlp_Q5, FLOATING *hlp_Q6, FLOATING *hlp_Q7,
		FLOATING *hlp_Q8, FLOATING *hlp_Q9, FLOATING *hlp_Q10, FLOATING *hlp_Q11,
		FLOATING *hlp_Q12, FLOATING *hlp_Q13, FLOATING *hlp_Q14, FLOATING *hlp_Q15,
		FLOATING *hlp_Q16, FLOATING *hlp_Q17, FLOATING *hlp_Q18,
		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18){


	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	//	int x,y,z;
	int x_e/*east*/,x_w/*west*/;
	int y_n/*north*/,y_s/*south*/;
	int z_l/*left*/,z_r/*right*/;

	//	__shared__ FLOATING shared_buffer[64];


	if(tid<end_of_memory){

		z_l =  (z+1)%lz; //1 8esh meta to trexon x
		z_r =   (z+lz-1) %(  lz) ;

		//				for (y = 0; y< ly; ++y){
		y_n =  (y+1)%ly; //1 8esh meta to trexon y
		y_s =   (y+ly-1) %(  ly) ;

		//					for (x = 0; x< lx; ++x){
		x_e =  (x+1)%lx; //1 8esh meta to trexon z
		x_w =   (x+lx-1) %(  lx) ;

		//regular streaming process

		//					.........density propagation
		//					.........zero: just copy
		hlp_Q0[index(z,y,x)]=Q0[index(z,y,x)];
		//				node_hlp_grid[index4D(z,y,x,0)]=node_grid[index4D(z,y,x,0)];

		//.........in the x,y and z directions
		hlp_Q1[index(z  ,y  ,x_e)] = Q1[index(z,y,x)];
		hlp_Q2[index(z  ,y_n,x  )] = Q2[index(z,y,x)];
		hlp_Q3[index(z  ,y  ,x_w)] = Q3[index(z,y,x)];
		hlp_Q4[index(z  ,y_s,x  )] = Q4[index(z,y,x)];
		hlp_Q5[index(z_l,y  ,x  )] = Q5[index(z,y,x)];
		hlp_Q6[index(z_r,y  ,x  )] = Q6[index(z,y,x)];

		//......... in the x,y diagonals
		hlp_Q7[ index(z  ,y_n,x_e)] = Q7[ index(z,y,x)];
		hlp_Q8[ index(z  ,y_n,x_w)] = Q8[ index(z,y,x)];
		hlp_Q9[ index(z  ,y_s,x_w)] = Q9[ index(z,y,x)];
		hlp_Q10[index(z  ,y_s,x_e)] = Q10[index(z,y,x)];

		//......... in the x,z diagonals
		hlp_Q11[index(z_r,y  ,x_e)] = Q11[index(z,y,x)];
		hlp_Q12[index(z_r,y  ,x_w)] = Q12[index(z,y,x)];
		hlp_Q13[index(z_l,y  ,x_w)] = Q13[index(z,y,x)];
		hlp_Q14[index(z_l,y  ,x_e)] = Q14[index(z,y,x)];

		//......... in the y,z diagonals
		hlp_Q15[index(z_l,y_n,x  )] = Q15[index(z,y,x)];
		hlp_Q16[index(z_r,y_n,x  )] = Q16[index(z,y,x)];
		hlp_Q17[index(z_r,y_s,x  )] = Q17[index(z,y,x)];
		hlp_Q18[index(z_l,y_s,x  )] = Q18[index(z,y,x)];
	}
}

__global__
void streaming_kernel_single_threaded_p1_shared(int lx, int ly, int lz,
		FLOATING *hlp_Q0, FLOATING *hlp_Q1, FLOATING *hlp_Q2, FLOATING *hlp_Q3,
		FLOATING *hlp_Q4, FLOATING *hlp_Q5, FLOATING *hlp_Q6, FLOATING *hlp_Q7,
		FLOATING *hlp_Q8, FLOATING *hlp_Q9, FLOATING *hlp_Q10, FLOATING *hlp_Q11,
		FLOATING *hlp_Q12, FLOATING *hlp_Q13, FLOATING *hlp_Q14, FLOATING *hlp_Q15,
		FLOATING *hlp_Q16, FLOATING *hlp_Q17, FLOATING *hlp_Q18,
		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18){


	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	//	int x,y,z;
	int x_e/*east*/,x_w/*west*/;
	int y_n/*north*/,y_s/*south*/;
	int z_l/*left*/,z_r/*right*/;

	extern __shared__ FLOATING shared_buffer[];
	FLOATING *shared_Q0=shared_buffer;
	FLOATING *shared_Q1 = &shared_Q0[blockDim.x];
	FLOATING *shared_Q2 = &shared_Q1[blockDim.x];
	FLOATING *shared_Q3 = &shared_Q2[blockDim.x];
	FLOATING *shared_Q4 = &shared_Q3[blockDim.x];
	FLOATING *shared_Q5 = &shared_Q4[blockDim.x];
	FLOATING *shared_Q6 = &shared_Q5[blockDim.x];
	FLOATING *shared_Q7 = &shared_Q6[blockDim.x];
	FLOATING *shared_Q8 = &shared_Q7[blockDim.x];
	FLOATING *shared_Q9 = &shared_Q8[blockDim.x];
	FLOATING *shared_Q10 = &shared_Q9[blockDim.x];
	FLOATING *shared_Q11 = &shared_Q10[blockDim.x];
	FLOATING *shared_Q12 = &shared_Q11[blockDim.x];
	FLOATING *shared_Q13 = &shared_Q12[blockDim.x];
	FLOATING *shared_Q14 = &shared_Q13[blockDim.x];
	FLOATING *shared_Q15 = &shared_Q14[blockDim.x];
	FLOATING *shared_Q16 = &shared_Q15[blockDim.x];
	FLOATING *shared_Q17 = &shared_Q16[blockDim.x];
	FLOATING *shared_Q18 = &shared_Q17[blockDim.x];



	shared_Q0[threadIdx.x]=Q0[index(z,y,x)];
	shared_Q1[threadIdx.x]=Q1[index(z,y,x)];
	shared_Q2[threadIdx.x]=Q2[index(z,y,x)];
	shared_Q3[threadIdx.x]=Q3[index(z,y,x)];
	shared_Q4[threadIdx.x]=Q4[index(z,y,x)];
	shared_Q5[threadIdx.x]=Q5[index(z,y,x)];
	shared_Q6[threadIdx.x]=Q6[index(z,y,x)];
	shared_Q7[threadIdx.x]=Q7[index(z,y,x)];
	shared_Q8[threadIdx.x]=Q8[index(z,y,x)];
	shared_Q9[threadIdx.x]=Q9[index(z,y,x)];
	shared_Q10[threadIdx.x]=Q10[index(z,y,x)];
	shared_Q11[threadIdx.x]=Q11[index(z,y,x)];
	shared_Q12[threadIdx.x]=Q12[index(z,y,x)];
	shared_Q13[threadIdx.x]=Q13[index(z,y,x)];
	shared_Q14[threadIdx.x]=Q14[index(z,y,x)];
	shared_Q15[threadIdx.x]=Q15[index(z,y,x)];
	shared_Q16[threadIdx.x]=Q16[index(z,y,x)];
	shared_Q17[threadIdx.x]=Q17[index(z,y,x)];
	shared_Q18[threadIdx.x]=Q18[index(z,y,x)];
	__syncthreads();
	if(tid<end_of_memory){

		z_l =  (z+1)%lz; //1 8esh meta to trexon x
		z_r =   (z+lz-1) %(  lz) ;

		//				for (y = 0; y< ly; ++y){
		y_n =  (y+1)%ly; //1 8esh meta to trexon y
		y_s =   (y+ly-1) %(  ly) ;

		//					for (x = 0; x< lx; ++x){
		x_e =  (x+1)%lx; //1 8esh meta to trexon z
		x_w =   (x+lx-1) %(  lx) ;

		//regular streaming process

		//					.........density propagation
		//					.........zero: just copy
		hlp_Q0[index(z,y,x)]=shared_Q0[threadIdx.x];
		//				node_hlp_grid[index4D(z,y,x,0)]=node_grid[index4D(z,y,x,0)];

		//.........in the x,y and z directions
		hlp_Q1[index(z  ,y  ,x_e)] =shared_Q1[threadIdx.x];
		hlp_Q2[index(z  ,y_n,x  )] =shared_Q2[threadIdx.x];
		hlp_Q3[index(z  ,y  ,x_w)] =shared_Q3[threadIdx.x];
		hlp_Q4[index(z  ,y_s,x  )] =shared_Q4[threadIdx.x];
		hlp_Q5[index(z_l,y  ,x  )] =shared_Q5[threadIdx.x];
		hlp_Q6[index(z_r,y  ,x  )] =shared_Q6[threadIdx.x];

		//......... in the x,y diagonals
		hlp_Q7[ index(z  ,y_n,x_e)] =shared_Q7[ threadIdx.x];
		hlp_Q8[ index(z  ,y_n,x_w)] =shared_Q8[ threadIdx.x];
		hlp_Q9[ index(z  ,y_s,x_w)] =shared_Q9[ threadIdx.x];
		hlp_Q10[index(z  ,y_s,x_e)] =shared_Q10[threadIdx.x];

		//......... in the x,z diagonals
		hlp_Q11[index(z_r,y  ,x_e)] =shared_Q11[threadIdx.x];
		hlp_Q12[index(z_r,y  ,x_w)] =shared_Q12[threadIdx.x];
		hlp_Q13[index(z_l,y  ,x_w)] =shared_Q13[threadIdx.x];
		hlp_Q14[index(z_l,y  ,x_e)] =shared_Q14[threadIdx.x];

		//......... in the y,z diagonals
		hlp_Q15[index(z_l,y_n,x  )] =shared_Q15[threadIdx.x];
		hlp_Q16[index(z_r,y_n,x  )] =shared_Q16[threadIdx.x];
		hlp_Q17[index(z_r,y_s,x  )] =shared_Q17[threadIdx.x];
		hlp_Q18[index(z_l,y_s,x  )] =shared_Q18[threadIdx.x];
	}
}

__global__
void streaming_kernel_first_part_Q0(int lx, int ly, int lz, const FLOATING *Q0,  FLOATING *hlp_Q0){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q0[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q0[index(z,y,x)] = shared_buffer[threadIdx.x];
}

__global__
void streaming_kernel_first_part_Q1(int lx, int ly, int lz, const FLOATING *Q1,  FLOATING *hlp_Q1){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/


	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int x_e/*east*/ =  (x+1)%lx; //1 8esh meta to trexon x

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q1[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q1[index(z  ,y  ,x_e)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q2(int lx, int ly, int lz, const FLOATING *Q2,  FLOATING *hlp_Q2){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/


	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int y_n/*north*/ =  (y+1)%ly; //1 8esh meta to trexon y

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q2[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q2[index(z  ,y_n,x  )] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q3(int lx, int ly, int lz, const FLOATING *Q3,  FLOATING *hlp_Q3){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int x_w/*west*/ =   (x+lx-1) %(  lx) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q3[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q3[index(z  ,y  ,x_w)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q4(int lx, int ly, int lz, const FLOATING *Q4,  FLOATING *hlp_Q4){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/


	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int y_s/*south*/ =   (y+ly-1) %(  ly) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q4[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q4[index(z  ,y_s,x  )] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q5(int lx, int ly, int lz, const FLOATING *Q5,  FLOATING *hlp_Q5){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;
	int z_l/*left*/ =  (z+1)%lz;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q5[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q5[index(z_l,y  ,x  )] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q6(int lx, int ly, int lz, const FLOATING *Q6,  FLOATING *hlp_Q6){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;
	int z_r/*right*/ =   (z+lz-1) %(  lz) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q6[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q6[index(z_r,y  ,x  )] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q7(int lx, int ly, int lz, const FLOATING *Q7,  FLOATING *hlp_Q7){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int y_n/*north*/ =  (y+1)%ly;
	int x_e/*east*/ =  (x+1)%lx;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q7[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q7[ index(z  ,y_n,x_e)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q8(int lx, int ly, int lz, const FLOATING *Q8,  FLOATING *hlp_Q8){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/


	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int y_n/*north*/ =  (y+1)%ly;
	int x_w/*west*/ =   (x+lx-1) %(  lx) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q8[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q8[ index(z  ,y_n,x_w)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q9(int lx, int ly, int lz, const FLOATING *Q9,  FLOATING *hlp_Q9){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int y_s/*south*/ =   (y+ly-1) %(  ly) ;
	int x_w/*west*/ =   (x+lx-1) %(  lx) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q9[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q9[ index(z  ,y_s,x_w)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q10(int lx, int ly, int lz, const FLOATING *Q10,  FLOATING *hlp_Q10){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int y_s/*south*/ =   (y+ly-1) %(  ly) ;
	int x_e/*east*/ =  (x+1)%lx;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q10[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q10[index(z  ,y_s,x_e)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q11(int lx, int ly, int lz, const FLOATING *Q11,  FLOATING *hlp_Q11){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int z_r/*right*/ =   (z+lz-1) %(  lz) ;
	int x_e/*east*/ =  (x+1)%lx;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q11[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q11[index(z_r,y  ,x_e)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q12(int lx, int ly, int lz, const FLOATING *Q12,  FLOATING *hlp_Q12){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int	z_r/*right*/ =   (z+lz-1) %(  lz) ;
	int x_w/*west*/ =   (x+lx-1) %(  lx) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q12[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q12[index(z_r,y  ,x_w)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q13(int lx, int ly, int lz, const FLOATING *Q13,  FLOATING *hlp_Q13){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;


	int z_l/*left*/ =  (z+1)%lz;
	int x_w/*west*/ =   (x+lx-1) %(  lx) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q13[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q13[index(z_l,y  ,x_w)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q14(int lx, int ly, int lz, const FLOATING *Q14,  FLOATING *hlp_Q14){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;



	int z_l/*left*/ =  (z+1)%lz;
	int x_e/*east*/ =  (x+1)%lx;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q14[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q14[index(z_l,y  ,x_e)] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q15(int lx, int ly, int lz, const FLOATING *Q15,  FLOATING *hlp_Q15){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int z_l/*left*/ =  (z+1)%lz;
	int y_n/*north*/ =  (y+1)%ly;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q15[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q15[index(z_l,y_n,x  )] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q16(int lx, int ly, int lz, const FLOATING *Q16,  FLOATING *hlp_Q16){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int z_r /*right*/ =   (z+lz-1) %(  lz) ;
	int y_n /*north*/=  (y+1)%ly;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q16[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q16[index(z_r,y_n,x  )] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q17(int lx, int ly, int lz, const FLOATING *Q17,  FLOATING *hlp_Q17){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int	z_r/*right*/ =   (z+lz-1) %(  lz) ;
	int y_s/*south*/ =   (y+ly-1) %(  ly) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q17[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q17[index(z_r,y_s,x  )] = shared_buffer[threadIdx.x];

}

__global__
void streaming_kernel_first_part_Q18(int lx, int ly, int lz, const FLOATING *Q18,  FLOATING *hlp_Q18){
	/*Propagate fluid densities to their next neighbour nodes */
	/*c....density propagation: all fluid densities are propagated from
	c       non-occupied nodes along the lattice connection lines
	c       to their next neighbours.*/
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int end_of_memory=lz*ly*(lx);
	int z=(int) (tid/(ly*lx));
	int	y=(tid-z*(ly*lx))/lx;
	int	x=tid-z*(ly*lx)-y*lx;

	int z_l/*left*/ =  (z+1)%lz;
	int y_s/*south*/ =   (y+ly-1) %(  ly) ;

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=Q18[index(z,y,x)];
	__syncthreads();

	if( tid<end_of_memory)
		hlp_Q18[index(z_l,y_s,x  )] = shared_buffer[threadIdx.x];
}



__global__
void streaming_kernel_single_threaded_p2_v2(int lx, int ly, int lz,
		FLOATING *hlp_Q0, FLOATING *hlp_Q1, FLOATING *hlp_Q2, FLOATING *hlp_Q3,
		FLOATING *hlp_Q4, FLOATING *hlp_Q5, FLOATING *hlp_Q6, FLOATING *hlp_Q7,
		FLOATING *hlp_Q8, FLOATING *hlp_Q9, FLOATING *hlp_Q10, FLOATING *hlp_Q11,
		FLOATING *hlp_Q12, FLOATING *hlp_Q13, FLOATING *hlp_Q14, FLOATING *hlp_Q15,
		FLOATING *hlp_Q16, FLOATING *hlp_Q17, FLOATING *hlp_Q18,
		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18){

	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int z=tid/ly;
	int  y=tid-z*ly;

	if(tid<ly*lz){


		//		for (z = 0 ; z< lz ; ++z){
		//loop for x=0 and x=lx-1!!!! (first and last slice)
		//			for (y = 0; y< ly; ++y){
		//toslice 0 antigrafetai sto 0
		hlp_Q0[index(z,y,0)]=Q0[index(z,y,0)];
		hlp_Q1[index(z,y,0)]=Q1[index(z,y,0)];
		hlp_Q2[index(z,y,0)]=Q2[index(z,y,0)];
		hlp_Q3[index(z,y,0)]=Q3[index(z,y,0)];
		hlp_Q4[index(z,y,0)]=Q4[index(z,y,0)];
		hlp_Q5[index(z,y,0)]=Q5[index(z,y,0)];
		hlp_Q6[index(z,y,0)]=Q6[index(z,y,0)];
		hlp_Q7[index(z,y,0)]=Q7[index(z,y,0)];
		hlp_Q8[index(z,y,0)]=Q8[index(z,y,0)];
		hlp_Q9[index(z,y,0)]=Q9[index(z,y,0)];
		hlp_Q10[index(z,y,0)]=Q10[index(z,y,0)];
		hlp_Q11[index(z,y,0)]=Q11[index(z,y,0)];
		hlp_Q12[index(z,y,0)]=Q12[index(z,y,0)];
		hlp_Q13[index(z,y,0)]=Q13[index(z,y,0)];
		hlp_Q14[index(z,y,0)]=Q14[index(z,y,0)];
		hlp_Q15[index(z,y,0)]=Q15[index(z,y,0)];
		hlp_Q16[index(z,y,0)]=Q16[index(z,y,0)];
		hlp_Q17[index(z,y,0)]=Q17[index(z,y,0)];
		hlp_Q18[index(z,y,0)]=Q18[index(z,y,0)];
		//at x= lx I set the incomming density as the one (equilibrum) calculated after the collision
		hlp_Q3[index(z,y,lx-1)] = Q3[index(z,y,lx-1)];
		hlp_Q8[index(z,y,lx-1)] = Q8[index(z,y,lx-1)] ;
		hlp_Q9[index(z,y,lx-1)] = Q9[index(z,y,lx-1)] ;
		hlp_Q12[index(z,y,lx-1)] = Q12[index(z,y,lx-1)] ;
		hlp_Q13[index(z,y,lx-1)] = Q13[index(z,y,lx-1)] ;
		//			gia x=lx-1: the densities 0,   2,4,5,6,15,16,17,18
		//				aplws 8a metadw8oune
		//				"ka8eta" sto slice kai de 8a ginoun propagate se
		//				alla slices (tuxainei na voleuei auto)
		//				auto to kommati exei HDH ginei pio panw!
		//
		//
		//				during streaming some of the indices at
		//				(fixed) x=lx-1, y=ly-1, do not get updated.
		//				it doesn't matter since the last slice on x is not useful.


	}
}

void LBM::cuda_streaming(){

	if(data_location==CPU)
		copy_data_from_host_to_device();

	dim3 threads_type2(threads_for_streaming_collision_and_relaxation,1,1);
	dim3 grid_type2(blocks_for_streaming_collision_and_relaxation,1,1);

	streaming_kernel_first_part_Q0<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q0,  D3_hlp_d.Q0);
	streaming_kernel_first_part_Q1<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q1,  D3_hlp_d.Q1);
	streaming_kernel_first_part_Q2<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q2,  D3_hlp_d.Q2);
	streaming_kernel_first_part_Q3<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q3,  D3_hlp_d.Q3);
	streaming_kernel_first_part_Q4<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q4,  D3_hlp_d.Q4);
	streaming_kernel_first_part_Q5<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q5,  D3_hlp_d.Q5);
	streaming_kernel_first_part_Q6<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q6,  D3_hlp_d.Q6);
	streaming_kernel_first_part_Q7<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q7,  D3_hlp_d.Q7);
	streaming_kernel_first_part_Q8<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q8,  D3_hlp_d.Q8);
	streaming_kernel_first_part_Q9<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q9,  D3_hlp_d.Q9);
	streaming_kernel_first_part_Q10<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q10,  D3_hlp_d.Q10);
	streaming_kernel_first_part_Q11<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q11,  D3_hlp_d.Q11);
	streaming_kernel_first_part_Q12<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q12,  D3_hlp_d.Q12);
	streaming_kernel_first_part_Q13<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q13,  D3_hlp_d.Q13);
	streaming_kernel_first_part_Q14<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q14,  D3_hlp_d.Q14);
	streaming_kernel_first_part_Q15<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q15,  D3_hlp_d.Q15);
	streaming_kernel_first_part_Q16<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q16,  D3_hlp_d.Q16);
	streaming_kernel_first_part_Q17<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q17,  D3_hlp_d.Q17);
	streaming_kernel_first_part_Q18<<<grid_type2, threads_type2, size_of_allocated_shared_memory_for_streaming_collision_and_relaxation>>>( lx,  ly,  lz, D3_d.Q18,  D3_hlp_d.Q18);
	hipDeviceSynchronize();

	if(data_location==CPU)
		copy_data_from_host_to_device();

	int n_of_threads=64;
	int n_of_blocks=(lz*ly)/n_of_threads;
	if ( (lattice_nodes%n_of_threads)!=0 )
		++n_of_blocks;

	dim3 threads_type3(n_of_threads,1,1);
	dim3 grid_type3(n_of_blocks,1,1);

	streaming_kernel_single_threaded_p2_v2<<<grid_type3, threads_type3>>>(lx,ly,lz,
			D3_hlp_d.Q0, D3_hlp_d.Q1, D3_hlp_d.Q2, D3_hlp_d.Q3,
			D3_hlp_d.Q4, D3_hlp_d.Q5, D3_hlp_d.Q6, D3_hlp_d.Q7,
			D3_hlp_d.Q8, D3_hlp_d.Q9, D3_hlp_d.Q10, D3_hlp_d.Q11,
			D3_hlp_d.Q12, D3_hlp_d.Q13, D3_hlp_d.Q14, D3_hlp_d.Q15,
			D3_hlp_d.Q16, D3_hlp_d.Q17, D3_hlp_d.Q18,
			D3_d.Q0, D3_d.Q1, D3_d.Q2, D3_d.Q3,
			D3_d.Q4, D3_d.Q5, D3_d.Q6, D3_d.Q7,
			D3_d.Q8, D3_d.Q9, D3_d.Q10, D3_d.Q11,
			D3_d.Q12, D3_d.Q13, D3_d.Q14, D3_d.Q15,
			D3_d.Q16, D3_d.Q17, D3_d.Q18);

	hipDeviceSynchronize();
}

