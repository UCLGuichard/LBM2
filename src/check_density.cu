#include "hip/hip_runtime.h"
#include "global_defines.cuh"
#include "kernels3.cuh"

FLOATING LBM::sum_microscopic_densities_for_a_single_node(const int x, const int y, const int z){
	FLOATING sum = 0.0;
	sum = sum + D3.Q0[index(z,y,x)];

	sum+=D3.Q1[index(z,y,x)];
	sum+=D3.Q2[index(z,y,x)];
	sum+=D3.Q3[index(z,y,x)];
	sum+=D3.Q4[index(z,y,x)];
	sum+=D3.Q5[index(z,y,x)];
	sum+=D3.Q6[index(z,y,x)];

	sum+=D3.Q7[index(z,y,x)];
	sum+=D3.Q8[index(z,y,x)];
	sum+=D3.Q9[index(z,y,x)];
	sum+=D3.Q10[index(z,y,x)];
	sum+=D3.Q11[index(z,y,x)];
	sum+=D3.Q12[index(z,y,x)];
	sum+=D3.Q13[index(z,y,x)];
	sum+=D3.Q14[index(z,y,x)];
	sum+=D3.Q15[index(z,y,x)];
	sum+=D3.Q16[index(z,y,x)];
	sum+=D3.Q17[index(z,y,x)];
	sum+=D3.Q18[index(z,y,x)];

	return sum;
}




int LBM::calculate_macroscopic_density_in_the_host(FLOATING &density){
	/*compute integral density*/

	if(data_location==GPU)
		copy_data_from_device_to_host();
	//.....local variables
	int  x,y,z;
	FLOATING n_sum=0.0;

	//.....loop over computational domain
	//...........loop over all densities

	//loop only for the last quarter of the domain
#pragma unroll
	for ( z =0 ; z < lz ; ++z){
#pragma unroll
		for (y = 0; y < ly ; ++y){
#pragma unroll
			for ( x = (lx*3/4)-1 ; x< lx ; ++x){

				n_sum+=sum_microscopic_densities_for_a_single_node(x,y,z);

			}
		}
	}





	cout.setf(ios::fixed,ios::floatfield);
	cout.precision(10);
	cout << "density check: Integral density=" << n_sum << " " << n_sum/(lz*ly*(lx/4))  << endl;
	//check for nan

	if( isnan(n_sum/(lx*lz*ly))==0 ){
		//it is NOT nan, it is a good number!
		density=n_sum/(lz*ly*(lx/4));
		return 1;
	}else{
		//NAN WAS FOUND:the rest of this branch is for debugging purposes (print the nan on an external file)
		cout << " nsum:" << n_sum << endl;
#pragma unroll
		for ( z =0 ; z < lz ; ++z){
#pragma unroll
			for (y = 0; y < ly ; ++y){
#pragma unroll
				for ( x = (lx*3/4)-1 ; x< lx ; ++x){
					n_sum+=sum_microscopic_densities_for_a_single_node(x,y,z);

					if(isnan(n_sum)==0)
						cout << " x,y,z: " << x << " " << y << " " << z << endl;
				}
			}
		}

		return 0;

	}

}


FLOATING sum_one_density(int big_array_length, int numThreads, int numBlocks, int whichKernel, FLOATING *D3_Q,FLOATING* h_odata, FLOATING* d_odata){
	FLOATING partial_sum=0.0;

	reduce<FLOATING>(big_array_length, numThreads, numBlocks, whichKernel, D3_Q, d_odata);
	hipDeviceSynchronize();
	hipMemcpy( h_odata, d_odata, numBlocks*sizeof(FLOATING), hipMemcpyDeviceToHost);
	for(int i=0; i<numBlocks; i++)
		partial_sum+= h_odata[i];

	return partial_sum;
}


__global__
void simple_check_density(const FLOATING *input_array, const int lx, const int ly, const int lz, FLOATING *output_array){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;

	int z=(int) (tid/(ly*lx));
	int	y=(int) (tid-z*(ly*lx))/lx;
	int	x=(int) tid-z*(ly*lx)-y*lx;

	if( (3/4*lx-1)<=x and x< lx)
		output_array[tid]=input_array[tid];
}

FLOATING LBM::calulate_partial_sum(const FLOATING *input_array_d){
	FLOATING partial_sum=0.0;
	//calculate cuda threads and blocks
	//	simple_check_density<<< 1,1 >>> (input_array_d, 680, 73, 73, temp_check_density_d_full);
	partial_sum=reduce_sum(temp_check_density_d_full, 680*73*73);
	return partial_sum;
}


__global__
void collect_data_convergence_interest(const CUDA_FLOATING *input_array, const int lx, const int ly, const int lz,
		FLOATING *output_array){

	//collects a range of elements from input_array and inserts them into the output
	//then the reduce will be applied upon these data
	int tid=blockIdx.x*blockDim.x+threadIdx.x;

	int z_short= (int)(4*tid)/(lx*ly);
	int y_short=(int) (4*tid-z_short*lx*ly) / lx ;
	int x_short=((4*tid+3)%lx);

	extern __shared__ FLOATING shared_buffer[];

	shared_buffer[threadIdx.x]=input_array[index(z_short,y_short,x_short)];
	__syncthreads();

	output_array[tid]=shared_buffer[threadIdx.x];

	//	output_array[tid]=input_array[index(z_short,y_short,x_short)];



	__syncthreads();

}

FLOATING LBM::reduce_a_fraction_of_the_domain(const CUDA_FLOATING *domain_of_interest_d){

	int lattice_nodes=lz*ly*lx/4;
	int n_of_threads=threads_per_kernel;
	int n_of_blocks= (lattice_nodes)/n_of_threads;

	if( (lattice_nodes%n_of_threads)!=0)
		++n_of_blocks;

	const int size_of_allocated_shared_memory=(n_of_threads)*sizeof(CUDA_FLOATING);

	dim3 threads_type2(n_of_threads,1,1);
	dim3 grid_type2(n_of_blocks,1,1);

	collect_data_convergence_interest<<<grid_type2, threads_type2,size_of_allocated_shared_memory>>>(domain_of_interest_d,
			lx, ly,  lz, temp_check_density_d);

	FLOATING partial_sum=reduce_sum(temp_check_density_d, lz*ly*lx/4);
	return partial_sum;


}

void LBM::cuda_check_density(const int iteration){

//	int const array_length=(lx*ly*lz);
//	int const FLOATING_array_size=array_length*sizeof(FLOATING);



	FLOATING n_sum=0.0;


	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q0);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q1);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q2);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q3);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q4);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q5);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q6);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q7);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q8);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q9);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q10);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q11);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q12);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q13);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q14);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q15);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q16);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q17);
	n_sum+=reduce_a_fraction_of_the_domain(D3_d.Q18);
	//	n_sum=1.1;

	FLOATING temp_cpu_density=0.0;
	//CPU WORKING
//	if(data_location==GPU)
//		copy_data_from_device_to_host();
//	if( check_density(temp_cpu_density)==0)
//		exit(10);

	//save results in file - start
	const int quarter_length=lx*ly*lz/4;

	char buffer1[256];
	snprintf(buffer1, sizeof(buffer1), "LBM2_%s_convergence.log", case_name.c_str());

#ifdef PRODUCE_OUTPUT_FILES
	ofstream convergence_file;
	convergence_file.open( buffer1);
	//convergence_file.open( buffer1 , ofstream::app);
	convergence_file.precision(10);
#endif //PRODUCE_OUTPUT_FILES

	if (n_sum!=0){
		cout <<" cuda Integral density:" << n_sum/quarter_length << endl;
#ifdef PRODUCE_OUTPUT_FILES
		convergence_file<< iteration << "\t" << n_sum/quarter_length << "\t" << temp_cpu_density << endl;
#endif //PRODUCE_OUTPUT_FILES
	}else{
		cout <<" n_sum=0! potentially wrong! check code!" << endl;
#ifdef PRODUCE_OUTPUT_FILES
		convergence_file<< iteration << "\t" << n_sum/quarter_length << " WARNING! "<< endl;
#endif //PRODUCE_OUTPUT_FILES
	}
#ifdef PRODUCE_OUTPUT_FILES
	convergence_file.close();
#endif //PRODUCE_OUTPUT_FILES
	//save results in file - end

}
