#include "hip/hip_runtime.h"
#include "global_defines.cuh"
#include "kernels3.cuh"


void	LBM::convective_BC(){

	if(data_location==GPU)
		copy_data_from_device_to_host();

	int  y,z,num;

	FLOATING  u_x,u_y,u_z,u_n[19];//,n_equ[19];
	FLOATING	u_squ,rho,Uc;



	//.....first compute the mean outflow velocity, Uc
	Uc = 0.0;
	num = 0;
	for (z = 0 ; z< lz ; ++z){
		for (y = 0 ; y< ly ; ++y){
			if (obstacles[index(z,y,(lx-1))]==0) {
				Uc +=   u_current[index2D(z,y)];
				++num ;
			}
		}
	}
	Uc /= num; //!if (num>0) check not needed


	cout << " CCCCPU(convective_BC, U_C_avg) Uc:"  << Uc << endl;

	for (z = 0 ; z< lz ; ++z){
		for (y = 0 ; y< ly ; ++y){
			if (!obstacles[index(z,y,(lx-1))]) {

				//.....compute the new velocities (based on convective BC)
				//originally proposed by Djenidi
//				u_current[index2D(z,y)] = (u_previous_temporal_boundary[index2D(z,y)] + Uc*u_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
//				u_previous_temporal_boundary[index2D(z,y)] = u_current[index2D(z,y)];
//				v_current[index2D(z,y)] = (v_previous_temporal_boundary[index2D(z,y)] + Uc*v_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
//				v_previous_temporal_boundary[index2D(z,y)] = v_current[index2D(z,y)];
//				w_current[index2D(z,y)] = (w_previous_temporal_boundary[index2D(z,y)] + Uc*w_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
//				w_previous_temporal_boundary[index2D(z,y)] = w_current[index2D(z,y)];

				////.....compute the new velocities (based on NON convective BC)
				//suggested by Timos on 04102015
				u_current[index2D(z,y)] = Uc;
				v_current[index2D(z,y)] = 0;
				w_current[index2D(z,y)] = 0;


				rho=0.0;
				rho+=D3_hlp.Q0[index(z,y,lx-1)]+D3_hlp.Q1[index(z,y,lx-1)]+D3_hlp.Q2[index(z,y,lx-1)]+D3_hlp.Q3[index(z,y,lx-1)];
				rho+=D3_hlp.Q4[index(z,y,lx-1)]+D3_hlp.Q5[index(z,y,lx-1)]+D3_hlp.Q6[index(z,y,lx-1)]+D3_hlp.Q7[index(z,y,lx-1)];
				rho+=D3_hlp.Q8[index(z,y,lx-1)]+D3_hlp.Q9[index(z,y,lx-1)]+D3_hlp.Q10[index(z,y,lx-1)]+D3_hlp.Q11[index(z,y,lx-1)];
				rho+=D3_hlp.Q12[index(z,y,lx-1)]+D3_hlp.Q13[index(z,y,lx-1)]+D3_hlp.Q14[index(z,y,lx-1)]+D3_hlp.Q15[index(z,y,lx-1)];
				rho+=D3_hlp.Q16[index(z,y,lx-1)]+D3_hlp.Q17[index(z,y,lx-1)]+D3_hlp.Q18[index(z,y,lx-1)];

				u_x = u_current[index2D(z,y)];
				u_y = v_current[index2D(z,y)];
				u_z = w_current[index2D(z,y)];

				//...........square velocity
				u_squ = u_x * u_x + u_y * u_y + u_z * u_z;
				/*
								c...........n- velocity compnents (n = lattice node connection vectors)
								c...........this is only necessary for clearence, and only 3 speeds would
								c...........be necessary
				 */
				u_n[0]= 0.0; //SHOULD NEVER USED!
				u_n[1] =   u_x; //u_xs
				u_n[2] =         u_y;
				u_n[3] = - u_x;
				u_n[4] =       - u_y;
				u_n[5] =   u_z;
				u_n[6] =       - u_z;
				u_n[7] =   u_x + u_y;
				u_n[8] = - u_x + u_y;
				u_n[9] = - u_x - u_y;
				u_n[10] =   u_x - u_y;
				u_n[11] =   u_x - u_z;
				u_n[12] = - u_x - u_z;
				u_n[13] = - u_x + u_z;
				u_n[14] =   u_x + u_z;
				u_n[15] =   u_z + u_y;
				u_n[16] = - u_z + u_y;
				u_n[17] = - u_z - u_y;
				u_n[18] =   u_z - u_y;


				/*c...........equilibrium densities
					c...........zero velocity density
					c*/
				//original part!
				//				n_equ[0] = (FLOATING) (t_0  * rho*(1.0 - u_squ / (2.0 * c_squ)));
				//
				//
				//				//...........axis speeds (factor: t_1)
				//				//TODO: NA GINEI SUGXWNEUSH SE ENA CASE ME FOR
				//				for (i = 1 ; i< 7 ; ++i)
				//					n_equ[i] = (FLOATING) (t_1 * rho*(1.0 + u_n[i] / c_squ
				//							+ ( u_n[i] * u_n[i]) / (2.0 * (c_squ * c_squ))
				//							- u_squ / (2.0 * c_squ)));
				//
				//				//...........diagonal speeds (factor: t_2)
				//				for (i = 7 ; i< 19 ; ++i)
				//					n_equ[i] =  (FLOATING) (t_2 * rho*(1.0 + u_n[i] / c_squ
				//							+ ( u_n[i] * u_n[i]) / (2.0 * (c_squ * c_squ))
				//							- u_squ / (2.0 * c_squ)));
				//
				//
				//
				//				D3.Q0[index(z,y,lx-1)]=(FLOATING) n_equ[0];
				//
				//				//...........axis speeds (factor: t_1)
				//
				//				D3.Q1[index(z,y,lx-1)]=(FLOATING) n_equ[1];
				//				D3.Q2[index(z,y,lx-1)]=(FLOATING) n_equ[2];
				//				D3.Q3[index(z,y,lx-1)]=(FLOATING) n_equ[3];
				//				D3.Q4[index(z,y,lx-1)]=(FLOATING) n_equ[4];
				//				D3.Q5[index(z,y,lx-1)]=(FLOATING) n_equ[5];
				//				D3.Q6[index(z,y,lx-1)]=(FLOATING) n_equ[6];
				//
				//				//...........diagonal speeds (factor: t_2)
				//				D3.Q7[index(z,y,lx-1)]=(FLOATING) n_equ[7];
				//				D3.Q8[index(z,y,lx-1)]=(FLOATING) n_equ[8];
				//				D3.Q9[index(z,y,lx-1)]=(FLOATING) n_equ[9];
				//				D3.Q10[index(z,y,lx-1)]=(FLOATING) n_equ[10];
				//				D3.Q11[index(z,y,lx-1)]=(FLOATING) n_equ[11];
				//				D3.Q12[index(z,y,lx-1)]=(FLOATING) n_equ[12];
				//				D3.Q13[index(z,y,lx-1)]=(FLOATING) n_equ[13];
				//				D3.Q14[index(z,y,lx-1)]=(FLOATING) n_equ[14];
				//				D3.Q15[index(z,y,lx-1)]=(FLOATING) n_equ[15];
				//				D3.Q16[index(z,y,lx-1)]=(FLOATING) n_equ[16];
				//				D3.Q17[index(z,y,lx-1)]=(FLOATING) n_equ[17];
				//				D3.Q18[index(z,y,lx-1)]=(FLOATING) n_equ[18];

				//optimised!
				D3.Q0[index(z,y,lx-1)]=(t_0  * rho*(1.0 - u_squ / (2.0 * c_squ)));
				//...........axis speeds (factor: t_1)
				D3.Q1[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[1] / c_squ	+ ( u_n[1] * u_n[1]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q2[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[2] / c_squ	+ ( u_n[2] * u_n[2]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q3[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[3] / c_squ	+ ( u_n[3] * u_n[3]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q4[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[4] / c_squ	+ ( u_n[4] * u_n[4]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q5[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[5] / c_squ	+ ( u_n[5] * u_n[5]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q6[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[6] / c_squ	+ ( u_n[6] * u_n[6]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));

				//...........diagonal speeds (factor: t_2)
				D3.Q7[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[7] / c_squ 	+ ( u_n[7] * u_n[7]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q8[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[8] / c_squ 	+ ( u_n[8] * u_n[8]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q9[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[9] / c_squ 	+ ( u_n[9] * u_n[9]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q10[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[10] / c_squ 	+ ( u_n[10] * u_n[10]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q11[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[11] / c_squ 	+ ( u_n[11] * u_n[11]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q12[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[12] / c_squ 	+ ( u_n[12] * u_n[12]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q13[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[13] / c_squ 	+ ( u_n[13] * u_n[13]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q14[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[14] / c_squ 	+ ( u_n[14] * u_n[14]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q15[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[15] / c_squ 	+ ( u_n[15] * u_n[15]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q16[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[16] / c_squ 	+ ( u_n[16] * u_n[16]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q17[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[17] / c_squ 	+ ( u_n[17] * u_n[17]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
				D3.Q18[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[18] / c_squ 	+ ( u_n[18] * u_n[18]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));

			}
		}
	}

#ifdef DEBUG
	cout << " #LBM convective_bc OK!" << endl;
#endif

}






__global__
void	new_convective_BC_kernel_v1(FLOATING Uc, const int free_lattices_at_U_direction,int end_of_memory, int lx, int ly, int lz, FLOATING reynolds, FLOATING nu, FLOATING r_small,
		FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ, FLOATING omega, FLOATING one_minus_omega,
		FLOATING reciprocal_c_squ,
		FLOATING *hlp_Q0, FLOATING *hlp_Q1, FLOATING *hlp_Q2, FLOATING *hlp_Q3,
		FLOATING *hlp_Q4, FLOATING *hlp_Q5, FLOATING *hlp_Q6, FLOATING *hlp_Q7,
		FLOATING *hlp_Q8, FLOATING *hlp_Q9, FLOATING *hlp_Q10, FLOATING *hlp_Q11,
		FLOATING *hlp_Q12, FLOATING *hlp_Q13, FLOATING *hlp_Q14, FLOATING *hlp_Q15,
		FLOATING *hlp_Q16, FLOATING *hlp_Q17, FLOATING *hlp_Q18,
		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18,
		int *obstacles,
		FLOATING *u_previous_spatial_boundary, FLOATING *v_previous_spatial_boundary, FLOATING *w_previous_spatial_boundary,
		FLOATING *u_current, FLOATING *v_current, FLOATING *w_current,
		FLOATING *u_previous_temporal_boundary, FLOATING *v_previous_temporal_boundary, FLOATING *w_previous_temporal_boundary){

	if(blockIdx.x*blockDim.x+threadIdx.x==0){
		int  y,z  ;

		FLOATING  u_x,u_y,u_z,u_n[19];//,n_equ[19];
		FLOATING	u_squ,rho ;



		//.....first compute the mean outflow velocity, Uc
		//		Uc = 0.0;
		//
		//		for (z = 0 ; z< lz ; ++z){
		//			for (y = 0 ; y< ly ; ++y){
		////				if (obstacles[index(z,y,(lx-1))]==0) {
		//					Uc +=   u_current[index2D(z,y)];
		//
		////				}
		//			}
		//		}
		//		Uc /= free_lattices_at_U_direction; //!if (num>0) check not needed
		printf( "u-wise free lattices:%d \n",free_lattices_at_U_direction);

		printf( "within convective BC, Uc: %f\n", Uc);



		for (z = 0 ; z< lz ; ++z){
			for (y = 0 ; y< ly ; ++y){
				if (!obstacles[index(z,y,(lx-1))]) {

					//.....compute the new velocities (based on convective BC)
					u_current[index2D(z,y)] = (u_previous_temporal_boundary[index2D(z,y)] + Uc*u_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
					u_previous_temporal_boundary[index2D(z,y)] = u_current[index2D(z,y)];
					v_current[index2D(z,y)] = (v_previous_temporal_boundary[index2D(z,y)] + Uc*v_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
					v_previous_temporal_boundary[index2D(z,y)] = v_current[index2D(z,y)];
					w_current[index2D(z,y)] = (w_previous_temporal_boundary[index2D(z,y)] + Uc*w_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
					w_previous_temporal_boundary[index2D(z,y)] = w_current[index2D(z,y)];

					rho=hlp_Q0[index(z,y,lx-1)]+hlp_Q1[index(z,y,lx-1)]+hlp_Q2[index(z,y,lx-1)]+hlp_Q3[index(z,y,lx-1)]+
							hlp_Q4[index(z,y,lx-1)]+hlp_Q5[index(z,y,lx-1)]+hlp_Q6[index(z,y,lx-1)]+hlp_Q7[index(z,y,lx-1)]+
							hlp_Q8[index(z,y,lx-1)]+hlp_Q9[index(z,y,lx-1)]+hlp_Q10[index(z,y,lx-1)]+hlp_Q11[index(z,y,lx-1)]+
							hlp_Q12[index(z,y,lx-1)]+hlp_Q13[index(z,y,lx-1)]+hlp_Q14[index(z,y,lx-1)]+hlp_Q15[index(z,y,lx-1)]+
							hlp_Q16[index(z,y,lx-1)]+hlp_Q17[index(z,y,lx-1)]+hlp_Q18[index(z,y,lx-1)];


					u_x = u_current[index2D(z,y)];
					u_y = v_current[index2D(z,y)];
					u_z = w_current[index2D(z,y)];

					//...........square velocity
					u_squ = u_x * u_x + u_y * u_y + u_z * u_z;
					/*
									c...........n- velocity compnents (n = lattice node connection vectors)
									c...........this is only necessary for clearence, and only 3 speeds would
									c...........be necessary
					 */
					u_n[0]= 0.0; //SHOULD NEVER USED!
					u_n[1] =   u_x; //u_xs
					u_n[2] =         u_y;
					u_n[3] = - u_x;
					u_n[4] =       - u_y;
					u_n[5] =   u_z;
					u_n[6] =       - u_z;
					u_n[7] =   u_x + u_y;
					u_n[8] = - u_x + u_y;
					u_n[9] = - u_x - u_y;
					u_n[10] =   u_x - u_y;
					u_n[11] =   u_x - u_z;
					u_n[12] = - u_x - u_z;
					u_n[13] = - u_x + u_z;
					u_n[14] =   u_x + u_z;
					u_n[15] =   u_z + u_y;
					u_n[16] = - u_z + u_y;
					u_n[17] = - u_z - u_y;
					u_n[18] =   u_z - u_y;


					/*c...........equilibrium densities
						c...........zero velocity density
						c*/


					//optimised!
					Q0[index(z,y,lx-1)]=(t_0  * rho*(1.0 - u_squ / (2.0 * c_squ)));
					//...........axis speeds (factor: t_1)
					Q1[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[1] / c_squ	+ ( u_n[1] * u_n[1]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q2[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[2] / c_squ	+ ( u_n[2] * u_n[2]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q3[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[3] / c_squ	+ ( u_n[3] * u_n[3]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q4[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[4] / c_squ	+ ( u_n[4] * u_n[4]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q5[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[5] / c_squ	+ ( u_n[5] * u_n[5]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q6[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[6] / c_squ	+ ( u_n[6] * u_n[6]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));

					//...........diagonal speeds (factor: t_2)
					Q7[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[7] / c_squ 	+ ( u_n[7] * u_n[7]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q8[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[8] / c_squ 	+ ( u_n[8] * u_n[8]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q9[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[9] / c_squ 	+ ( u_n[9] * u_n[9]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q10[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[10] / c_squ 	+ ( u_n[10] * u_n[10]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q11[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[11] / c_squ 	+ ( u_n[11] * u_n[11]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q12[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[12] / c_squ 	+ ( u_n[12] * u_n[12]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q13[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[13] / c_squ 	+ ( u_n[13] * u_n[13]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q14[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[14] / c_squ 	+ ( u_n[14] * u_n[14]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q15[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[15] / c_squ 	+ ( u_n[15] * u_n[15]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q16[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[16] / c_squ 	+ ( u_n[16] * u_n[16]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q17[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[17] / c_squ 	+ ( u_n[17] * u_n[17]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
					Q18[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[18] / c_squ 	+ ( u_n[18] * u_n[18]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));

				}
			}
		}

	}
}

void compute_mean_outflow_velocity(const FLOATING *u_current_d, const int *obstacles_d){


}

__global__
void	new_convective_BC_kernel_v2(const FLOATING Uc, int end_of_memory, int lx, int ly, int lz, FLOATING reynolds, FLOATING nu, FLOATING r_small,
		FLOATING t_0, FLOATING t_1, FLOATING t_2, FLOATING c_squ, FLOATING omega, FLOATING one_minus_omega,
		FLOATING reciprocal_c_squ,
		const FLOATING *hlp_Q0, const FLOATING *hlp_Q1, const FLOATING *hlp_Q2, const FLOATING *hlp_Q3,
		const FLOATING *hlp_Q4, const FLOATING *hlp_Q5, const FLOATING *hlp_Q6, const FLOATING *hlp_Q7,
		const FLOATING *hlp_Q8, const FLOATING *hlp_Q9, const FLOATING *hlp_Q10, const FLOATING *hlp_Q11,
		const FLOATING *hlp_Q12, const FLOATING *hlp_Q13, const FLOATING *hlp_Q14, const FLOATING *hlp_Q15,
		const FLOATING *hlp_Q16, const FLOATING *hlp_Q17, const FLOATING *hlp_Q18,
		FLOATING *Q0, FLOATING *Q1, FLOATING *Q2, FLOATING *Q3,
		FLOATING *Q4, FLOATING *Q5, FLOATING *Q6, FLOATING *Q7,
		FLOATING *Q8, FLOATING *Q9, FLOATING *Q10, FLOATING *Q11,
		FLOATING *Q12, FLOATING *Q13, FLOATING *Q14, FLOATING *Q15,
		FLOATING *Q16, FLOATING *Q17, FLOATING *Q18,
		const int *obstacles,
		FLOATING *u_previous_spatial_boundary, FLOATING *v_previous_spatial_boundary, FLOATING *w_previous_spatial_boundary,
		FLOATING *u_current, FLOATING *v_current, FLOATING *w_current,
		FLOATING *u_previous_temporal_boundary, FLOATING *v_previous_temporal_boundary, FLOATING *w_previous_temporal_boundary){

	int tid=blockIdx.x*blockDim.x+threadIdx.x;

	int z=(int) tid / ly;
	int y=(int) tid % ly;

	if(tid<ly*lz){


		FLOATING  u_x,u_y,u_z,u_n[19];//,n_equ[19];
		FLOATING	u_squ,rho ;






		//		for (z = 0 ; z< lz ; ++z){
		//			for (y = 0 ; y< ly ; ++y){
		//				if (!obstacles[index(z,y,(lx-1))]) {

		//.....compute the new velocities (based on convective BC)
		u_current[index2D(z,y)] = (u_previous_temporal_boundary[index2D(z,y)] + Uc*u_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
		u_previous_temporal_boundary[index2D(z,y)] = u_current[index2D(z,y)];
		v_current[index2D(z,y)] = (v_previous_temporal_boundary[index2D(z,y)] + Uc*v_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
		v_previous_temporal_boundary[index2D(z,y)] = v_current[index2D(z,y)];
		w_current[index2D(z,y)] = (w_previous_temporal_boundary[index2D(z,y)] + Uc*w_previous_spatial_boundary[index2D(z,y)])/(1.0+Uc);
		w_previous_temporal_boundary[index2D(z,y)] = w_current[index2D(z,y)];

		rho=hlp_Q0[index(z,y,lx-1)]+hlp_Q1[index(z,y,lx-1)]+hlp_Q2[index(z,y,lx-1)]+hlp_Q3[index(z,y,lx-1)]+
				hlp_Q4[index(z,y,lx-1)]+hlp_Q5[index(z,y,lx-1)]+hlp_Q6[index(z,y,lx-1)]+hlp_Q7[index(z,y,lx-1)]+
				hlp_Q8[index(z,y,lx-1)]+hlp_Q9[index(z,y,lx-1)]+hlp_Q10[index(z,y,lx-1)]+hlp_Q11[index(z,y,lx-1)]+
				hlp_Q12[index(z,y,lx-1)]+hlp_Q13[index(z,y,lx-1)]+hlp_Q14[index(z,y,lx-1)]+hlp_Q15[index(z,y,lx-1)]+
				hlp_Q16[index(z,y,lx-1)]+hlp_Q17[index(z,y,lx-1)]+hlp_Q18[index(z,y,lx-1)];


		u_x = u_current[index2D(z,y)];
		u_y = v_current[index2D(z,y)];
		u_z = w_current[index2D(z,y)];

		//...........square velocity
		u_squ = u_x * u_x + u_y * u_y + u_z * u_z;
		/*
									c...........n- velocity compnents (n = lattice node connection vectors)
									c...........this is only necessary for clearence, and only 3 speeds would
									c...........be necessary
		 */
		u_n[0]= 0.0; //SHOULD NEVER USED!
		u_n[1] =   u_x; //u_xs
		u_n[2] =         u_y;
		u_n[3] = - u_x;
		u_n[4] =       - u_y;
		u_n[5] =   u_z;
		u_n[6] =       - u_z;
		u_n[7] =   u_x + u_y;
		u_n[8] = - u_x + u_y;
		u_n[9] = - u_x - u_y;
		u_n[10] =   u_x - u_y;
		u_n[11] =   u_x - u_z;
		u_n[12] = - u_x - u_z;
		u_n[13] = - u_x + u_z;
		u_n[14] =   u_x + u_z;
		u_n[15] =   u_z + u_y;
		u_n[16] = - u_z + u_y;
		u_n[17] = - u_z - u_y;
		u_n[18] =   u_z - u_y;


		/*c...........equilibrium densities
						c...........zero velocity density
						c*/


		//optimised!
		Q0[index(z,y,lx-1)]=(t_0  * rho*(1.0 - u_squ / (2.0 * c_squ)));
		//...........axis speeds (factor: t_1)
		Q1[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[1] / c_squ	+ ( u_n[1] * u_n[1]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q2[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[2] / c_squ	+ ( u_n[2] * u_n[2]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q3[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[3] / c_squ	+ ( u_n[3] * u_n[3]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q4[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[4] / c_squ	+ ( u_n[4] * u_n[4]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q5[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[5] / c_squ	+ ( u_n[5] * u_n[5]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q6[index(z,y,lx-1)]=(t_1 * rho*(1.0 + u_n[6] / c_squ	+ ( u_n[6] * u_n[6]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));

		//...........diagonal speeds (factor: t_2)
		Q7[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[7] / c_squ 	+ ( u_n[7] * u_n[7]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q8[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[8] / c_squ 	+ ( u_n[8] * u_n[8]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q9[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[9] / c_squ 	+ ( u_n[9] * u_n[9]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q10[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[10] / c_squ 	+ ( u_n[10] * u_n[10]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q11[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[11] / c_squ 	+ ( u_n[11] * u_n[11]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q12[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[12] / c_squ 	+ ( u_n[12] * u_n[12]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q13[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[13] / c_squ 	+ ( u_n[13] * u_n[13]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q14[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[14] / c_squ 	+ ( u_n[14] * u_n[14]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q15[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[15] / c_squ 	+ ( u_n[15] * u_n[15]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q16[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[16] / c_squ 	+ ( u_n[16] * u_n[16]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q17[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[17] / c_squ 	+ ( u_n[17] * u_n[17]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));
		Q18[index(z,y,lx-1)]=(t_2 * rho*(1.0 + u_n[18] / c_squ 	+ ( u_n[18] * u_n[18]) / (2.0 * (c_squ * c_squ)) - u_squ / (2.0 * c_squ)));

		//} //if (!obstacles[index(z,y,(lx-1))])
		//			} // for y
		//		} // for z

	}
}


void LBM::cuda_convective_BC(){
	if(data_location==CPU)
		copy_data_from_host_to_device();


	dim3 threads_type2(threads_per_kernel,1,1);
	dim3 grid_type2(convective_boundary_conditions_blocks,1,1);



	FLOATING temp_Uc=reduce_sum(u_current_temp_d, lz*ly)/no_obstacle_lattices_at_penultimate_x_slice;



	//simplest working implementation
	//		new_convective_BC_kernel_v1<<<grid_type1, threads_type1>>>(temp_Uc,no_obstacle_lattices_at_penultimate_x_slice, ly*lz,  lx,  ly,  lz,
	//				reynolds,  nu,  r_small, t_0,  t_1,  t_2,
	//				c_squ,  omega,  one_minus_omega, reciprocal_c_squ,
	//				D3_hlp_d.Q0, D3_hlp_d.Q1, D3_hlp_d.Q2, D3_hlp_d.Q3,
	//				D3_hlp_d.Q4, D3_hlp_d.Q5, D3_hlp_d.Q6, D3_hlp_d.Q7,
	//				D3_hlp_d.Q8, D3_hlp_d.Q9, D3_hlp_d.Q10, D3_hlp_d.Q11,
	//				D3_hlp_d.Q12, D3_hlp_d.Q13, D3_hlp_d.Q14, D3_hlp_d.Q15,
	//				D3_hlp_d.Q16, D3_hlp_d.Q17, D3_hlp_d.Q18,
	//				D3_d.Q0, D3_d.Q1, D3_d.Q2, D3_d.Q3,
	//				D3_d.Q4, D3_d.Q5, D3_d.Q6, D3_d.Q7,
	//				D3_d.Q8, D3_d.Q9, D3_d.Q10, D3_d.Q11,
	//				D3_d.Q12, D3_d.Q13, D3_d.Q14, D3_d.Q15,
	//				D3_d.Q16, D3_d.Q17, D3_d.Q18,
	//				obstacles_d,
	//				u_previous_spatial_boundary_d,  v_previous_spatial_boundary_d,  w_previous_spatial_boundary_d,
	//				u_current_d,  v_current_d,  w_current_d,
	//				u_previous_temporal_boundary_d,  v_previous_temporal_boundary_d,  w_previous_temporal_boundary_d);

	//average of u_current_d with no obstacles
	//
	//
	new_convective_BC_kernel_v2<<<grid_type2, threads_type2>>>(temp_Uc, ly*lz,  lx,  ly,  lz,
			reynolds,  nu,  r_small, t_0,  t_1,  t_2,
			c_squ,  omega,  one_minus_omega, reciprocal_c_squ,
			D3_hlp_d.Q0, D3_hlp_d.Q1, D3_hlp_d.Q2, D3_hlp_d.Q3,
			D3_hlp_d.Q4, D3_hlp_d.Q5, D3_hlp_d.Q6, D3_hlp_d.Q7,
			D3_hlp_d.Q8, D3_hlp_d.Q9, D3_hlp_d.Q10, D3_hlp_d.Q11,
			D3_hlp_d.Q12, D3_hlp_d.Q13, D3_hlp_d.Q14, D3_hlp_d.Q15,
			D3_hlp_d.Q16, D3_hlp_d.Q17, D3_hlp_d.Q18,
			D3_d.Q0, D3_d.Q1, D3_d.Q2, D3_d.Q3,
			D3_d.Q4, D3_d.Q5, D3_d.Q6, D3_d.Q7,
			D3_d.Q8, D3_d.Q9, D3_d.Q10, D3_d.Q11,
			D3_d.Q12, D3_d.Q13, D3_d.Q14, D3_d.Q15,
			D3_d.Q16, D3_d.Q17, D3_d.Q18,
			obstacles_d,
			u_previous_spatial_boundary_d,  v_previous_spatial_boundary_d,  w_previous_spatial_boundary_d,
			u_current_d,  v_current_d,  w_current_d,
			u_previous_temporal_boundary_d,  v_previous_temporal_boundary_d,  w_previous_temporal_boundary_d);


	hipDeviceSynchronize();

}
